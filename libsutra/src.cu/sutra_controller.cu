#include "hip/hip_runtime.h"
#include <hip/hip_math_constants.h>
#include <sutra_controller.h>
/*
  _  __                    _
 | |/ /___ _ __ _ __   ___| |___
 | ' // _ \ '__| '_ \ / _ \ / __|
 | . \  __/ |  | | | |  __/ \__ \
 |_|\_\___|_|  |_| |_|\___|_|___/

 */

__global__ void shift_krnl(float *data, int offset, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    data[tid] = data[tid + offset * N];
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void mult_krnl(float *i_data, float *scale, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    i_data[tid] = i_data[tid] * scale[tid];
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void mult_krnl(float *i_data, float *scale, float gain, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    i_data[tid] = i_data[tid] * scale[tid] * gain;
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void mult_krnl(float *i_data, float gain, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    i_data[tid] = i_data[tid] * gain;
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void mult_int_krnl(float *o_data, float *i_data, float gain, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    o_data[tid] = gain * i_data[tid] + o_data[tid];
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void mult_int_krnl(float *o_data, float *i_data, float *scale,
                              float gain, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    o_data[tid] = gain * (i_data[tid] * scale[tid]) + o_data[tid];
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void mult_int_krnl(float *o_data, float *i_data, float *scale,
                              float gain, int N, int istart) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  tid += istart;

  while (tid < N) {
    o_data[tid] = gain * (i_data[tid] * scale[tid]) + o_data[tid];
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void add_md_krnl(float *o_matrix, float *i_matrix, float *i_vector,
                            int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    o_matrix[tid * (N + 1)] = i_matrix[tid * (N + 1)] + i_vector[tid];
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void TT_filt_krnl(float *mat, int n, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid < N) {
    tid % (n + 1) ? mat[tid] *= -1.0f : mat[tid] = (1.0f - mat[tid]);
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void fill_filtmat_krnl(float *filter, int nactu, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid < N) {
    filter[tid] =
        tid % (nactu + 1) ? (float)-1. / nactu : (float)(1. - 1. / nactu);
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void fill_cmat_krnl(float *cmat, float *wtt, float *Mtt, long nact,
                               long nslope, long N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int i, j;
  while (tid < N) {
    i = tid / nact;
    j = tid - i * nact;
    // if(j < nact-2) cmat[tid] = 1.0f;
    cmat[tid] =
        (j < nact - 2) ? wtt[j + i * (nact - 2)] : Mtt[j - (nact - 2) + i * 2];
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void do_statcov_krnl(float *statcov, float *xpos, float *ypos,
                                float norm, long dim, long N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int i, j;
  while (tid < N) {
    i = tid / dim;
    j = tid - i * dim;
    statcov[i * dim + j] =
        6.88 *
        powf(sqrtf((xpos[i] - xpos[j]) * (xpos[i] - xpos[j]) +
                   (ypos[i] - ypos[j]) * (ypos[i] - ypos[j])),
             5. / 3.) *
        norm;
    tid += blockDim.x * gridDim.x;
  }
}
template <class T>
__global__ void pupphase_krnl(T *o_data, float *i_data, int *indx_pup, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid < N) {
    o_data[tid] = (T)i_data[indx_pup[tid]];
    tid += blockDim.x * gridDim.x;
  }
}

__device__ hipComplex exp_complex(hipComplex z) {
  hipComplex res;
  float t = expf(z.x);
  sincosf(z.y, &res.y, &res.x);
  res.x *= t;
  res.y *= t;

  return res;
}

__global__ void compute_Hcor_krnl(float *o_data, int nrow, int ncol, float Fs,
                                  float Te, float gmin, float gmax,
                                  float delay) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int j = tid / nrow;
  int i = tid - j * nrow;

  float step = (Fs / 2.0f - (Fs / (2 * ncol))) / (ncol - 1);
  float f = Fs / (2 * ncol) + j * step;
  float G = gmin + i * (gmax - gmin) / (nrow - 1);
  hipFloatComplex pTe = make_hipFloatComplex(0.0f, 2 * HIP_PI_F * f * Te);
  hipFloatComplex moins_pTe =
      make_hipFloatComplex(0.0f, -2 * HIP_PI_F * f * Te);
  hipFloatComplex pTe2 = hipCmulf(pTe, pTe);
  hipFloatComplex UnMoinsepTe = make_hipFloatComplex(
      1.0f - exp_complex(moins_pTe).x, -exp_complex(moins_pTe).y);
  hipFloatComplex pdelay =
      make_hipFloatComplex(0.0f, -2 * HIP_PI_F * f * Te * delay);

  hipFloatComplex res = hipCdivf(UnMoinsepTe, pTe2);
  hipFloatComplex Hbo = hipCmulf(res, exp_complex(pdelay));
  Hbo.x = 1 + G * Hbo.x;
  Hbo.y *= G;
  float mod = hipCabsf(Hbo);

  o_data[tid] = 1.0f / (mod * mod);
}

__global__ void absnormfft_krnl(hipFloatComplex *idata, float *odata, int N,
                                float norm) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  hipFloatComplex cache;
  while (tid < N) {
    cache = idata[tid + 1];  // Reject first element (0 frequency)
    odata[tid] = (cache.x * cache.x + cache.y * cache.y) * norm;
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void adjust_csrcol_krnl(int *colind, int *nnz, int Nphi,
                                   int nnztot) {
  int tid = nnz[0] + threadIdx.x + blockIdx.x * blockDim.x;
  int i = 1;
  int N = nnz[0] + nnz[i];

  if (tid < nnztot) {
    while (tid > N) {
      i++;
      N += nnz[i];
    }
    __syncthreads();

    colind[tid] += i * Nphi;
  }
}

__global__ void adjust_csrrow_krnl(int *rowind, int *nact, int *nnz,
                                   int nact_tot) {
  int tid = nact[0] + threadIdx.x + blockIdx.x * blockDim.x;
  int i = 1;
  int N = nact[0] + nact[1];

  if (tid < nact_tot) {
    while (tid > N) {
      i++;
      N += nact[i];
    }
    __syncthreads();

    rowind[tid] += nnz[i - 1];
  }
}

/*
  _                           _
 | |    __ _ _   _ _ __   ___| |__   ___ _ __ ___
 | |   / _` | | | | '_ \ / __| '_ \ / _ \ '__/ __|
 | |__| (_| | |_| | | | | (__| | | |  __/ |  \__ \
 |_____\__,_|\__,_|_| |_|\___|_| |_|\___|_|  |___/

 */

int shift_buf(float *d_data, int offset, int N, carma_device *device) {
  int nBlocks, nThreads;
  getNumBlocksAndThreads(device, N, nBlocks, nThreads);
  dim3 grid(nBlocks), threads(nThreads);

  shift_krnl<<<grid, threads>>>(d_data, offset, N);

  carmaCheckMsg("shift_kernel<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}

int mult_vect(float *d_data, float *scale, int N, carma_device *device) {
  int nBlocks, nThreads;
  getNumBlocksAndThreads(device, N, nBlocks, nThreads);
  dim3 grid(nBlocks), threads(nThreads);

  mult_krnl<<<grid, threads>>>(d_data, scale, N);

  carmaCheckMsg("mult_kernel<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}

int mult_vect(float *d_data, float *scale, float gain, int N,
              carma_device *device) {
  int nBlocks, nThreads;
  getNumBlocksAndThreads(device, N, nBlocks, nThreads);
  dim3 grid(nBlocks), threads(nThreads);

  mult_krnl<<<grid, threads>>>(d_data, scale, gain, N);

  carmaCheckMsg("mult_kernel<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}

int mult_vect(float *d_data, float gain, int N, carma_device *device) {
  int nBlocks, nThreads;
  getNumBlocksAndThreads(device, N, nBlocks, nThreads);
  dim3 grid(nBlocks), threads(nThreads);

  mult_krnl<<<grid, threads>>>(d_data, gain, N);

  carmaCheckMsg("mult_kernel<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}

int mult_int(float *o_data, float *i_data, float *scale, float gain, int N,
             carma_device *device, carma_streams *streams) {
  int nthreads = 0, nblocks = 0;

  int nstreams = streams->get_nbStreams();
  getNumBlocksAndThreads(device, N / nstreams, nblocks, nthreads);

  dim3 grid(nblocks), threads(nthreads);

  for (int i = 0; i < nstreams; i++) {
    mult_int_krnl<<<grid, threads, 0, streams->get_stream(i)>>>(
        o_data, i_data, scale, gain, N, i * nblocks * nthreads);
    carmaCheckMsg("multint_kernel<<<>>> execution failed\n");
  }

  return EXIT_SUCCESS;
}

int mult_int(float *o_data, float *i_data, float *scale, float gain, int N,
             carma_device *device) {
  int nthreads = 0, nblocks = 0;

  getNumBlocksAndThreads(device, N, nblocks, nthreads);

  dim3 grid(nblocks), threads(nthreads);

  mult_int_krnl<<<grid, threads>>>(o_data, i_data, scale, gain, N);
  carmaCheckMsg("multint_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

int mult_int(float *o_data, float *i_data, float gain, int N,
             carma_device *device) {
  int nthreads = 0, nblocks = 0;

  getNumBlocksAndThreads(device, N, nblocks, nthreads);

  dim3 grid(nblocks), threads(nthreads);

  mult_int_krnl<<<grid, threads>>>(o_data, i_data, gain, N);
  carmaCheckMsg("multint_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

int add_md(float *o_matrix, float *i_matrix, float *i_vector, int N,
           carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  add_md_krnl<<<grid, threads>>>(o_matrix, i_matrix, i_vector, N);
  carmaCheckMsg("add_md_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

int fill_filtmat(float *filter, int nactu, int N, carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  fill_filtmat_krnl<<<grid, threads>>>(filter, nactu, N);
  carmaCheckMsg("fill_filtmat_krnl<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}
int TT_filt(float *mat, int n, carma_device *device) {
  int nthreads = 0, nblocks = 0;
  int N = n * n;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);
  TT_filt_krnl<<<grid, threads>>>(mat, n, N);
  carmaCheckMsg("TT_filt_krnl<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

int fill_cmat(float *cmat, float *wtt, float *Mtt, int nactu, int nslopes,
              carma_device *device) {
  int nthreads = 0, nblocks = 0;
  int N = nactu * nslopes;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  fill_cmat_krnl<<<grid, threads>>>(cmat, wtt, Mtt, nactu, nslopes, N);
  carmaCheckMsg("fill_cmat_krnl<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

int do_statmat(float *statcov, long dim, float *xpos, float *ypos, float norm,
               carma_device *device) {
  int nthreads = 0, nblocks = 0;
  int N = (dim * dim);
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);
  do_statcov_krnl<<<grid, threads>>>(statcov, xpos, ypos, norm, dim, N);
  carmaCheckMsg("do_statcov_krnl<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

template <class T>
int get_pupphase(T *o_data, float *i_data, int *indx_pup, int Nphi,
                 carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, Nphi, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);
  pupphase_krnl<<<grid, threads>>>(o_data, i_data, indx_pup, Nphi);
  carmaCheckMsg("pupphase_krnl<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}
template int get_pupphase<float>(float *o_data, float *i_data, int *indx_pup,
                                 int Nphi, carma_device *device);
template int get_pupphase<double>(double *o_data, float *i_data, int *indx_pup,
                                  int Nphi, carma_device *device);

int compute_Hcor_gpu(float *o_data, int nrow, int ncol, float Fs, float gmin,
                     float gmax, float delay, carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, nrow * ncol, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  compute_Hcor_krnl<<<grid, threads>>>(o_data, nrow, ncol, Fs, 1.0f / Fs, gmin,
                                       gmax, delay);
  carmaCheckMsg("compute_Hcor_krnl<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

int absnormfft(hipFloatComplex *idata, float *odata, int N, float norm,
               carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  absnormfft_krnl<<<grid, threads>>>(idata, odata, N, norm);
  carmaCheckMsg("absnormfft_krnl<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}

int adjust_csr_index(int *rowind, int *NNZ, int *nact, int nact_tot,
                     int row_off, carma_device *device) {
  int N = nact_tot - row_off;
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid2(nblocks), threads2(nthreads);

  adjust_csrrow_krnl<<<grid2, threads2>>>(rowind, nact, NNZ, nact_tot);

  return EXIT_SUCCESS;
}
