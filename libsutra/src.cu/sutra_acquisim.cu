#include "hip/hip_runtime.h"
#include <sutra_acquisim.h>
#include <sutra_ao_utils.h>

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template<class T> struct SharedMemory
{
    __device__ inline operator       T*()
    {
        extern __shared__ int __smem[];
        return (T*)__smem;
    }

    __device__ inline operator const T*() const
    {
        extern __shared__ int __smem[];
        return (T*)__smem;
    }
};

// specialize for double to avoid unaligned memory
// access compile errors
template<> struct SharedMemory<double>
{
    __device__ inline operator       double*()
    {
        extern __shared__ double __smem_d[];
        return (double*)__smem_d;
    }

    __device__ inline operator const double*() const
    {
        extern __shared__ double __smem_d[];
        return (double*)__smem_d;
    }
};

__global__ void bcube_krnl(float *bimage, float *bcube, int npix, int npix2, int nsub, int *ivalid, int *jvalid, int N)
{
  /*
    indx is an array nrebin^2 * npix^2
    it gives the nrebin x nrebin pixels in the hrimage per npix x npix pixels of the subap
    Npix = npix x npix
   */
  int tid     = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    int nim = tid / npix2;
    int tidim = tid - nim * npix2;
    int xim = tidim % npix;
    int yim = tidim / npix;

    int idbin = xim + yim * nsub + ivalid[nim] * npix + jvalid[nim] * npix * nsub;
    bcube[tid] = bimage[idbin];
    tid  += blockDim.x * gridDim.x;
  }
}

int fillbincube(float *bimage, float *bcube, int npix, int nsub, int Nsub, int *ivalid, int *jvalid, int device)
{
  int Npix = npix * npix;
  int N = Npix * nsub;
  int nthreads = 0,nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);

  dim3 grid(nblocks), threads(nthreads);

  bcube_krnl<<<grid, threads>>>(bimage,bcube,npix,Npix,Nsub,ivalid,jvalid,N);

  cutilCheckMsg("binimg_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void bcube_krnl_async(float *bimage, float *bcube, int npix, int npix2, int nsub, int *ivalid, int *jvalid, int N, int idstart)
{
  /*
    indx is an array nrebin^2 * npix^2
    it gives the nrebin x nrebin pixels in the hrimage per npix x npix pixels of the subap
    Npix = npix x npix
   */
  int tid     = threadIdx.x + blockIdx.x * blockDim.x;
  tid += idstart;

  while (tid < N) {
    int nim = tid / npix2;
    int tidim = tid - nim * npix2;
    int xim = tidim % npix;
    int yim = tidim / npix;

    int idbin = xim + yim * nsub + ivalid[nim] * npix + jvalid[nim] * npix * nsub;
    bcube[tid] = bimage[idbin];
    tid  += blockDim.x * gridDim.x;
  }
}

int fillbincube_async(carma_host_obj<float> *image_telemetry, float *bimage, float *bcube, int npix, int nsub, int Nsub, int *ivalid, int *jvalid, int nim, int device)
{
  float *hdata = image_telemetry->getData();
  int nstreams = image_telemetry->get_nbStreams();

  int Npix = npix * npix;
  int N = Npix * nsub;
  int nthreads = 0,nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);

  // here nstreams should be : final image size / npix
  dim3 threads(nthreads);
  dim3 grid(N/(nstreams*threads.x));

  // asynchronously launch nstreams kernels, each operating on its own portion of data
  for(int i = 0; i < nstreams; i++){
	  hipMemcpyAsync(&(bimage[i*nim/nstreams]), &(hdata[i*nim/nstreams]), sizeof(float) * nim / nstreams,
			  hipMemcpyHostToDevice, image_telemetry->get_cudaStream_t(i));
    bcube_krnl_async<<<grid, threads, 0, image_telemetry->get_cudaStream_t(i)>>>(bimage,bcube,npix,Npix,Nsub,ivalid,jvalid,N,i*N/nstreams);
    // asynchronously launch nstreams memcopies.  Note that memcopy in stream x will only
    //   commence executing when all previous CUDA calls in stream x have completed
  }
  //hipStreamSynchronize(image_telemetry->get_cudaStream_t(nstreams-1));
  cutilCheckMsg("binimg_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

int fillbincube_async(carma_streams *streams, carma_obj<float> *bimage, carma_obj<float> *bcube, int npix, int nsub, int Nsub, int *ivalid, int *jvalid, int device)
{
  float *g_image = bimage->getData();
  float *g_cube = bcube->getData();
  int nstreams = streams->get_nbStreams();

  int Npix = npix * npix;
  int N = Npix * nsub;
  int nthreads = 0,nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);

  // here nstreams should be : final image size / npix
  dim3 threads(nthreads);
  dim3 grid(N/(nstreams*threads.x));

  // asynchronously launch nstreams kernels, each operating on its own portion of data
  for(int i = 0; i < nstreams; i++){

    bcube_krnl_async<<<grid, threads, 0, streams->get_stream(i)>>>(g_image,g_cube,npix,Npix,Nsub,ivalid,jvalid,N,i*N/nstreams);
    // asynchronously launch nstreams memcopies.  Note that memcopy in stream x will only
    //   commence executing when all previous CUDA calls in stream x have completed
  }

  cutilCheckMsg("binimg_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}


