#include "hip/hip_runtime.h"
#include <sutra_centroider_wcog.h>
#include <carma_utils.cuh>

template <class T>
__global__ void fillweights_krnl(T *d_out, T *weights, int Npix, int N) {
  int nim, idx;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    nim = tid / Npix;
    idx = tid - nim * Npix;
    d_out[tid] = weights[idx];
    tid += blockDim.x * gridDim.x;
  }
}

template <class T>
int fillweights(T *d_out, T *d_in, int npix, int N, carma_device *device) {
  int nBlocks, nThreads;
  getNumBlocksAndThreads(device, N, nBlocks, nThreads);
  dim3 grid(nBlocks), threads(nThreads);

  fillweights_krnl<<<grid, threads>>>(d_out, d_in, npix * npix, N);
  carmaCheckMsg("<<<fillweights_krnl>>> execution failed\n");

  return EXIT_SUCCESS;
}

template int fillweights<float>(float *d_out, float *d_in, int npix, int N,
                                carma_device *device);

template int fillweights<double>(double *d_out, double *d_in, int npix, int N,
                                 carma_device *device);

template <class T>
__global__ void centroidx(T *g_idata, T *g_odata, T *alpha, T *weights,
                          unsigned int n, unsigned int N, T scale, T offset,
                          unsigned int nelem_thread) {
  T *sdata = SharedMemory<T>();

  // load shared mem
  unsigned int tid = threadIdx.x;
  // unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  // unsigned int x = (tid % n) + 1;
  unsigned int x;
  int idim;
  sdata[tid] = 0.0f;
  for (int cc = 0; cc < nelem_thread; cc++) {
    x = ((tid * nelem_thread + cc) % n);
    idim = tid * nelem_thread + cc + (blockDim.x * nelem_thread) * blockIdx.x;
    if (idim < N)
      sdata[tid] += g_idata[idim] * x * weights[idim];
    else
      sdata[tid] += 0.0f;
  }
  __syncthreads();

  // sdata[tid] = (i < N) ? g_idata[i] * x * weights[i] : 0;

  __syncthreads();

  reduce_krnl(sdata, blockDim.x, tid);
  // if(tid == 0)
  //	printf("blockIdx %d sdata %f \n",blockIdx.x,sdata[tid]);
  // write result for this block to global mem
  if (tid == 0)
    g_odata[blockIdx.x] =
        ((sdata[tid] * 1.0 / (alpha[blockIdx.x] + 1.e-6)) - offset) * scale;
}

template <class T>
__global__ void centroidy(T *g_idata, T *g_odata, T *alpha, T *weights,
                          unsigned int n, unsigned int N, T scale, T offset,
                          unsigned int nelem_thread) {
  T *sdata = SharedMemory<T>();

  // load shared mem
  unsigned int tid = threadIdx.x;
  // unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  // unsigned int y = (tid / n) + 1;
  unsigned int y;
  int idim;
  sdata[tid] = 0.0f;
  for (int cc = 0; cc < nelem_thread; cc++) {
    y = ((tid * nelem_thread + cc) / n);
    idim = tid * nelem_thread + cc + (blockDim.x * nelem_thread) * blockIdx.x;
    if (idim < N)
      sdata[tid] += g_idata[idim] * y * weights[idim];
    else
      sdata[tid] += 0.0f;
  }

  // sdata[tid] = (i < N) ? g_idata[i] * y * weights[i] : 0;

  __syncthreads();

  reduce_krnl(sdata, blockDim.x, tid);

  // write result for this block to global mem
  if (tid == 0)
    g_odata[blockIdx.x] =
        ((sdata[tid] * 1.0 / (alpha[blockIdx.x] + 1.e-6)) - offset) * scale;
}

template <class T>
void get_centroids(int size, int threads, int blocks, int n, T *d_idata,
                   T *d_odata, T *alpha, T *weights, T scale, T offset,
                   carma_device *device) {
  int maxThreads = device->get_properties().maxThreadsPerBlock;
  unsigned int nelem_thread = 1;
  while ((threads / nelem_thread > maxThreads) ||
         (threads % nelem_thread != 0)) {
    nelem_thread++;
  }

  threads /= nelem_thread;
  dim3 dimBlock(threads, 1, 1);
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  int smemSize =
      (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);
  centroidx<T><<<dimGrid, dimBlock, smemSize>>>(
      d_idata, d_odata, alpha, weights, n, size, scale, offset, nelem_thread);

  carmaCheckMsg("centroidx_kernel<<<>>> execution failed\n");

  centroidy<T><<<dimGrid, dimBlock, smemSize>>>(d_idata, &(d_odata[blocks]),
                                                alpha, weights, n, size, scale,
                                                offset, nelem_thread);

  carmaCheckMsg("centroidy_kernel<<<>>> execution failed\n");
}

template void get_centroids<float>(int size, int threads, int blocks, int n,
                                   float *d_idata, float *d_odata, float *alpha,
                                   float *weights, float scale, float offset,
                                   carma_device *device);

template void get_centroids<double>(int size, int threads, int blocks, int n,
                                    double *d_idata, double *d_odata,
                                    double *alpha, double *weights,
                                    double scale, double offset,
                                    carma_device *device);
