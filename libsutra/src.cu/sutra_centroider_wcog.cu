#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
//  This file is part of COMPASS <https://anr-compass.github.io/compass/>
//
//  Copyright (C) 2011-2019 COMPASS Team <https://github.com/ANR-COMPASS>
//  All rights reserved.
//  Distributed under GNU - LGPL
//
//  COMPASS is free software: you can redistribute it and/or modify it under the terms of the GNU Lesser
//  General Public License as published by the Free Software Foundation, either version 3 of the License,
//  or any later version.
//
//  COMPASS: End-to-end AO simulation tool using GPU acceleration
//  The COMPASS platform was designed to meet the need of high-performance for the simulation of AO systems.
//
//  The final product includes a software package for simulating all the critical subcomponents of AO,
//  particularly in the context of the ELT and a real-time core based on several control approaches,
//  with performances consistent with its integration into an instrument. Taking advantage of the specific
//  hardware architecture of the GPU, the COMPASS tool allows to achieve adequate execution speeds to
//  conduct large simulation campaigns called to the ELT.
//
//  The COMPASS platform can be used to carry a wide variety of simulations to both testspecific components
//  of AO of the E-ELT (such as wavefront analysis device with a pyramid or elongated Laser star), and
//  various systems configurations such as multi-conjugate AO.
//
//  COMPASS is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the
//  implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
//  See the GNU Lesser General Public License for more details.
//
//  You should have received a copy of the GNU Lesser General Public License along with COMPASS.
//  If not, see <https://www.gnu.org/licenses/lgpl-3.0.txt>.
// -----------------------------------------------------------------------------

//! \file      sutra_centroider_wcog.cu
//! \ingroup   libsutra
//! \class     SutraCentroiderWcog
//! \brief     this class provides the centroider_wcog features to COMPASS
//! \author    COMPASS Team <https://github.com/ANR-COMPASS>
//! \version   5.1.0
//! \date      2011/01/28
//! \copyright GNU Lesser General Public License

#include <sutra_centroider_wcog.h>
#include <carma_utils.cuh>

template <class T>
__global__ void fillweights_krnl(T *d_out, T *weights, int Npix, int N) {
  int nim, idx;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    nim = tid / Npix;
    idx = tid - nim * Npix;
    d_out[tid] = weights[idx];
    tid += blockDim.x * gridDim.x;
  }
}

template <class T>
int fill_weights(T *d_out, T *d_in, int npix, int N, CarmaDevice *device) {
  int nb_blocks, nb_threads;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);
  dim3 grid(nb_blocks), threads(nb_threads);

  fillweights_krnl<<<grid, threads>>>(d_out, d_in, npix * npix, N);
  carma_check_msg("<<<fillweights_krnl>>> execution failed\n");

  return EXIT_SUCCESS;
}

template int fill_weights<float>(float *d_out, float *d_in, int npix, int N,
                                CarmaDevice *device);

template int fill_weights<double>(double *d_out, double *d_in, int npix, int N,
                                 CarmaDevice *device);

template <class T, int nb_threads>
__global__ void centroids(float *d_img, T *d_centroids, T *ref, int *validx,
                          int *validy, float *d_intensities, float *weights,
                          unsigned int npix, unsigned int size, float scale,
                          float offset, unsigned int nelem_thread) {
  if (blockDim.x > nb_threads) {
    if (threadIdx.x == 0) printf("Wrong size argument\n");
    return;
  }
  // Specialize BlockReduce for a 1D block of 128 threads on type int
  typedef hipcub::BlockReduce<float, nb_threads> BlockReduce;
  // Allocate shared memory for BlockReduce
  __shared__ typename BlockReduce::TempStorage temp_storage;

  float idata = 0;
  float xdata = 0;
  float ydata = 0;
  // load shared mem
  unsigned int tid = threadIdx.x;
  unsigned int xvalid = validx[blockIdx.x];
  unsigned int yvalid = validy[blockIdx.x];
  unsigned int x, y;
  int idim, wdim;

  for (int cc = 0; cc < nelem_thread; cc++) {
    x = ((tid * nelem_thread + cc) % npix);
    y = ((tid * nelem_thread + cc) / npix);
    // idim = tid * nelem_thread + cc + (blockDim.x * nelem_thread) *
    // blockIdx.x;
    idim = (x + xvalid) + (y + yvalid) * size;
    wdim = x + y * npix + blockIdx.x * npix * npix;
    if (idim < size * size) {
      idata += d_img[idim] * weights[wdim];
      xdata += d_img[idim] * x * weights[wdim];
      ydata += d_img[idim] * y * weights[wdim];
    }
  }

  // sdata[tid] = (i < N) ? g_idata[i] * x : 0;
  __syncthreads();

  float intensity = BlockReduce(temp_storage).Sum(idata, blockDim.x);
  __syncthreads();
  float slopex = BlockReduce(temp_storage).Sum(xdata, blockDim.x);
  __syncthreads();
  float slopey = BlockReduce(temp_storage).Sum(ydata, blockDim.x);

  // write result for this block to global mem
  if (tid == 0) {
    d_centroids[blockIdx.x] =
        (T(slopex * 1.0 / (intensity + 1.e-6)) - offset) * scale -
        ref[blockIdx.x];
    d_centroids[blockIdx.x + gridDim.x] =
        (T(slopey * 1.0 / (intensity + 1.e-6)) - offset) * scale -
        ref[blockIdx.x + gridDim.x];
    d_intensities[blockIdx.x] = intensity;
  }
}

template <class T>
void get_centroids(int size, int threads, int blocks, int npix, float *d_img,
                   T *d_centroids, T *ref, int *validx, int *validy,
                   float *intensities, float *weights, float scale,
                   float offset, CarmaDevice *device) {
  int maxThreads = device->get_properties().maxThreadsPerBlock;
  unsigned int nelem_thread = 1;
  while ((threads / nelem_thread > maxThreads) ||
         (threads % nelem_thread != 0)) {
    nelem_thread++;
  }

  threads /= nelem_thread;
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  if (threads <= 16)
    centroids<T, 16><<<dimGrid, 16>>>(d_img, d_centroids, ref, validx, validy,
                                      intensities, weights, npix, size, scale,
                                      offset, nelem_thread);
  else if (threads <= 36)
    centroids<T, 36><<<dimGrid, 36>>>(d_img, d_centroids, ref, validx, validy,
                                      intensities, weights, npix, size, scale,
                                      offset, nelem_thread);

  else if (threads <= 64)
    centroids<T, 64><<<dimGrid, 64>>>(d_img, d_centroids, ref, validx, validy,
                                      intensities, weights, npix, size, scale,
                                      offset, nelem_thread);
  else if (threads <= 100)
    centroids<T, 100><<<dimGrid, 100>>>(d_img, d_centroids, ref, validx, validy,
                                        intensities, weights, npix, size, scale,
                                        offset, nelem_thread);
  else if (threads <= 144)
    centroids<T, 144><<<dimGrid, 144>>>(d_img, d_centroids, ref, validx, validy,
                                        intensities, weights, npix, size, scale,
                                        offset, nelem_thread);
  else if (threads <= 256)
    centroids<T, 256><<<dimGrid, 256>>>(d_img, d_centroids, ref, validx, validy,
                                        intensities, weights, npix, size, scale,
                                        offset, nelem_thread);
  else if (threads <= 512)
    centroids<T, 512><<<dimGrid, 512>>>(d_img, d_centroids, ref, validx, validy,
                                        intensities, weights, npix, size, scale,
                                        offset, nelem_thread);
  else if (threads <= 1024)
    centroids<T, 1024><<<dimGrid, 1024>>>(d_img, d_centroids, ref, validx,
                                          validy, intensities, weights, npix,
                                          size, scale, offset, nelem_thread);
  else
    printf("SH way too big !!!\n");

  carma_check_msg("centroids_kernel<<<>>> execution failed\n");
}

template void get_centroids<float>(int size, int threads, int blocks, int npix,
                                   float *d_img, float *d_centroids, float *ref,
                                   int *validx, int *validy, float *intensities,
                                   float *weights, float scale, float offset,
                                   CarmaDevice *device);

template void get_centroids<double>(int size, int threads, int blocks, int npix,
                                    float *d_img, double *d_centroids,
                                    double *ref, int *validx, int *validy,
                                    float *intensities, float *weights,
                                    float scale, float offset,
                                    CarmaDevice *device);

// template <class T>
// __global__ void centroidx(T *g_idata, T *g_odata, T *alpha, T *weights,
//                           unsigned int n, unsigned int N, float scale, float
//                           offset, unsigned int nelem_thread) {
//   T *sdata = SharedMemory<T>();

//   // load shared mem
//   unsigned int tid = threadIdx.x;
//   // unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
//   // unsigned int x = (tid % n) + 1;
//   unsigned int x;
//   int idim;
//   sdata[tid] = 0.0f;
//   for (int cc = 0; cc < nelem_thread; cc++) {
//     x = ((tid * nelem_thread + cc) % n);
//     idim = tid * nelem_thread + cc + (blockDim.x * nelem_thread) *
//     blockIdx.x; if (idim < N)
//       sdata[tid] += g_idata[idim] * x * weights[idim];
//     else
//       sdata[tid] += 0.0f;
//   }
//   __syncthreads();

//   // sdata[tid] = (i < N) ? g_idata[i] * x * weights[i] : 0;

//   __syncthreads();

//   reduce_krnl(sdata, blockDim.x, tid);
//   // if(tid == 0)
//   //	printf("blockIdx %d sdata %f \n",blockIdx.x,sdata[tid]);
//   // write result for this block to global mem
//   if (tid == 0)
//     g_odata[blockIdx.x] =
//         ((sdata[tid] * 1.0 / (alpha[blockIdx.x] + 1.e-6)) - offset) * scale;
// }

// template <class T>
// __global__ void centroidy(T *g_idata, T *g_odata, T *alpha, T *weights,
//                           unsigned int n, unsigned int N, float scale, float
//                           offset, unsigned int nelem_thread) {
//   T *sdata = SharedMemory<T>();

//   // load shared mem
//   unsigned int tid = threadIdx.x;
//   // unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
//   // unsigned int y = (tid / n) + 1;
//   unsigned int y;
//   int idim;
//   sdata[tid] = 0.0f;
//   for (int cc = 0; cc < nelem_thread; cc++) {
//     y = ((tid * nelem_thread + cc) / n);
//     idim = tid * nelem_thread + cc + (blockDim.x * nelem_thread) *
//     blockIdx.x; if (idim < N)
//       sdata[tid] += g_idata[idim] * y * weights[idim];
//     else
//       sdata[tid] += 0.0f;
//   }

//   // sdata[tid] = (i < N) ? g_idata[i] * y * weights[i] : 0;

//   __syncthreads();

//   reduce_krnl(sdata, blockDim.x, tid);

//   // write result for this block to global mem
//   if (tid == 0)
//     g_odata[blockIdx.x] =
//         ((sdata[tid] * 1.0 / (alpha[blockIdx.x] + 1.e-6)) - offset) * scale;
// }

// template <class T>
// void get_centroids(int size, int threads, int blocks, int n, T *d_idata,
//                    T *d_odata, T *alpha, T *weights, float scale, float
//                    offset, CarmaDevice *device) {
//   int maxThreads = device->get_properties().maxThreadsPerBlock;
//   unsigned int nelem_thread = 1;
//   while ((threads / nelem_thread > maxThreads) ||
//          (threads % nelem_thread != 0)) {
//     nelem_thread++;
//   }

//   threads /= nelem_thread;
//   dim3 dimBlock(threads, 1, 1);
//   dim3 dimGrid(blocks, 1, 1);

//   // when there is only one warp per block, we need to allocate two warps
//   // worth of shared memory so that we don't index shared memory out of
//   bounds int smemSize =
//       (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);
//   centroidx<T><<<dimGrid, dimBlock, smemSize>>>(
//       d_idata, d_odata, alpha, weights, n, size, scale, offset,
//       nelem_thread);

//   carma_check_msg("centroidx_kernel<<<>>> execution failed\n");

//   centroidy<T><<<dimGrid, dimBlock, smemSize>>>(d_idata, &(d_odata[blocks]),
//                                                 alpha, weights, n, size,
//                                                 scale, offset, nelem_thread);

//   carma_check_msg("centroidy_kernel<<<>>> execution failed\n");
// }

// template void get_centroids<float>(int size, int threads, int blocks, int n,
//                                    float *d_idata, float *d_odata, float
//                                    *alpha, float *weights, float scale, float
//                                    offset, CarmaDevice *device);

// template void get_centroids<double>(int size, int threads, int blocks, int n,
//                                     double *d_idata, double *d_odata,
//                                     double *alpha, double *weights,
//                                     double scale, double offset,
//                                     CarmaDevice *device);
