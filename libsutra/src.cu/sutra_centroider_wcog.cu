#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
//  This file is part of COMPASS <https://anr-compass.github.io/compass/>
//
//  Copyright (C) 2011-2023 COMPASS Team <https://github.com/ANR-COMPASS>
//  All rights reserved.

// -----------------------------------------------------------------------------

//! \file      sutra_centroider_wcog.cu
//! \ingroup   libsutra
//! \class     SutraCentroiderWcog
//! \brief     this class provides the centroider_wcog features to COMPASS
//! \author    COMPASS Team <https://github.com/ANR-COMPASS>
//! \version   5.4.4
//! \date      2022/01/24

#include <sutra_centroider_wcog.h>
#include <sutra_centroider_utils.cuh>
#include <carma_utils.cuh>

template <class T>
__global__ void fillweights_krnl(T *d_out, T *weights, int Npix, int N) {
  int nim, idx;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    nim = tid / Npix;
    idx = tid - nim * Npix;
    d_out[tid] = weights[idx];
    tid += blockDim.x * gridDim.x;
  }
}

template <class T>
int fill_weights(T *d_out, T *d_in, int npix, int N, CarmaDevice *device) {
  int nb_blocks, nb_threads;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);
  dim3 grid(nb_blocks), threads(nb_threads);

  fillweights_krnl<<<grid, threads>>>(d_out, d_in, npix * npix, N);
  carma_check_msg("<<<fillweights_krnl>>> execution failed\n");

  return EXIT_SUCCESS;
}

template int fill_weights<float>(float *d_out, float *d_in, int npix, int N,
                                CarmaDevice *device);

template int fill_weights<double>(double *d_out, double *d_in, int npix, int N,
                                 CarmaDevice *device);

template <int nb_threads, typename T>
__global__ void centroids(float *d_img, T *d_centroids, T *ref, int *validx,
                          int *validy, float *d_intensities, float *weights, float threshold, 
                          unsigned int npix, sutra::SlopesIndex si, unsigned int size, float scale,
                          float offset, unsigned int nelem_thread) {
  if (blockDim.x > nb_threads) {
    if (threadIdx.x == 0) printf("Wrong size argument\n");
    return;
  }
  // Specialize BlockReduce for a 1D block of 128 threads on type int
  typedef hipcub::BlockReduce<float, nb_threads> BlockReduce;
  // Allocate shared memory for BlockReduce
  __shared__ typename BlockReduce::TempStorage temp_storage;

  float idata = 0;
  float xdata = 0;
  float ydata = 0;
  // load shared mem
  unsigned int tid = threadIdx.x;
  unsigned int xvalid = validx[blockIdx.x];
  unsigned int yvalid = validy[blockIdx.x];
  unsigned int x, y;
  int idim, wdim;

  for (int cc = 0; cc < nelem_thread; cc++) {
    x = ((tid * nelem_thread + cc) % npix);
    y = ((tid * nelem_thread + cc) / npix);
    // idim = tid * nelem_thread + cc + (blockDim.x * nelem_thread) *
    // blockIdx.x;
    idim = (x + xvalid) + (y + yvalid) * size;
    wdim = x + y * npix + blockIdx.x * npix * npix;
    if (idim < size * size) {
      float data_thresh =
          (d_img[idim] > threshold) ? d_img[idim] - threshold : 1.e-6;
      idata += data_thresh * weights[wdim];
      xdata += data_thresh * x * weights[wdim];
      ydata += data_thresh * y * weights[wdim];
    }
  }

  // sdata[tid] = (i < N) ? g_idata[i] * x : 0;
  __syncthreads();

  float intensity = BlockReduce(temp_storage).Sum(idata, npix * npix);
  __syncthreads();
  float slopex = BlockReduce(temp_storage).Sum(xdata, npix * npix);
  __syncthreads();
  float slopey = BlockReduce(temp_storage).Sum(ydata, npix * npix);

  // write result for this block to global mem
  if (tid == 0) {
    d_centroids[si.x(blockIdx.x)] = (T(slopex * 1.0 / (intensity + 1.e-6)) - offset) * scale - ref[si.x(blockIdx.x)];
    d_centroids[si.y(blockIdx.x)] = (T(slopey * 1.0 / (intensity + 1.e-6)) - offset) * scale - ref[si.y(blockIdx.x)];
    d_intensities[blockIdx.x] = intensity;
  }
}

template <class T>
void get_centroids(int size, int threads, int blocks, int npix, float *d_img,
                   T *d_centroids, T *ref, int *validx, int *validy,
                   float *intensities, float *weights, float threshold, float scale,
                   float offset,
                   SlopeOrder slope_order, CarmaDevice *device, hipStream_t stream) {
  int maxThreads = device->get_properties().maxThreadsPerBlock;
  unsigned int nelem_thread = 1;
  while ((threads / nelem_thread > maxThreads) ||
         (threads % nelem_thread != 0)) {
    nelem_thread++;
  }

  sutra::SlopesIndex si{blocks, slope_order};

  threads /= nelem_thread;
  dim3 dimGrid(blocks, 1, 1);
  // when there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  if (threads <= 16)
    centroids<  16><<<dimGrid, threads, 0, stream>>>(d_img, d_centroids, ref, validx, validy,
                                             intensities, weights, threshold, npix, si, size, scale,
                                             offset, nelem_thread);
  else if (threads <= 36)
    centroids<  36><<<dimGrid, threads, 0, stream>>>(d_img, d_centroids, ref, validx, validy,
                                             intensities, weights, threshold, npix, si, size, scale,
                                             offset, nelem_thread);
  else if (threads <= 64)
    centroids<  64><<<dimGrid, threads, 0, stream>>>(d_img, d_centroids, ref, validx, validy,
                                             intensities, weights, threshold, npix, si, size, scale,
                                             offset, nelem_thread);
  else if (threads <= 100)
    centroids< 100><<<dimGrid, threads, 0, stream>>>(d_img, d_centroids, ref, validx, validy,
                                             intensities, weights, threshold, npix, si, size, scale,
                                             offset, nelem_thread);
  else if (threads <= 144)
    centroids< 144><<<dimGrid, threads, 0, stream>>>(d_img, d_centroids, ref, validx, validy,
                                             intensities, weights, threshold, npix, si, size, scale,
                                             offset, nelem_thread);
  else if (threads <= 256)
    centroids< 256><<<dimGrid, threads, 0, stream>>>(d_img, d_centroids, ref, validx, validy,
                                             intensities, weights, threshold, npix, si, size, scale,
                                             offset, nelem_thread);
  else if (threads <= 512)
    centroids< 512><<<dimGrid, threads, 0, stream>>>(d_img, d_centroids, ref, validx, validy,
                                             intensities, weights, threshold, npix, si, size, scale,
                                             offset, nelem_thread);
  else if (threads <= 1024)
    centroids<1024><<<dimGrid, threads, 0, stream>>>(d_img, d_centroids, ref, validx,
                                             validy, intensities, weights, threshold, npix, si,
                                             size, scale, offset, nelem_thread);
  else
    printf("SH way too big !!!\n");

  carma_check_msg("centroids_kernel<<<>>> execution failed\n");
}

template void get_centroids<float>(int size, int threads, int blocks, int npix,
                                   float *d_img, float *d_centroids, float *ref,
                                   int *validx, int *validy, float *intensities,
                                   float *weights, float threshold, float scale, float offset,
                                   SlopeOrder slope_order,
                                   CarmaDevice *device, hipStream_t stream);

template void get_centroids<double>(int size, int threads, int blocks, int npix,
                                    float *d_img, double *d_centroids,
                                    double *ref, int *validx, int *validy,
                                    float *intensities, float *weights, float threshold,
                                    float scale, float offset,
                                    SlopeOrder slope_order,
                                    CarmaDevice *device, hipStream_t stream);
