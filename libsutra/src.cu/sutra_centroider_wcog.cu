#include "hip/hip_runtime.h"
// This file is part of COMPASS <https://github.com/COSMIC-RTC/compass>
//
// COMPASS is free software: you can redistribute it and/or modify it under the terms of the GNU Lesser
// General Public License as published by the Free Software Foundation, either version 3 of the 
// License, or any later version.
//
// COMPASS is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; 
// without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. 
// See the GNU Lesser General Public License for more details.
//
// You should have received a copy of the GNU Lesser General Public License along with COMPASS. 
// If not, see <https://www.gnu.org/licenses/>
//
//  Copyright (C) 2011-2024 COSMIC Team <https://github.com/COSMIC-RTC/compass>

//! \file      sutra_centroider_wcog.cu
//! \ingroup   libsutra
//! \class     SutraCentroiderWcog
//! \brief     this class provides the centroider_wcog features to COMPASS
//! \author    COSMIC Team <https://github.com/COSMIC-RTC/compass>
//! \date      2022/01/24

#include <sutra_centroider_wcog.hpp>
#include <sutra_centroider_utils.cuh>
#include <carma_utils.cuh>

template <class T>
__global__ void fillweights_krnl(T *d_out, T *weights, int32_t Npix, int32_t N) {
  int32_t nim, idx;
  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    nim = tid / Npix;
    idx = tid - nim * Npix;
    d_out[tid] = weights[idx];
    tid += blockDim.x * gridDim.x;
  }
}

template <class T>
int32_t fill_weights(T *d_out, T *d_in, int32_t npix, int32_t N, CarmaDevice *device) {
  int32_t nb_blocks, nb_threads;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);
  dim3 grid(nb_blocks), threads(nb_threads);

  fillweights_krnl<<<grid, threads>>>(d_out, d_in, npix * npix, N);
  carma_check_msg("<<<fillweights_krnl>>> execution failed\n");

  return EXIT_SUCCESS;
}

template int32_t fill_weights<float>(float *d_out, float *d_in, int32_t npix, int32_t N,
                                CarmaDevice *device);

template int32_t fill_weights<double>(double *d_out, double *d_in, int32_t npix, int32_t N,
                                 CarmaDevice *device);

template <int32_t nb_threads, typename T>
__global__ void centroids(float *d_img, T *d_centroids, T *ref, int32_t *validx,
                          int32_t *validy, float *d_intensities, float *weights, float threshold,
                          uint32_t npix, sutra::SlopesIndex si, uint32_t size, float scale,
                          float offset, uint32_t nelem_thread) {
  if (blockDim.x > nb_threads) {
    if (threadIdx.x == 0) printf("Wrong size argument\n");
    return;
  }
  // Specialize BlockReduce for a 1D block of 128 threads on type int32_t
  typedef hipcub::BlockReduce<float, nb_threads> BlockReduce;
  // Allocate shared memory for BlockReduce
  __shared__ typename BlockReduce::TempStorage temp_storage;

  float idata = 0;
  float xdata = 0;
  float ydata = 0;
  // load shared mem
  uint32_t tid = threadIdx.x;
  uint32_t xvalid = validx[blockIdx.x];
  uint32_t yvalid = validy[blockIdx.x];
  uint32_t x, y;
  int32_t idim, wdim;

  for (int32_t cc = 0; cc < nelem_thread; cc++) {
    x = ((tid * nelem_thread + cc) % npix);
    y = ((tid * nelem_thread + cc) / npix);
    // idim = tid * nelem_thread + cc + (blockDim.x * nelem_thread) *
    // blockIdx.x;
    idim = (x + xvalid) + (y + yvalid) * size;
    wdim = x + y * npix + blockIdx.x * npix * npix;
    if (idim < size * size) {
      float data_thresh =
          (d_img[idim] > threshold) ? d_img[idim] - threshold : 1.e-6;
      idata += data_thresh * weights[wdim];
      xdata += data_thresh * x * weights[wdim];
      ydata += data_thresh * y * weights[wdim];
    }
  }

  // sdata[tid] = (i < N) ? g_idata[i] * x : 0;
  __syncthreads();

  float intensity = BlockReduce(temp_storage).Sum(idata, npix * npix);
  __syncthreads();
  float slopex = BlockReduce(temp_storage).Sum(xdata, npix * npix);
  __syncthreads();
  float slopey = BlockReduce(temp_storage).Sum(ydata, npix * npix);

  // write result for this block to global mem
  if (tid == 0) {
    d_centroids[si.x(blockIdx.x)] = (T(slopex * 1.0 / (intensity + 1.e-6)) - offset) * scale - ref[si.x(blockIdx.x)];
    d_centroids[si.y(blockIdx.x)] = (T(slopey * 1.0 / (intensity + 1.e-6)) - offset) * scale - ref[si.y(blockIdx.x)];
    d_intensities[blockIdx.x] = intensity;
  }
}

template <class T>
void get_centroids(int32_t size, int32_t threads, int32_t blocks, int32_t npix, float *d_img,
                   T *d_centroids, T *ref, int32_t *validx, int32_t *validy,
                   float *intensities, float *weights, float threshold, float scale,
                   float offset,
                   SlopeOrder slope_order, CarmaDevice *device, hipStream_t stream) {
  int32_t maxThreads = device->get_properties().maxThreadsPerBlock;
  uint32_t nelem_thread = 1;
  while ((threads / nelem_thread > maxThreads) ||
         (threads % nelem_thread != 0)) {
    nelem_thread++;
  }

  sutra::SlopesIndex si{blocks, slope_order};

  threads /= nelem_thread;
  dim3 dimGrid(blocks, 1, 1);
  // when there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  if (threads <= 16)
    centroids<  16><<<dimGrid, threads, 0, stream>>>(d_img, d_centroids, ref, validx, validy,
                                             intensities, weights, threshold, npix, si, size, scale,
                                             offset, nelem_thread);
  else if (threads <= 36)
    centroids<  36><<<dimGrid, threads, 0, stream>>>(d_img, d_centroids, ref, validx, validy,
                                             intensities, weights, threshold, npix, si, size, scale,
                                             offset, nelem_thread);
  else if (threads <= 64)
    centroids<  64><<<dimGrid, threads, 0, stream>>>(d_img, d_centroids, ref, validx, validy,
                                             intensities, weights, threshold, npix, si, size, scale,
                                             offset, nelem_thread);
  else if (threads <= 100)
    centroids< 100><<<dimGrid, threads, 0, stream>>>(d_img, d_centroids, ref, validx, validy,
                                             intensities, weights, threshold, npix, si, size, scale,
                                             offset, nelem_thread);
  else if (threads <= 144)
    centroids< 144><<<dimGrid, threads, 0, stream>>>(d_img, d_centroids, ref, validx, validy,
                                             intensities, weights, threshold, npix, si, size, scale,
                                             offset, nelem_thread);
  else if (threads <= 256)
    centroids< 256><<<dimGrid, threads, 0, stream>>>(d_img, d_centroids, ref, validx, validy,
                                             intensities, weights, threshold, npix, si, size, scale,
                                             offset, nelem_thread);
  else if (threads <= 512)
    centroids< 512><<<dimGrid, threads, 0, stream>>>(d_img, d_centroids, ref, validx, validy,
                                             intensities, weights, threshold, npix, si, size, scale,
                                             offset, nelem_thread);
  else if (threads <= 1024)
    centroids<1024><<<dimGrid, threads, 0, stream>>>(d_img, d_centroids, ref, validx,
                                             validy, intensities, weights, threshold, npix, si,
                                             size, scale, offset, nelem_thread);
  else
    printf("SH way too big !!!\n");

  carma_check_msg("centroids_kernel<<<>>> execution failed\n");
}

template void get_centroids<float>(int32_t size, int32_t threads, int32_t blocks, int32_t npix,
                                   float *d_img, float *d_centroids, float *ref,
                                   int32_t *validx, int32_t *validy, float *intensities,
                                   float *weights, float threshold, float scale, float offset,
                                   SlopeOrder slope_order,
                                   CarmaDevice *device, hipStream_t stream);

template void get_centroids<double>(int32_t size, int32_t threads, int32_t blocks, int32_t npix,
                                    float *d_img, double *d_centroids,
                                    double *ref, int32_t *validx, int32_t *validy,
                                    float *intensities, float *weights, float threshold,
                                    float scale, float offset,
                                    SlopeOrder slope_order,
                                    CarmaDevice *device, hipStream_t stream);
