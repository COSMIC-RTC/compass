#include "hip/hip_runtime.h"
#include <sutra_atmos.h>
#include <sutra_controller_utils.h>

/*  Tuning parameters of tbulateDPHI kernel*/
#define tabDPHI_thread_x (256)

/*	Tuning parameters of matcov GPU Kernel */
// Thread block size (x, y),
// max #threads per block is 512 for fermi and 1024 for kepler
#define matcov_thread_x (8)
#define matcov_thread_y (8)

//#define CUDA_ERROR_CHECK

//#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError() __cudaCheckError(__FILE__, __LINE__)
/*
inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}
*/
inline void __cudaCheckError(const char *file, const int line) {
#ifdef CUDA_ERROR_CHECK
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n", file, line,
            hipGetErrorString(err));
    exit(-1);
  }

  // More careful checking. However, this will affect performance.
  // Comment away if needed.
  err = hipDeviceSynchronize();
  if (hipSuccess != err) {
    fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n", file,
            line, hipGetErrorString(err));
    exit(-1);
  }
#endif

  return;
}

//============================================================================================
//================================= AUX FUNCTIONS
//============================================
//============================================================================================
#define VERBOSE 0
void process_err(hipError_t e, const char *str) {
  if (VERBOSE) printf("%s\n", str);
  if (e != hipSuccess) {
    printf("*** Error %s: %s \n", str, hipGetErrorString(e));
    exit(1);
  }
}
//-----------------------------------------------------------------------
double *arr2dAlloc_gpu_gb(long nbLin, long nbCol)
/* DOCUMENT  array = arr2dAlloc(nblin,nbcol)

 Allocates a 2d array (double).
 */
{
  hipError_t e;
  double *tableau;
  e = hipMalloc((void **)&tableau, sizeof(double) * nbCol * nbLin);
  process_err(e, "gpu alloc tableau2");
  return tableau;
}

void arr2dFree_gpu_gb(double *tableau)
/* DOCUMENT  arr2dFree(array)

 Free a 2d array (double).
 */
{
  if (tableau) hipFree(tableau);
}

//============================================================================================
//============================= tabDPHI FUNCTIONS/KERNEL(s)
//==================================
//============================================================================================
__device__ double macdo_x56_gpu_gb(double x, int k)
/* DOCUMENT  macdo_x56_gpu_gb(x)

 Computation of the function
 f(x) = x^(5/6)*K_{5/6}(x)
 using a series for the esimation of K_{5/6}, taken from Rod Conan thesis :
 K_a(x)=1/2 \sum_{n=0}^\infty \frac{(-1)^n}{n!}
 \left(\Gamma(-n-a) (x/2)^{2n+a} + \Gamma(-n+a) (x/2)^{2n-a} \right) ,
 with a = 5/6.

 Setting x22 = (x/2)^2, setting uda = (1/2)^a, and multiplying by x^a,
 this becomes :
 x^a * Ka(x) = 0.5 $ -1^n / n! [ G(-n-a).uda x22^(n+a) + G(-n+a)/uda x22^n ]
 Then we use the following recurrence formulae on the following quantities :
 G(-(n+1)-a) = G(-n-a) / -a-n-1
 G(-(n+1)+a) = G(-n+a) /  a-n-1
 (n+1)! = n! * (n+1)
 x22^(n+1) = x22^n * x22
 and at each iteration on n, one will use the values already computed at step
 (n-1). The values of G(a) and G(-a) are hardcoded instead of being computed.

 The first term of the series has also been skipped, as it
 vanishes with another term in the expression of Dphi.

 SEE ALSO:
 */
{
  const double a = 5. / 6.;
  const double x2a = pow(x, (double)2. * a), x22 = x * x / 4.;
  double x2n;  // x^2.a, etc
  double s = 0.0;
  int n;

  const double Ga[11] = {0,
                         12.067619015983075,
                         5.17183672113560444,
                         0.795667187867016068,
                         0.0628158306210802181,
                         0.00301515986981185091,
                         9.72632216068338833e-05,
                         2.25320204494595251e-06,
                         3.93000356676612095e-08,
                         5.34694362825451923e-10,
                         5.83302941264329804e-12};

  const double Gma[11] = {-3.74878707653729304,     -2.04479295083852408,
                          -0.360845814853857083,    -0.0313778969438136685,
                          -0.001622994669507603,    -5.56455315259749673e-05,
                          -1.35720808599938951e-06, -2.47515152461894642e-08,
                          -3.50257291219662472e-10, -3.95770950530691961e-12,
                          -3.65327031259100284e-14};

  x2n = 0.5;  // init (1/2) * x^0

  s = Gma[0] * x2a;
  s *= x2n;

  // prepare recurrence iteration for next step
  x2n *= x22;  // x^n

#pragma unroll
  for (n = 1; n <= 10; n++) {
    s += (Gma[n] * x2a + Ga[n]) * x2n;
    // prepare recurrence iteration for next step
    x2n *= x22;  // x^n
  }
  return s;
}
//------------------------------------------------------------------------------------
__device__ double asymp_macdo_gpu_gb(double x)
/* DOCUMENT asymp_macdo_gpu_gb(x)

 Computes a term involved in the computation of the phase struct
 function with a finite outer scale according to the Von-Karman
 model. The term involves the MacDonald function (modified bessel
 function of second kind) K_{5/6}(x), and the algorithm uses the
 asymptotic form for x ~ infinity.
 Warnings :
 - This function makes a doubleing point interrupt for x=0
 and should not be used in this case.
 - Works only for x>0.

 SEE ALSO:
 */
{
  // k2 is the value for
  // gamma_R(5./6)*2^(-1./6)
  const double k2 = 1.00563491799858928388289314170833;
  const double k3 = 1.25331413731550012081;   //  sqrt(pi/2)
  const double a1 = 0.22222222222222222222;   //  2/9
  const double a2 = -0.08641975308641974829;  //  -7/89
  const double a3 = 0.08001828989483310284;   // 175/2187
  double res;
  double x_1;

  x_1 = 1. / x;
  res = k2 - k3 * exp(-x) * pow(x, (double)(1 / 3.)) *
                 (1.0 + x_1 * (a1 + x_1 * (a2 + x_1 * a3)));
  return res;
}
//------------------------------------------------------------------------------------
__device__ double rodconan_gpu_gb(double r, double L0, int k)
/* DOCUMENT rodconan_gpu_gb(r,L0,k=)
 The phase structure function is computed from the expression
 Dphi(r) = k1  * L0^(5./3) * (k2 - (2.pi.r/L0)^5/6 K_{5/6}(2.pi.r/L0))

 For small r, the expression is computed from a development of
 K_5/6 near 0. The value of k2 is not used, as this same value
 appears in the series and cancels with k2.
 For large r, the expression is taken from an asymptotic form.

 SEE ALSO:
 */
{
  const double pi = 3.1415926535897932384626433;
  double res = 0;

  // k1 is the value of :
  // 2*gamma_R(11./6)*2^(-5./6)*pi^(-8./3)*(24*gamma_R(6./5)/5.)^(5./6);
  const double k1 = 0.1716613621245709486;
  const double dprf0 = (2 * pi / L0) * r;
  // k2 is the value for gamma_R(5./6)*2^(-1./6),
  // but is now unused
  // k2 = 1.0056349179985892838;

  // Xlim = 0.75*2*pi;   // = 4.71239
  if (dprf0 > 4.71239)
    res = asymp_macdo_gpu_gb(dprf0);
  else
    res = -macdo_x56_gpu_gb(dprf0, k);

  res *= k1 * pow(L0, (double)5. / 3);

  return res;
}

__global__ void tabulateDPHI_gpu_gb_kernel(double *tabDPHI_d, double *L0diff_d,
                                           long Nl0, long Ndphi,
                                           double convert) {
  const int tx = threadIdx.x;
  const int ty = blockIdx.x;

  const int tid = ty * blockDim.x + tx;
  int l = tid / Ndphi;
  int j = tid % Ndphi;

  if (tid >= (Nl0 * Ndphi)) return;

  tabDPHI_d[tid] = rodconan_gpu_gb((double)j / convert, L0diff_d[l], 10);

  // double* mytabDPHI = tabDPHI_d + (l * Ndphi);
  //
  // int j, k;
  //#pragma unroll
  // for(k = 0; k < (Ndphi/tabDPHI_thread_x); k++)
  //{
  //	j = k * tabDPHI_thread_x + tx;
  //	mytabDPHI[j] = rodconan_gpu_gb(rr_d[j], L0diff_d[l], 10);
  //}
  //
  // k = (Ndphi/tabDPHI_thread_x);
  // if(tx < (Ndphi%tabDPHI_thread_x) )
  //{
  //	j = k * tabDPHI_thread_x + tx;
  //	mytabDPHI[j] = rodconan_gpu_gb(rr_d[j], L0diff_d[l], 10);
  //}
}

__device__ double Ij0t83_gb(double x, double *tab_x, double *tab_y, long npts) {
  if (x <= exp(-3.0))
    return (double)(0.75 * pow((double)x, 1 / 3.) * (1 - x * x / 112.));
  else {
    double dt = 14. / (npts - 1);  // 14 = tmax - tmin
    double convert = (log(x) + 4.) / dt;
    long i0 = (long)convert;
    long i1 = i0 + 1;
    /*
    long i1 = 0;
    while(x > tab_x[i1])
        i1++;
    long i0 = i1 - 1;
    */
    // long i0 = (long) ((logf(x) + 4.0f) * convert);
    // long i1 = i0 + 1;
    // return  ((x - (double)i0 / convert) * tab_y[indexL0 * npts + i1]
    //    + ((double)i1 / convert - x) * tab_y[indexL0 * npts + i0]);
    return (((x - tab_x[i0]) * tab_y[/*indexL0 * npts +*/ i1] +
             (tab_x[i1] - x) * tab_y[/*indexL0 * npts +*/ i0])) /
           (tab_x[i1] - tab_x[i0]);
  }
}

__global__ void tabulateDPHI_lowpass_kernel(double *tabDPHI_d,
                                            double *tab_int_x,
                                            double *tab_int_y, double *dx,
                                            double *L0diff_d, long Nl0,
                                            long Ndphi, double convert,
                                            double convert_int, long npts) {
  const double pi = 3.1415926535897932384626433;
  const int tx = threadIdx.x;
  const int ty = blockIdx.x;

  const int tid = ty * blockDim.x + tx;
  int dm = tid / (Ndphi * Nl0);
  int pos = tid - dm * (Ndphi * Nl0);
  int l = pos / Ndphi;
  // int j = tid % Ndphi;
  int j = pos - l * Ndphi;
  double r = (double)j / convert;
  // long indexL0 = 0;

  if (tid >= (Nl0 * Ndphi)) return;

  // tabDPHI_d[tid] = Ij0t83_gb((double)(r*(pi/dx)), indexL0, tab_int_x,
  // tab_int_y, convert_int,npts);

  tabDPHI_d[tid] =
      pow(r, (double)(5. / 3.)) *
      Ij0t83_gb((double)(r * (pi / dx[dm])), tab_int_x, tab_int_y, npts) *
      (double)((2 * pow((2 * pi), (double)(8 / 3.)) * 0.0228956));
}

//------------------------------------------------------------------------------------
__device__ double DPHI_gpu_gb(double x, double y, double L0)
/* DOCUMENT dphi = DPHI(x,y,indexL0,rr,tabDPHI,convert) * r0^(-5./3)
 <x> & <y>         :  separation between apertures
 <indexL0>         :  index for the L0 taken into account
 <rr>              :  array of distance between apertures
 <tabDPHI>         :  array of precomputed DPHI
 <convert>         :  relation between the index on tabDPHI and (x,y)

 Computes the phase structure function for a separation (x,y).
 The r0 is not taken into account : the final result of DPHI(x,y,L0)
 has to be scaled with r0^-5/3, with r0 expressed in meters, to get
 the right value.

 SEE ALSO:
 */
{
  double r = sqrt(x * x + y * y);

  return rodconan_gpu_gb(r, L0, 10);
  /*
  long i0 = (long) (r * convert);
  long i1 = i0 + 1;

  return ((r - (double)i0 / convert) * tabDPHI[indexL0 * Ndphi + i1]
    + ((double)i1 / convert - r) * tabDPHI[indexL0 * Ndphi + i0]);
    */
}

//============================================================================================
//============================= SUBAP POSITION KERNELS/FUNCTIONS
//=============================
//============================================================================================
__global__ void subposition_gpu_gb_kernel(
    long Nw, long Nx, long *Nsubap, long Nlayer, double *alphaX, double *alphaY,
    double *h, double *GsAlt, long *Nssp, double *diamPup, double *thetaML,
    long *ioff, double *X, double *Y, double *XPup, double *YPup, double *u,
    double *v) {
  const int tx = threadIdx.x;
  const int ty = blockIdx.x;

  const int tid = ty * blockDim.x + tx;
  long i;      // subaperture i
  long n = 0;  // WFS n
  long l;
  const double rad = 3.14159265358979323846 / 180.;

  if (tid >= (Nx * Nlayer)) return;

  l = tid / Nx;

  const int pos = tid - l * Nx;
  long Nsubapx = Nsubap[0];

  while (pos >= Nsubapx) {
    n++;
    Nsubapx += Nsubap[n];
  }
  Nsubapx -= Nsubap[n];

  i = pos - Nsubapx;

  // tid = n + i * Nw + l * Nw * Nsubap

  const double dX = alphaX[n] * h[l];
  const double dY = alphaY[n] * h[l];

  const double rr = 1. - h[l] * GsAlt[n];

  const long nssp = Nssp[n];

  // magnification factor
  const double G = diamPup[n] / (double)(nssp);

  // rotation angle
  const double th = thetaML[n] * rad;

  // taking magnification factor into account
  const double xtp = X[ioff[n] + i] * G;
  const double ytp = Y[ioff[n] + i] * G;

  // taking rotation into account
  double uu = xtp * cos(th) - ytp * sin(th);
  double vv = xtp * sin(th) + ytp * cos(th);

  // taking pupil offset into account
  uu += XPup[n];
  vv += YPup[n];

  // Projection onto  the layer

  u[tid] = uu * rr + dX;
  v[tid] = vv * rr + dY;
}

//============================================================================================
//============================= MATCOV ELEMENTARY FUNCTIONS
//==================================
//============================================================================================
__device__ double cov_XX_gpu_gb(double du, double dv, double ac, double ad,
                                double bc, double bd, double *tabDPHI,
                                double L0, double convert, int Ndphi)
/* DOCUMENT
  Compute the XX-covariance with the distance sqrt(du2+dv2). DPHI is precomputed
  on tabDPHI.
*/
{
  return -DPHI_gpu_gb(du + ac, dv, L0) + DPHI_gpu_gb(du + ad, dv, L0) +
         DPHI_gpu_gb(du + bc, dv, L0) - DPHI_gpu_gb(du + bd, dv, L0);
}

//------------------------------------------------------------------------------------
__device__ double cov_YY_gpu_gb(double du, double dv, double ac, double ad,
                                double bc, double bd, double *tabDPHI,
                                double L0, double convert, int Ndphi)
/* DOCUMENT
   Compute the YY-covariance with the distance sqrt(du2+dv2). DPHI is
   precomputed on tabDPHI.
 */
{
  return -DPHI_gpu_gb(du, dv + ac, L0) + DPHI_gpu_gb(du, dv + ad, L0) +
         DPHI_gpu_gb(du, dv + bc, L0) - DPHI_gpu_gb(du, dv + bd, L0);
}

//------------------------------------------------------------------------------------
__device__ double cov_XY_gpu_gb(double du, double dv, double s0,
                                double *tabDPHI, double L0, double convert,
                                int Ndphi)
/* DOCUMENT
   Compute the XY-covariance with the distance sqrt(du2+dv2). DPHI is
   precomputed on tabDPHI.
 */
{
  return -DPHI_gpu_gb(du + s0, dv - s0, L0) +
         DPHI_gpu_gb(du + s0, dv + s0, L0) + DPHI_gpu_gb(du - s0, dv - s0, L0) -
         DPHI_gpu_gb(du - s0, dv + s0, L0);
}

//============================================================================================
//============================= CPHIM ELEMENTARY FUNCTIONS
//==================================
//============================================================================================
__device__ double DPHI_highpass_gb(double r, double fc, double *tab_x,
                                   double *tab_y, long npts) {
  const double pi = 3.1415926535897932384626433;
  return pow(r, 5 / 3.) *
         (1.1183343328701949 - Ij0t83_gb(2 * pi * fc * r, tab_x, tab_y, npts)) *
         pow(2 * pi, 8 / 3.) * 2 * 0.0228956;
}
__device__ double DPHI_lowpass_gb(double x, double y, double L0, double fc,
                                  double *tab_int_x, double *tab_int_y,
                                  long npts) {
  /*
  double r = sqrt(x * x + y * y);
  const double pi = 3.1415926535897932384626433;
  int npts = (int)1/pas;
  double du = 2*pi*fc*r/npts;
  double dphi = 0;
  double u = 0;
  //for(double u=pas ; u <= 2*pi*fc*r ; u+=pas ){
  for (int i=1 ; i < npts ; i++){
        u += du;
        dphi += pow(u*u + pow(2*pi*r/L0,2),-11/6.) * u * (1- j0(u)) * du;
  }

  return 2*pow((2*pi),8/3.)*0.0228956*pow(r,5/3.) * dphi;
  */
  double r = sqrt(x * x + y * y);

  return rodconan_gpu_gb(r, L0, 10) -
         DPHI_highpass_gb(r, fc, tab_int_x, tab_int_y, npts);
}
__device__ double cphim_XX(double du, double dv, double posx, double posy,
                           double xref, double yref, double s2, double L0,
                           double fc, long npts, double *tab_int_x,
                           double *tab_int_y)
/* DOCUMENT
  Compute the XX-covariance with the distance sqrt(du2+dv2). DPHI is precomputed
  on tabDPHI.
*/
{
  return -DPHI_lowpass_gb(du - 2 * s2, dv - s2, L0, fc, tab_int_x, tab_int_y,
                          npts) +
         DPHI_lowpass_gb(du, dv - s2, L0, fc, tab_int_x, tab_int_y, npts) +
         DPHI_lowpass_gb(posx + 2 * s2 - xref, posy + s2 - yref, L0, fc,
                         tab_int_x, tab_int_y, npts) -
         DPHI_lowpass_gb(posx - xref, posy + s2 - yref, L0, fc, tab_int_x,
                         tab_int_y, npts);

  /*
                return -DPHI_gpu_gb(du - 2*s2, dv - s2, L0)
                                + DPHI_gpu_gb(du, dv - s2, L0)
                                + DPHI_gpu_gb(posx+ 2*s2, posy + s2, L0)
                                - DPHI_gpu_gb(posx, posy+s2, L0);*/
}

//------------------------------------------------------------------------------------
__device__ double cphim_YY(double du, double dv, double posx, double posy,
                           double xref, double yref, double s2, double L0,
                           double fc, long npts, double *tab_int_x,
                           double *tab_int_y)
/* DOCUMENT
   Compute the YY-covariance with the distance sqrt(du2+dv2). DPHI is
   precomputed on tabDPHI.
 */
{
  return -DPHI_lowpass_gb(du - s2, dv - 2 * s2, L0, fc, tab_int_x, tab_int_y,
                          npts) +
         DPHI_lowpass_gb(du - s2, dv, L0, fc, tab_int_x, tab_int_y, npts) +
         DPHI_lowpass_gb(posx + s2 - xref, posy + 2 * s2 - yref, L0, fc,
                         tab_int_x, tab_int_y, npts) -
         DPHI_lowpass_gb(posx + s2 - xref, posy - yref, L0, fc, tab_int_x,
                         tab_int_y, npts);

  /*
    return  -DPHI_gpu_gb(du-s2, dv - 2*s2, L0)
      + DPHI_gpu_gb(du-s2, dv, L0)
      + DPHI_gpu_gb(posx+s2, posy + 2*s2, L0)
      - DPHI_gpu_gb(posx+s2, posy, L0);*/
}
//============================================================================================
//============================= MATCOV 3 FUNCTIONS/KERNEL
//====================================
//============================================================================================
__device__ double compute_element_3(int ipos, int jpos, double convert,
                                    double *sspSizeL, long *Nssp, double *u,
                                    double *v, double pasDPHI, double *tabDPHI,
                                    long *indexL0, double *cn2, int Ndphi,
                                    int Nw, int Nlayer, int Nsubap,
                                    int type_mat, double teldiam) {
  /* *** Covariance matrix per-element generation ***
   *   Arguments
   *   =========
   *	ipos:		Integer: global x-coordinate of the element w.r.t. the
   *entire matrix jpos:		Integer: global y-coordinate of the element
   *w.r.t. the entire matrix
   */

  const double lambda2 = 0.00026942094446267851;
  const int nslps = Nsubap * 2;

  // WFS m
  int m = ipos / nslps;  // tab_wfs[ipos];
  if (type_mat == 3) m = Nw - 1;
  // WFS n
  int n = jpos / nslps;  // tab_wfs[jpos];
  if (type_mat == 2) n = Nw - 1;

  // subap i
  int i = ipos % (nslps / 2);  // tab_subap[ipos];
  // subap j
  int j = jpos % (nslps / 2);  // tab_subap[jpos];;

  // xy i
  int xy_i = (ipos / (nslps / 2)) % 2;  // tab_xy[ipos];
  // xy j
  int xy_j = (jpos / (nslps / 2)) % 2;  // tab_xy[jpos];

  const double sspSizem = teldiam / Nssp[m];
  const double sspSizen = teldiam / Nssp[n];

  const double kk = lambda2 / (sspSizem * sspSizen);

  int type = xy_i * 2 + xy_j;

  // Layer l
  double covar = 0.0;
#pragma unroll
  for (int l = 0; l < Nlayer; l++) {
    const double sspSizeml = sspSizeL[m * Nlayer + l];
    const double sspSizenl = sspSizeL[n * Nlayer + l];
    // test if the altitude layers is not higher than the LGS altitude
    if ((sspSizeml > 0) && (sspSizenl > 0)) {
      const int pos1 = m + i * Nw + l * Nw * Nsubap;
      const int pos2 = n + j * Nw + l * Nw * Nsubap;
      const double du = u[pos1] - u[pos2];
      const double dv = v[pos1] - v[pos2];

      const double s1 = sspSizeml * 0.5;
      const double s2 = sspSizenl * 0.5;

      const double ac = s1 - s2;
      const double ad = s1 + s2;
      const double bc = -ad;  // initially -s1-s2;
      const double bd = -ac;  // initially -s1+s2;

      if (type == 0)
        covar += 0.5 * pasDPHI *
                 cov_XX_gpu_gb(du, dv, ac, ad, bc, bd, tabDPHI, indexL0[l],
                               convert, Ndphi) *
                 kk * cn2[l];
      else if (type == 3)
        covar += 0.5 * pasDPHI *
                 cov_YY_gpu_gb(du, dv, ac, ad, bc, bd, tabDPHI, indexL0[l],
                               convert, Ndphi) *
                 kk * cn2[l];
      else {  // if ((type == 1) || (type == 2))
        const double s0 =
            sqrt(s1 * s1 + s2 * s2);  // half size of the subaperture equivalent
                                      // to a convolution by s1 and s2
        const double dd =
            (s1 > s2) ? 1. - s2 / s1 : 1. - s1 / s2;  // Nono's style ....
        covar +=
            0.25 * pasDPHI *
            cov_XY_gpu_gb(du, dv, s0, tabDPHI, indexL0[l], convert, Ndphi) *
            kk * cn2[l] * (1. - dd * dd);
      }
    }
  }
  return (double)covar;
}

__global__ void matcov_kernel_3(char uplo, char copy, double *data, int nrows,
                                int ncols, int xoffset, int yoffset, int lda,
                                double convert, double *sspSizeL, long *Nssp,
                                double *u, double *v, double pasDPHI,
                                double *tabDPHI, long *indexL0, double *cn2,
                                int Ndphi, int Nw, int Nlayer, int Nsubap,
                                int type_mat, double teldiam) {
  /* *** covariance matrix generation kernel ***
   *	The kernel generates the element values in a given matrix/submatrix
   *   The generation function can be any function, as long as each element
   *   can be computed both individually and independently
   *
   *	see argument description in the kernel driver
   */

  // local thread coordinates w.r.t. thread block
  const int tx_ = threadIdx.x;
  const int ty_ = threadIdx.y;

  // local thread block coordinates w.r.t. kernel grid
  const int bx_ = blockIdx.x;
  const int by_ = blockIdx.y;

  // local coordinates of the element w.r.t. submatrix
  int lx = bx_ * blockDim.x + tx_;
  int ly = by_ * blockDim.y + ty_;

  // global coordinates of the elemnt w.r.t. the entire matrix
  int gx = lx + xoffset;
  int gy = ly + yoffset;

  // out-of-bound threads should terminate
  if ((lx >= nrows) || (ly >= ncols)) return;

  double value;
  if (uplo == 'l') {
    if (gy <= gx) {
      value = compute_element_3(gx, gy, convert, sspSizeL, Nssp, u, v, pasDPHI,
                                tabDPHI, indexL0, cn2, Ndphi, Nw, Nlayer,
                                Nsubap, type_mat, teldiam);
      data[ly * lda + lx] = value;
      if (copy == 'c') data[lx * lda + ly] = value;
    }
  } else if (uplo == 'u') {  // upper
    if (gx <= gy) {
      value = compute_element_3(gx, gy, convert, sspSizeL, Nssp, u, v, pasDPHI,
                                tabDPHI, indexL0, cn2, Ndphi, Nw, Nlayer,
                                Nsubap, type_mat, teldiam);
      data[ly * lda + lx] = value;
      if (copy == 'c') data[lx * lda + ly] = value;
    }
  } else {  // uplo = 'f' full generation
    value = compute_element_3(gx, gy, convert, sspSizeL, Nssp, u, v, pasDPHI,
                              tabDPHI, indexL0, cn2, Ndphi, Nw, Nlayer, Nsubap,
                              type_mat, teldiam);
    data[ly * lda + lx] = value;
  }

  // if ((type_mat == 3) || (gx <= gy))
  //{
  // call the generation function
  // data[0] = compute_element_3(gx, gy, tab_wfs, tab_subap,
  // tab_xy,convert,sspSizeL,Nssp,u,v,pasDPHI,tabDPHI,
  //		      indexL0,cn2,Ndphi,Nw,Nlayer,Nsubap,type_mat,teldiam);
  // printf("gx = %d, gy = %d ----- %.2f \n", gx, gy, data[0]);
  //}
}

//============================================================================================
//============================= MATCOV TS FUNCTIONS/KERNEL
//===================================
//============================================================================================
__device__ double compute_element_ts_(int ipos, int jpos, double convert,
                                      double *X, double *Y, long *Nssp,
                                      double pasDPHI, double *tabDPHI,
                                      long *indexL0, double *cn2, int Ndphi,
                                      int Nw, int Nlayer, int Nsubap,
                                      double teldiam) {
  /* *** Covariance matrix per-element generation ***
   *   Arguments
   *   =========
   *	ipos:		Integer: global x-coordinate of the element w.r.t. the
   *entire matrix jpos:		Integer: global y-coordinate of the element
   *w.r.t. the entire matrix
   */

  // for now return a dummy value

  double lambda2 = 0.00026942094446267851;
  // WFS Nw-1
  // subap i
  int i = ipos < Nsubap ? ipos : ipos - Nsubap;
  // subap j
  int j = jpos < Nsubap ? jpos : jpos - Nsubap;
  // xy i
  int xy_i = ipos < Nsubap ? 0 : 1;
  // xy j
  int xy_j = jpos < Nsubap ? 0 : 1;

  double sspSize = teldiam / Nssp[Nw - 1];

  double kk = lambda2 / (sspSize * sspSize);

  int type = xy_i * 2 + xy_j;

  double s = sspSize * 0.5;

  double ac = 0.0;
  double ad = 2.0 * s;
  double bc = -ad;
  double bd = 0.0;

  double du = X[(Nsubap * (Nw - 1) + i)] - X[(Nsubap * (Nw - 1) + j)];
  double dv = Y[(Nsubap * (Nw - 1) + i)] - Y[(Nsubap * (Nw - 1) + j)];

  // if(ipos < 10)printf("ipos = %d - %d\n", ipos, (Nsubap*(Nw-1)+i));
  // if(jpos < 10)printf("jpos = %d - %d\n", jpos, (Nsubap*(Nw-1)+j));

  // const double du = X[0] - X[1];
  // const double dv = Y[0] - Y[1];

  // Layer l
  double covar = 0.0;
#pragma unroll
  for (int l = 0; l < Nlayer; l++) {
    // test if the altitude layers is not higher than the LGS altitude
    if (sspSize > 0) {
      if (type == 0)
        covar += 0.5 * pasDPHI *
                 cov_XX_gpu_gb(du, dv, ac, ad, bc, bd, tabDPHI, indexL0[l],
                               convert, Ndphi) *
                 kk * cn2[l];
      else if (type == 3)
        covar += 0.5 * pasDPHI *
                 cov_YY_gpu_gb(du, dv, ac, ad, bc, bd, tabDPHI, indexL0[l],
                               convert, Ndphi) *
                 kk * cn2[l];
      else {
        double s0 = 1.41421 * s;  // half size of the subaperture equivalent to
                                  // a convolution by s1 and s2
        double dd = 0;
        covar +=
            0.25 * pasDPHI *
            cov_XY_gpu_gb(du, dv, s0, tabDPHI, indexL0[l], convert, Ndphi) *
            kk * cn2[l] * (1. - dd * dd);
      }
    }
  }
  return (double)covar;
}
//--------------------------------------------------------------------------------------------
__global__ void matcov_ts_kernel(double *data, int nrows, int ncols,
                                 int xoffset, int yoffset, int lda,
                                 double convert, double *X, double *Y,
                                 long *Nssp, double pasDPHI, double *tabDPHI,
                                 long *indexL0, double *cn2, int Ndphi, int Nw,
                                 int Nlayer, int Nsubap, double teldiam) {
  /* *** covariance matrix generation kernel ***
   *	The kernel generates the element values in a given matrix/submatrix
   *   The generation function can be any function, as long as each element
   *   can be computed both individually and independently
   *
   *	see argument description in the kernel driver
   */

  // local thread coordinates w.r.t. thread block
  const int tx_ = threadIdx.x;
  const int ty_ = threadIdx.y;

  // local thread block coordinates w.r.t. kernel grid
  const int bx_ = blockIdx.x;
  const int by_ = blockIdx.y;

  // local coordinates of the element w.r.t. submatrix
  int lx = bx_ * blockDim.x + tx_;
  int ly = by_ * blockDim.y + ty_;

  // global coordinates of the elemnt w.r.t. the entire matrix
  int gx = lx + xoffset;
  int gy = ly + yoffset;

  // out-of-bound threads should terminate
  if ((lx >= nrows) || (ly >= ncols)) return;

  // Advance the data pointer accordingly
  data += ly * lda + lx;

  // call the generation function
  data[0] =
      compute_element_ts_(gx, gy, convert, X, Y, Nssp, pasDPHI, tabDPHI,
                          indexL0, cn2, Ndphi, Nw, Nlayer, Nsubap, teldiam);
  // printf("gx = %d, gy = %d ----- %.2f \n", gx, gy, data[0]);
}

//============================================================================================
//============================= MATCOV TS
//===================================================
//============================================================================================
/*
void matts_gpu_gb(double* data, int nrows, int ncols, int xoffset, int yoffset,
int lda, struct tomo_struct tomo, struct gtomo_struct *tomo_gpu)
{
        /* *** matcov gpu kernel driver ***
        *  Arguments
        *  ==========
        *  data		double pointer: A pointer to the matrix/submatrix to be
generated. It
        *  			should always point to the first element in a
matrix/submatrix
        *
        *  nrows	integer: The number of rows of the matrix/submatrix to
be generated
        *
        *  ncols	integer: The number of columns of the matrix/submatrix
to be generated
        *
        *  xoffset	integer: The x-offset of the submatrix, must be zero if
the entire matrix
        *			is generated. Its the x-coordinate of the first
element in the matrix/submatrix
        *
        *  yoffset  integer: The y-offset of the submatrix, must be zero if the
entire matrix
        *			is generated. Its the y-coordinate of the first
element in the matrix/submatrix
        *
        *  lda		integer: The leading dimension of the matrix/submatrix
        */
/*
  const long Nw = tomo.Nw;
  const double crmax = tomo.rmax;
  const double pasDPHI = 1./tomo.pasDPHI; //inverse du pas de rr
  const long Ndphi = floor(crmax*pasDPHI)+1;
  const double convert = (double)(Ndphi-1)/(crmax+1./pasDPHI);


  int nbx = nrows / matcov_thread_x + (nrows%matcov_thread_x != 0);
  int nby = ncols / matcov_thread_y + (ncols%matcov_thread_y != 0);

  dim3 dimBlock(matcov_thread_x, matcov_thread_y);
  dim3 dimGrid(nbx, nby);
  const long Nsubap = tomo.Nsubap[Nw-1];

  matcov_ts_kernel<<<dimGrid, dimBlock, 0, tomo_gpu->matcov_stream>>>(data,
nrows, ncols, xoffset, yoffset, lda,
                                           convert,tomo_gpu->X_d,tomo_gpu->Y_d,tomo_gpu->Nssp_d,
                                           pasDPHI,tomo_gpu->tabDPHI_d,tomo_gpu->indexL0_d,tomo_gpu->cn2_d,
                                           Ndphi,tomo.Nw,atmos->nscreens,Nsubap,tomo.DiamTel);
  //CudaCheckError();
}
*/
//============================================================================================
//============================= MATCOV COPY KERNEL
//===========================================
//============================================================================================
__global__ void matcov_kernel_copy(double *data, int nrows, int ncols,
                                   int xoffset, int yoffset, int lda) {
  /* *** covariance matrix generation kernel ***
   *	The kernel generates the element values in a given matrix/submatrix
   *   The generation function can be any function, as long as each element
   *   can be computed both individually and independently
   *
   *	see argument description in the kernel driver
   */

  // local thread coordinates w.r.t. thread block
  const int tx_ = threadIdx.x;
  const int ty_ = threadIdx.y;

  // local thread block coordinates w.r.t. kernel grid
  const int bx_ = blockIdx.x;
  const int by_ = blockIdx.y;

  // local coordinates of the element w.r.t. submatrix
  int lx = bx_ * blockDim.x + tx_;
  int ly = by_ * blockDim.y + ty_;

  // global coordinates of the elemnt w.r.t. the entire matrix
  int gx = lx + xoffset;
  int gy = ly + yoffset;

  // out-of-bound threads should terminate
  if ((lx >= nrows) || (ly >= ncols)) return;

  // Advance the data pointer accordingly
  // data += ly * lda + lx;

  if (gx > gy) {
    // call the generation function
    data[ly * lda + lx] = data[ly + lx * lda];
    // printf("gx = %d, gy = %d ----- %.2f \n", gx, gy, data[0]);
  }
}

//============================================================================================
//============================= MATCOV 1
//=====================================================
//============================================================================================
//************************** OBSOLETE - REMOVED
//********************************************//

//============================================================================================
//============================= MATCOV 2
//=====================================================
//============================================================================================
//************************** OBSOLETE - REMOVED
//********************************************//

//============================================================================================
//=============================== TOMO INIT/FIN FUNCTIONS
//====================================
//============================================================================================
void init_tomo_gpu_gb(struct gtomo_struct *tomo_gpu, sutra_atmos *atmos,
                      sutra_sensors *sensors, double diamTel, double cobs) {
  hipError_t e;

  tomo_gpu->DiamTel = diamTel;
  tomo_gpu->obs = cobs;
  tomo_gpu->Nw = sensors->nsensors();  // Adding TS for debug

  tomo_gpu->lgs_cst = 0.;
  tomo_gpu->spot_width = 1.;
  tomo_gpu->lgs_depth = 10000.;
  tomo_gpu->lgs_alt = 90000.;
  tomo_gpu->nlgs = 0;
  tomo_gpu->pasDPHI = 0.0001;

  tomo_gpu->Nx = 0;
  tomo_gpu->Nssp = (long *)malloc(tomo_gpu->Nw * sizeof(long));
  tomo_gpu->Nsubap = (long *)malloc(tomo_gpu->Nw * sizeof(long));
  tomo_gpu->diamPup = (double *)malloc(tomo_gpu->Nw * sizeof(double));
  tomo_gpu->XPup = (double *)malloc(tomo_gpu->Nw * sizeof(double));
  tomo_gpu->YPup = (double *)malloc(tomo_gpu->Nw * sizeof(double));
  tomo_gpu->thetaML = (double *)malloc(tomo_gpu->Nw * sizeof(double));
  tomo_gpu->GsAlt = (double *)malloc(tomo_gpu->Nw * sizeof(double));

  for (int i = 0; i < tomo_gpu->Nw; i++) {
    tomo_gpu->Nssp[i] = sensors->d_wfs[i]->nxsub;
    tomo_gpu->Nsubap[i] = sensors->d_wfs[i]->nvalid;
    tomo_gpu->diamPup[i] = (double)tomo_gpu->Nssp[i];
    tomo_gpu->XPup[i] = 0.;
    tomo_gpu->YPup[i] = 0.;
    tomo_gpu->thetaML[i] = 0.;
    if (sensors->d_wfs[i]->d_gs->lgs) {
      tomo_gpu->nlgs += 1;
      tomo_gpu->GsAlt[i] = 1.0 / tomo_gpu->lgs_alt;
    } else
      tomo_gpu->GsAlt[i] = 0.0;
    tomo_gpu->Nx += sensors->d_wfs[i]->nvalid;
  }

  e = hipMalloc((void **)&(tomo_gpu->indexL0_d),
                 atmos->nscreens * sizeof(long));
  process_err(e, "alloc gpu indexL0_d");

  e = hipMalloc((void **)&(tomo_gpu->u_d),
                 atmos->nscreens * tomo_gpu->Nx * sizeof(double));
  process_err(e, "alloc gpu u_d");

  e = hipMalloc((void **)&(tomo_gpu->v_d),
                 atmos->nscreens * tomo_gpu->Nx * sizeof(double));
  process_err(e, "alloc gpu v_d");

  e = hipMalloc((void **)&(tomo_gpu->sspSizeL_d),
                 tomo_gpu->Nw * atmos->nscreens * sizeof(double));
  process_err(e, "alloc gpu sspSizeL_d");

  e = hipMalloc((void **)&(tomo_gpu->cn2_d), atmos->nscreens * sizeof(double));
  process_err(e, "alloc gpu cn2_d");

  e = hipMalloc((void **)&(tomo_gpu->h_d), atmos->nscreens * sizeof(double));
  process_err(e, "alloc gpu h_d");

  e = hipMalloc((void **)&(tomo_gpu->Nssp_d), tomo_gpu->Nw * sizeof(long));
  process_err(e, "alloc gpu Nssp_d");

  e = hipMalloc((void **)&(tomo_gpu->Nsubap_d), tomo_gpu->Nw * sizeof(long));
  process_err(e, "alloc gpu Nsubap_d");

  e = hipMalloc((void **)&(tomo_gpu->ioff_d), tomo_gpu->Nw * sizeof(long));
  process_err(e, "alloc gpu ioff_d");

  e = hipMalloc((void **)&(tomo_gpu->alphaX_d), tomo_gpu->Nw * sizeof(double));
  process_err(e, "alloc gpu alphaX_d");

  e = hipMalloc((void **)&(tomo_gpu->alphaY_d), tomo_gpu->Nw * sizeof(double));
  process_err(e, "alloc gpu alphaY_d");

  e = hipMalloc((void **)&(tomo_gpu->GsAlt_d), tomo_gpu->Nw * sizeof(double));
  process_err(e, "alloc gpu GsAlt_d");

  e = hipMalloc((void **)&(tomo_gpu->diamPup_d),
                 tomo_gpu->Nw * sizeof(double));
  process_err(e, "alloc gpu diamPup_d");

  e = hipMalloc((void **)&(tomo_gpu->thetaML_d),
                 tomo_gpu->Nw * sizeof(double));
  process_err(e, "alloc gpu thetaML_d");

  e = hipMalloc((void **)&(tomo_gpu->X_d), tomo_gpu->Nx * sizeof(double));
  process_err(e, "alloc gpu X_d");

  e = hipMalloc((void **)&(tomo_gpu->Y_d), tomo_gpu->Nx * sizeof(double));
  process_err(e, "alloc gpu Y_d");

  e = hipMalloc((void **)&(tomo_gpu->XPup_d), tomo_gpu->Nw * sizeof(double));
  process_err(e, "alloc gpu XPup_d");

  e = hipMalloc((void **)&(tomo_gpu->YPup_d), tomo_gpu->Nw * sizeof(double));
  process_err(e, "alloc gpu YPup_d");

  tomo_gpu->L0diff_d = NULL;
  tomo_gpu->tabDPHI_d = NULL;

  e = hipStreamCreate(&(tomo_gpu->matcov_stream));
  process_err(e, "create matcov stream");
}

void free_tomo_gpu_gb(struct gtomo_struct *tomo_gpu) {
  hipError_t e;

  if ((tomo_gpu->u_d)) e = hipFree(tomo_gpu->u_d);
  process_err(e, "free gpu u_d");

  if (tomo_gpu->v_d) e = hipFree(tomo_gpu->v_d);
  process_err(e, "free gpu v_d");

  if (tomo_gpu->sspSizeL_d) e = hipFree(tomo_gpu->sspSizeL_d);
  process_err(e, "free gpu sspSizeL_d");

  if (tomo_gpu->cn2_d) e = hipFree(tomo_gpu->cn2_d);
  process_err(e, "free gpu cn2_d");

  if (tomo_gpu->h_d) e = hipFree(tomo_gpu->h_d);
  process_err(e, "free gpu h_d");

  if (tomo_gpu->Nsubap_d) e = hipFree(tomo_gpu->Nsubap_d);
  process_err(e, "free gpu Nsubap_d");

  if (tomo_gpu->indexL0_d) e = hipFree(tomo_gpu->indexL0_d);
  process_err(e, "free gpu indexL0_d");

  if (tomo_gpu->Nssp_d) e = hipFree(tomo_gpu->Nssp_d);
  process_err(e, "free gpu Nssp_d");

  if (tomo_gpu->ioff_d) e = hipFree(tomo_gpu->ioff_d);
  process_err(e, "free gpu ioff_d");

  if (tomo_gpu->alphaX_d) e = hipFree(tomo_gpu->alphaX_d);
  process_err(e, "free gpu alphaX_d");

  if (tomo_gpu->alphaY_d) e = hipFree(tomo_gpu->alphaY_d);
  process_err(e, "free gpu alphaY_d");

  if (tomo_gpu->GsAlt_d) e = hipFree(tomo_gpu->GsAlt_d);
  process_err(e, "free gpu GsAlt_d");

  if (tomo_gpu->diamPup_d) e = hipFree(tomo_gpu->diamPup_d);
  process_err(e, "free gpu diamPup_d");

  if (tomo_gpu->thetaML_d) e = hipFree(tomo_gpu->thetaML_d);
  process_err(e, "free gpu thetaML_d");

  if (tomo_gpu->X_d) e = hipFree(tomo_gpu->X_d);
  process_err(e, "free gpu X_d");

  if (tomo_gpu->Y_d) e = hipFree(tomo_gpu->Y_d);
  process_err(e, "free gpu Y_d");

  if (tomo_gpu->XPup_d) e = hipFree(tomo_gpu->XPup_d);
  process_err(e, "free gpu XPup_d");

  if (tomo_gpu->YPup_d) e = hipFree(tomo_gpu->YPup_d);
  process_err(e, "free gpu YPup_d");

  /*
  if (tomo_gpu->Cmm_d) e = hipFree(tomo_gpu->Cmm_d);
  process_err(e, "free gpu YPup_d");

  if (tomo_gpu->Cpm_d) e = hipFree(tomo_gpu->Cpm_d);
  process_err(e, "free gpu YPup_d");

  if (tomo_gpu->R_d) e = hipFree(tomo_gpu->R_d);
  process_err(e, "free gpu YPup_d");
  */

  if ((tomo_gpu->tabDPHI_d) != NULL) e = hipFree(tomo_gpu->tabDPHI_d);
  process_err(e, "free gpu tabDPHI_d");

  if ((tomo_gpu->L0diff_d) != NULL) e = hipFree(tomo_gpu->L0diff_d);
  process_err(e, "free gpu L0diff_d");

  // destroy matcov stream
  e = hipStreamDestroy(tomo_gpu->matcov_stream);
  process_err(e, "destroy matcov stream");
}

//============================================================================================
//============================ CPHIM DPHI FUNCTIONS
//=============================
//============================================================================================
void tab_dphi_lowpass(double *tab_dphi, struct cphim_struct *cphim_struct,
                      long Ndphi, double *L0diff_d, int Nl0, double convert,
                      double convert_int)
// void tabulateDPHI_gpu_gb(double* tabDPHI_d, double* rr_d,struct tomo_struct
// tomo, long Ndphi, long *indexL0_h)
/* DOCUMENT tabDPHI = tabulateDPHI(rr,tomo,Ndphi, indexL0)
 <tomo>            :  structure with all the needed information
 <Ndphi>           :  size of rr
 <indexL0>         :  link between the index of the studied layer and the index
 of the precomputed one.

 Computes the phase structure function for a separation rr(x,y).
 The r0 is not taken into account : the final result of DPHI(x,y,L0)
 has to be scaled with r0^-5/3, with r0 expressed in meters, to get
 the right value.

 Computes the phase structure for each different L0 and give a array (indexL0)
 to link the index of the layer i and the index of tabDPHI : for the layer l,
 DPHI = DPHI( du, dv, indexL0[l],rr,tabDPHI, convert). SEE ALSO: DPHI
 */
{
  // Assume one thread per element
  int nblocks = (Ndphi * Nl0) / tabDPHI_thread_x +
                (((Ndphi * Nl0) % tabDPHI_thread_x) != 0);
  dim3 dimBlock(tabDPHI_thread_x, 1);
  dim3 dimGrid(nblocks, 1);

  // tabulateDPHI_lowpass_kernel<<<dimGrid, dimBlock, 0,
  // cphim_struct->cphim_stream>>>(tab_dphi,cphim_struct->tab_int_x,
  // cphim_struct->tab_int_y, cphim_struct->dx, L0diff_d, Nl0, Ndphi, convert,
  // convert_int, cphim_struct->int_npts);
  carmaCheckMsg("tabulateDPHI_gpu_gb_kernel<<<>>> execution failed\n");
  // CudaCheckError();
}

//============================================================================================
//============================ MATCOV V3/V4 DPHI/SUBAP FUNCTIONS
//=============================
//============================================================================================
void tab_dphi_gpu_gb(double *tab_dphi, struct gtomo_struct *tomo_gpu,
                     long Ndphi, double *L0diff_d, int Nl0, double convert)
// void tabulateDPHI_gpu_gb(double* tabDPHI_d, double* rr_d,struct tomo_struct
// tomo, long Ndphi, long *indexL0_h)
/* DOCUMENT tabDPHI = tabulateDPHI(rr,tomo,Ndphi, indexL0)
 <tomo>            :  structure with all the needed information
 <Ndphi>           :  size of rr
 <indexL0>         :  link between the index of the studied layer and the index
 of the precomputed one.

 Computes the phase structure function for a separation rr(x,y).
 The r0 is not taken into account : the final result of DPHI(x,y,L0)
 has to be scaled with r0^-5/3, with r0 expressed in meters, to get
 the right value.

 Computes the phase structure for each different L0 and give a array (indexL0)
 to link the index of the layer i and the index of tabDPHI : for the layer l,
 DPHI = DPHI( du, dv, indexL0[l],rr,tabDPHI, convert). SEE ALSO: DPHI
 */
{
  // Assume one thread per element
  int nblocks = (Ndphi * Nl0) / tabDPHI_thread_x +
                (((Ndphi * Nl0) % tabDPHI_thread_x) != 0);
  dim3 dimBlock(tabDPHI_thread_x, 1);
  dim3 dimGrid(nblocks, 1);

  tabulateDPHI_gpu_gb_kernel<<<dimGrid, dimBlock, 0, tomo_gpu->matcov_stream>>>(
      tab_dphi, L0diff_d, Nl0, Ndphi, convert);
  carmaCheckMsg("tabulateDPHI_gpu_gb_kernel<<<>>> execution failed\n");
  // CudaCheckError();
}
//------------------------------------------------------------------------------------
// extern "C"
void sub_pos_gpu_gb(struct gtomo_struct *tomo_gpu, long Nlayer)
// void subap_position_gpu_gb(struct tomo_struct tomo, double ***u, double ***v)
/* DOCUMENT DOCUMENT         subap_position(tomo, u, v)
   <tomo>                : structure with all the needed information.
   <u> and <v>           : 3d arrays containing the sub-apertures projected
   coordinates onto all the layers. u[0][2][1] is the X-coordinate of the subap
   2 of the WFS 0 on the layer 1.

   Computes the projected coordinates of all subapertures  projected onto all
   the layer
 */
{
  int msize = Nlayer * tomo_gpu->Nx;
  int nblocks = msize / tabDPHI_thread_x + ((msize % tabDPHI_thread_x) != 0);
  dim3 dimBlock(tabDPHI_thread_x, 1);
  dim3 dimGrid(nblocks, 1);
  /*
    int nb = (int)(2);
      long *tmp;
      tmp=(long*)malloc((nb)*sizeof(long));
      carmaSafeCall(hipMemcpy(tmp, tomo_gpu->ioff_d, sizeof(long) * nb,
                    hipMemcpyDeviceToHost));
      for (int ii = 0 ; ii < nb ; ii++){
          printf("%5.5d \n",tmp[ii]);
      }
  */
  // std::cout << "Nsubap : " << Nsubap << std::endl;
  subposition_gpu_gb_kernel<<<dimGrid, dimBlock, 0, tomo_gpu->matcov_stream>>>(
      tomo_gpu->Nw, tomo_gpu->Nx, tomo_gpu->Nsubap_d, Nlayer,
      tomo_gpu->alphaX_d, tomo_gpu->alphaY_d, tomo_gpu->h_d, tomo_gpu->GsAlt_d,
      tomo_gpu->Nssp_d, tomo_gpu->diamPup_d, tomo_gpu->thetaML_d,
      tomo_gpu->ioff_d, tomo_gpu->X_d, tomo_gpu->Y_d, tomo_gpu->XPup_d,
      tomo_gpu->YPup_d, tomo_gpu->u_d, tomo_gpu->v_d);
  carmaCheckMsg("subposition_gpu_gb_kernel<<<>>> execution failed\n");
  /*
     int nb = (int)tomo_gpu->Nx * Nlayer;
        double *tmpp;
        tmpp=(double*)malloc((nb)*sizeof(double));
        carmaSafeCall(hipMemcpy(tmpp, tomo_gpu->v_d, sizeof(double) * nb,
                            hipMemcpyDeviceToHost));
        for (int ii = 0 ; ii < nb ; ii++){
          printf("%5.5f \n",tmpp[ii]);
        }
        */
  // CudaCheckError();
}

//============================================================================================
//=============================== TOMO UPDATE FUNCTIONS
//======================================
//============================================================================================
void update_tomo_atm_gpu_gb(struct gtomo_struct *tomo_gpu,
                            sutra_sensors *sensors, sutra_atmos *atmos,
                            double *L0, double *cn2, double *alphaX,
                            double *alphaY) {
  hipError_t e;

  double h[atmos->nscreens];
  int ii = 0;
  for (map<float, sutra_tscreen *>::iterator it = atmos->d_screens.begin();
       it != atmos->d_screens.end(); ++it) {
    h[ii] = (double)it->second->altitude;
    ii++;
  }
  // DEBUG_TRACE("Here !\n");
  double dmax = 0.0;
  double maxalt = h[atmos->nscreens - 1];
  long minssp = tomo_gpu->Nssp[0];
  for (int cc = 0; cc < tomo_gpu->Nw; cc++) {
    double tmp = sqrtf(alphaX[cc] * alphaX[cc] + alphaY[cc] * alphaY[cc]);
    if (tmp > dmax) dmax = tmp;
    if (minssp > tomo_gpu->Nssp[cc]) minssp = tomo_gpu->Nssp[cc];
  }
  const double crmax =
      dmax * 2 * maxalt + (1 + 1. / minssp) * tomo_gpu->DiamTel;
  const double pasDPHI = 1. / tomo_gpu->pasDPHI;  // inverse du pas de rr
  const long Ndphi = floor(crmax * pasDPHI) + 1;
  // const double convert = (double)(Ndphi-1)/(crmax+1./pasDPHI);

  e = hipMemcpyAsync(tomo_gpu->h_d, h, atmos->nscreens * sizeof(double),
                      hipMemcpyHostToDevice, tomo_gpu->matcov_stream);
  process_err(e, "copy gpu h_d");
  // DEBUG_TRACE("HERE !");

  e = hipMemcpyAsync(tomo_gpu->cn2_d, cn2, atmos->nscreens * sizeof(double),
                      hipMemcpyHostToDevice, tomo_gpu->matcov_stream);
  process_err(e, "copy gpu cn2_d");
  double *sspSizeL =
      (double *)malloc(sizeof(double) * tomo_gpu->Nw * atmos->nscreens);
  for (int cc = 0; cc < tomo_gpu->Nw * atmos->nscreens; cc++) {
    int n = cc / atmos->nscreens;
    int l = cc - n * atmos->nscreens;
    if (n >= sensors->nsensors()) n -= 1;
    sspSizeL[cc] = (((double)(tomo_gpu->DiamTel / sensors->d_wfs[n]->nxsub)) *
                    (1. - tomo_gpu->GsAlt[n] * h[l]));
  }
  // DEBUG_TRACE("HERE !");

  e = hipMemcpyAsync(tomo_gpu->sspSizeL_d, sspSizeL,
                      tomo_gpu->Nw * atmos->nscreens * sizeof(double),
                      hipMemcpyHostToDevice, tomo_gpu->matcov_stream);
  process_err(e, "copy gpu sspSizeL_d");
  hipStreamSynchronize(tomo_gpu->matcov_stream);
  // Search the different L0 and build indexL0
  const long Nlayer = atmos->nscreens;
  long i, j;
  int cpt = 1;
  double tmp[Nlayer];
  long indexL0[Nlayer];
  tmp[0] = L0[0];
  indexL0[0] = 0;
  // DEBUG_TRACE("HERE !");

  for (i = 1; i < Nlayer; i++) {
    j = 0;
    const double l0 = L0[i];

    while ((j < cpt) && (tmp[j] != l0)) {
      j++;
    }

    indexL0[i] = j;

    if (j == cpt) {
      tmp[j] = l0;
      cpt++;
    }
  }
  e = hipMemcpyAsync((tomo_gpu->indexL0_d), indexL0,
                      atmos->nscreens * sizeof(long), hipMemcpyHostToDevice,
                      tomo_gpu->matcov_stream);
  process_err(e, "copy gpu indexL0_d");
  // DEBUG_TRACE("HERE !");

  int Nl0 = cpt;
  /*
  double L0diff[Nl0];
   //DEBUG_TRACE("Cpt = %d ",cpt);
  // allocate space for L0
  if ((tomo_gpu->L0diff_d) != NULL){hipFree(tomo_gpu->L0diff_d);
  //DEBUG_TRACE("HERE !");

  e = hipMalloc((void**)&(tomo_gpu->L0diff_d), Nlayer*sizeof(double));
  process_err(e, "alloc gpu L0diff_d");
  for (i = 0; i < Nl0; i++)  {
    L0diff[i] = tmp[i];
  }

  */
  if ((tomo_gpu->L0diff_d) != NULL) hipFree(tomo_gpu->L0diff_d);
  e = hipMalloc((void **)&(tomo_gpu->L0diff_d), Nlayer * sizeof(double));
  // DEBUG_TRACE("HERE !");

  // offload L0diff
  e = hipMemcpyAsync(tomo_gpu->L0diff_d, L0, Nlayer * sizeof(double),
                      hipMemcpyHostToDevice, tomo_gpu->matcov_stream);
  process_err(e, "offload L0diff");
  // précalcul de DPHI : que pour chaque différent L0
  if ((tomo_gpu->tabDPHI_d) != NULL) {
    hipFree(tomo_gpu->tabDPHI_d);
  }
  // printf("tabDPHI alloc \n");
  e = hipMalloc((void **)&(tomo_gpu->tabDPHI_d), Nl0 * Ndphi * sizeof(double));
  process_err(e, "alloc gpu tabDPHI_d");
  // DEBUG_TRACE("HERE !");
  // tab_dphi_gpu_gb(tomo_gpu->tabDPHI_d, tomo_gpu, Ndphi, tomo_gpu->L0diff_d,
  // Nl0,convert);
  // carmaSafeCall(hipDeviceSynchronize());

  // %%%%%%% Computation of the sub-apertures positions and sizes %%%%%%%%%%%
  // u, v :arrays containing all the sub-apertures coordinates of all WFS, one
  // after the other u[0][1][3] is the X-coordinate of subap number 3 of wfs
  // number 0 at altitude 3
  // DEBUG_TRACE("HERE !");

  // Computes  u and v
  sub_pos_gpu_gb(tomo_gpu, (long)atmos->nscreens);
  // DEBUG_TRACE("HERE !");

  carmaSafeCall(hipDeviceSynchronize());

  if (sspSizeL) free(sspSizeL);
  // DEBUG_TRACE("Here !\n");
}
//---------------------------------------------------------------------------------
void update_tomo_sys_gpu_gb(struct gtomo_struct *tomo_gpu,
                            sutra_sensors *sensors, double *alphaX,
                            double *alphaY) {
  hipError_t e;

  long ioff[tomo_gpu->Nw];
  ioff[0] = 0;
  for (int i = 1; i < tomo_gpu->Nw; i++) {
    ioff[i] = ioff[i - 1] + sensors->d_wfs[i - 1]->nvalid;
  }

  e = hipMemcpyAsync(tomo_gpu->ioff_d, ioff, tomo_gpu->Nw * sizeof(long),
                      hipMemcpyHostToDevice, tomo_gpu->matcov_stream);
  process_err(e, "copy gpu ioff_d");

  e = hipMemcpyAsync(tomo_gpu->alphaX_d, alphaX, tomo_gpu->Nw * sizeof(double),
                      hipMemcpyHostToDevice, tomo_gpu->matcov_stream);
  process_err(e, "copy gpu alphaX_d");

  e = hipMemcpyAsync(tomo_gpu->alphaY_d, alphaY, tomo_gpu->Nw * sizeof(double),
                      hipMemcpyHostToDevice, tomo_gpu->matcov_stream);
  process_err(e, "copy gpu alphaY_d");

  e = hipMemcpyAsync(tomo_gpu->GsAlt_d, tomo_gpu->GsAlt,
                      tomo_gpu->Nw * sizeof(double), hipMemcpyHostToDevice,
                      tomo_gpu->matcov_stream);
  process_err(e, "copy gpu GsAlt_d");

  e = hipMemcpyAsync(tomo_gpu->Nssp_d, tomo_gpu->Nssp,
                      tomo_gpu->Nw * sizeof(long), hipMemcpyHostToDevice,
                      tomo_gpu->matcov_stream);
  process_err(e, "copy gpu Nssp_d");

  e = hipMemcpyAsync(tomo_gpu->Nsubap_d, tomo_gpu->Nsubap,
                      tomo_gpu->Nw * sizeof(long), hipMemcpyHostToDevice,
                      tomo_gpu->matcov_stream);
  process_err(e, "copy gpu Nsubap_d");

  e = hipMemcpyAsync(tomo_gpu->diamPup_d, tomo_gpu->diamPup,
                      tomo_gpu->Nw * sizeof(double), hipMemcpyHostToDevice,
                      tomo_gpu->matcov_stream);
  process_err(e, "copy gpu diamPup_d");

  e = hipMemcpyAsync(tomo_gpu->XPup_d, tomo_gpu->XPup,
                      tomo_gpu->Nw * sizeof(double), hipMemcpyHostToDevice,
                      tomo_gpu->matcov_stream);
  process_err(e, "copy gpu XPup_d");

  e = hipMemcpyAsync(tomo_gpu->YPup_d, tomo_gpu->YPup,
                      tomo_gpu->Nw * sizeof(double), hipMemcpyHostToDevice,
                      tomo_gpu->matcov_stream);
  process_err(e, "copy gpu YPup_d");

  e = hipMemcpyAsync(tomo_gpu->thetaML_d, tomo_gpu->thetaML,
                      tomo_gpu->Nw * sizeof(double), hipMemcpyHostToDevice,
                      tomo_gpu->matcov_stream);
  process_err(e, "copy gpu thetaML_d");
  // DEBUG_TRACE("Update \n");

  double *X;
  double *Y;
  int *tmpX;
  int *tmpY;
  X = (double *)malloc((tomo_gpu->Nx) * sizeof(double));
  Y = (double *)malloc((tomo_gpu->Nx) * sizeof(double));
  tmpX = (int *)malloc((tomo_gpu->Nx) * sizeof(int));
  tmpY = (int *)malloc((tomo_gpu->Nx) * sizeof(int));
  int ind = 0;
  double p2m;
  for (int i = 0; i < tomo_gpu->Nw; i++) {
    e = hipMemcpyAsync(tmpX, sensors->d_wfs[i]->d_validsubsx->getData(),
                        sizeof(int) * sensors->d_wfs[i]->nvalid,
                        hipMemcpyDeviceToHost, tomo_gpu->matcov_stream);
    process_err(e, "copy tmpX");
    e = hipMemcpyAsync(tmpY, sensors->d_wfs[i]->d_validsubsy->getData(),
                        sizeof(int) * sensors->d_wfs[i]->nvalid,
                        hipMemcpyDeviceToHost, tomo_gpu->matcov_stream);
    process_err(e, "copy tmpY");
    p2m = (tomo_gpu->DiamTel / (double)sensors->d_wfs[i]->nxsub) /
          ((double)(tmpX[1] - tmpX[0]));

    for (int j = 0; j < sensors->d_wfs[i]->nvalid; j++) {
      X[ind + j] = ((double)tmpX[j] * p2m) -
                   (double)((tomo_gpu->DiamTel / 2.) *
                            (1. - 1. / (double)sensors->d_wfs[i]->nxsub));
      Y[ind + j] = ((double)tmpY[j] * p2m) -
                   (double)((tomo_gpu->DiamTel / 2.) *
                            (1. - 1. / (double)sensors->d_wfs[i]->nxsub));
    }
    ind += sensors->d_wfs[i]->nvalid;
  }
  /*
    for (int ii = 0; ii<tomo_gpu->Nx ; ii++){
          std::cout << "X : " << X[ii] << std::endl;
    }
    for (int jj = 0; jj<tomo_gpu->Nx ; jj++){
          std::cout << "Y : " << Y[jj] << std::endl;
    }
  */
  // generateXY(tomo_gpu,sensors);
  e = hipMemcpyAsync(tomo_gpu->X_d, X, tomo_gpu->Nx * sizeof(double),
                      hipMemcpyHostToDevice, tomo_gpu->matcov_stream);
  process_err(e, "copy gpu X_d");
  e = hipMemcpyAsync(tomo_gpu->Y_d, Y, tomo_gpu->Nx * sizeof(double),
                      hipMemcpyHostToDevice, tomo_gpu->matcov_stream);
  process_err(e, "copy gpu Y_d");
  // hipStreamSynchronize(tomo_gpu->matcov_stream);
  // DEBUG_TRACE("Update \n");
  hipStreamSynchronize(tomo_gpu->matcov_stream);
  /*
    int nb = (int)(408);
    double *tmp;
    tmp=(double*)malloc((nb)*sizeof(double));
    carmaSafeCall(hipMemcpy(tmp, tomo_gpu->Y_d, sizeof(double) * nb,
                    hipMemcpyDeviceToHost));
    for (int ii = 0 ; ii < nb ; ii++){
          printf("%5.5f \n",tmp[ii]);
    }
  */
}

void generateXY(struct gtomo_struct *tomo, sutra_sensors *sensors)
/* DOCUMENT  generateXY(struct tomo_struct tomo, double *Nsubap)
 <tomo>               :  structure with all the needed information
 <tomo.X> & <tomo.Y>            :   arrays containing all the sub-apertures
coordinates of all WFS, one after the other <tomo.Nsubap>              :  number
of subaperture of ezach WFS Generate the position (X,Y) of each subapertures of
each WFS on the telescope pupil and the number of subaperture of ezach WFS
(Nsubap)
 */
{
  const double bornemin = -tomo->DiamTel / 2.;
  const double Rtel2 = (tomo->DiamTel * tomo->DiamTel) / 4.;
  long NsubapTot = 0;
  long n;

  // Total number of subapertures (without obstruction)
  for (n = 0; n < tomo->Nw; n++) {
    NsubapTot += tomo->Nssp[n] * tomo->Nssp[n];
  }
  // DEBUG_TRACE("Here !\n");
  const long cNsubapTot = NsubapTot;
  double x[cNsubapTot], y[cNsubapTot];
  int index[cNsubapTot];

  int cpt = 0;
  int ioff = 0;

  // Computation of all the subapertures' positions
  for (n = 0; n < tomo->Nw; n++) {
    long Nsap = 0;
    double pas = tomo->DiamTel / (1. * tomo->Nssp[n]);
    int i;
    double Robs2;

    // to avoid some bug that eliminates useful central subapertures when
    // obs=0.286
    if (tomo->Nssp[n] != 7 || (tomo->obs <= 0.285 || tomo->obs >= 0.29)) {
      Robs2 = tomo->DiamTel * tomo->obs / 2. * tomo->DiamTel * tomo->obs / 2.;
    } else {
      Robs2 = tomo->DiamTel * 0.285 / 2. * tomo->DiamTel * 0.285 / 2.;
    }
    // DEBUG_TRACE("Here !\n");
    if (tomo->Nssp[n] != 1) {
      for (i = 0; i < tomo->Nssp[n]; i++) {
        double tp =
            bornemin + pas / 2. * (2. * i + 1.);  // y-coord of current subap
        int j;

        for (j = 0; j < tomo->Nssp[n]; j++) {
          x[ioff + j] =
              bornemin + pas / 2. * (2. * j + 1.);  // x-coord of current subap
          y[ioff + j] = tp;

          double r2 = x[ioff + j] * x[ioff + j] + y[ioff + j] * y[ioff + j];
          // DEBUG_TRACE("Here !\n");
          // Search the non-valid subapertures
          if (r2 < Robs2 || r2 >= Rtel2) {
            index[cpt] = j + ioff;  // list of the useless subapertures index
            cpt++;
          } else {
            Nsap++;
          }
        }
        ioff += tomo->Nssp[n];
      }
      // tomo->Nsubap[n] = Nsap;
    } else {         // Special case (Nssp = 1)
      x[ioff] = 0.;  // x-coord of current subap
      y[ioff] = 0.;
      ioff += tomo->Nssp[n];
      // tomo->Nsubap[n] = 1;
    }
  }

  double *X;
  double *Y;
  std::cout << "sizeX = " << cNsubapTot - cpt << std::endl;
  X = (double *)malloc((cNsubapTot - cpt) * sizeof(double));
  Y = (double *)malloc((cNsubapTot - cpt) * sizeof(double));
  tomo->Nx = cNsubapTot - cpt;

  int a = 0;
  int off = 0;
  int borne = 0;
  int i;
  // Suppress the non-valid subapertures
  while (a <= cpt) {
    if (a == cpt) {
      borne = cNsubapTot;
    } else {
      borne = index[a];
    }

    for (i = off; i < borne; i++) {
      X[i - a] = x[i];
      Y[i - a] = y[i];
    }

    off = index[a] + 1;
    a++;
  }
  hipError_t e;
  e = hipMemcpyAsync(tomo->X_d, X, tomo->Nx * sizeof(double),
                      hipMemcpyHostToDevice, tomo->matcov_stream);
  process_err(e, "copy gpu X_d");
  e = hipMemcpyAsync(tomo->Y_d, Y, tomo->Nx * sizeof(double),
                      hipMemcpyHostToDevice, tomo->matcov_stream);
  process_err(e, "copy gpu Y_d");
}

//============================================================================================
//============================= MATCOV 3
//=====================================================
//============================================================================================
// extern "C"
/*
void matcov_gpu_3(double* data, int nrows, int ncols, int xoffset, int yoffset,
int lda, struct tomo_struct tomo, struct gtomo_struct *tomo_gpu)
{
        /* *** matcov gpu kernel driver ***
        *  Arguments
        *  ==========
        *  data		double pointer: A pointer to the matrix/submatrix to be
generated. It
        *  			should always point to the first element in a
matrix/submatrix
        *
        *  nrows	integer: The number of rows of the matrix/submatrix to
be generated
        *
        *  ncols	integer: The number of columns of the matrix/submatrix
to be generated
        *
        *  xoffset	integer: The x-offset of the submatrix, must be zero if
the entire matrix
        *			is generated. Its the x-coordinate of the first
element in the matrix/submatrix
        *
        *  yoffset  integer: The y-offset of the submatrix, must be zero if the
entire matrix
        *			is generated. Its the y-coordinate of the first
element in the matrix/submatrix
        *
        *  lda		integer: The leading dimension of the matrix/submatrix
        */
/*
  //hipError_t e;

  char uplo, copy;

  uplo = 'f';	// full generation is enabled by default
  copy = 'c';

  int type_mat = tomo.part;

  if(type_mat == 1) // Caa matrix
  {
        // check if a square diagonal tile is generated then we set uplo to 'l'
or 'u'
        // and then enable the copy
        // This also applies if the entire matrix will be generated
        // otherwise (off diagonal tile or non square submatrix) - full
generation is assumed
        if((xoffset == yoffset) && (nrows == ncols))	// if sqaure & diagonal
        {
                uplo = 'l';
                if(type_mat == 1)copy = 'c';
        }
        else	// full generation, copy is ignored
        {
                uplo = 'f';
        }
  }
  else if(type_mat == 2 || type_mat == 3) // Cmaa matrix
  {
        uplo = 'f';		// full generation, copy is ignored
  }
  else
  {
        printf("ERROR: unrecognized type_mat %d \n", type_mat); exit(1);
  }

  // %%%%%%% Pre-computation of DPHI %%%%%%%%%%
  //Computes an array of DPHI (tabDPHI) for an array of subaperture distance rr
for each DIFFERENT L0
  //const long Nw = tomo.Nw;
  const double crmax = tomo.rmax;
  const double pasDPHI = 1./tomo.pasDPHI; //inverse du pas de rr
  const long Ndphi = floor(crmax*pasDPHI)+1;
  const double convert = (double)(Ndphi-1)/(crmax+1./pasDPHI);

  //int size = tomo.Nslopes - 2 * tomo.Nsubap[tomo.Nw-1];

  int nbx = nrows / matcov_thread_x + (nrows%matcov_thread_x != 0);
  int nby = ncols / matcov_thread_y + (ncols%matcov_thread_y != 0);

  dim3 dimBlock(matcov_thread_x, matcov_thread_y);
  dim3 dimGrid(nbx, nby);
  const long Nsubap = tomo.Nsubap[0];

  // generate a full matrix
  matcov_kernel_3<<<dimGrid, dimBlock, 0, tomo_gpu->matcov_stream>>>(uplo, copy,
data, nrows, ncols, xoffset, yoffset, lda,
                                           convert,tomo_gpu->sspSizeL_d,tomo_gpu->Nssp_d,tomo_gpu->u_d,tomo_gpu->v_d,
                                           pasDPHI,tomo_gpu->tabDPHI_d,tomo_gpu->indexL0_d,tomo_gpu->cn2_d,
                                           Ndphi,tomo.Nw,atmos->nscreens,Nsubap,type_mat,tomo.DiamTel);

  //if (type_mat == 1)
  //  matcov_kernel_copy<<<dimGrid, dimBlock>>>(data, nrows, ncols, xoffset,
yoffset, lda);

  //hipStreamSynchronize(tomo_gpu->matcov_stream);
}
*/

//============================================================================================
//=========================== MATCOV 4 (NOISE) KERNELS/FUNCTION
//==============================
//============================================================================================
__device__ double compute_element_4(
    int ipos, int jpos, double convert, double *sspSizeL, long *Nssp, double *u,
    double *v, double pasDPHI, double *tabDPHI, double *indexL0, double *cn2,
    int Ndphi, int Nw, int Nlayer, long *Nsubap_wfs, long Nx, double *alphaX,
    double *alphaY, double lgs_cst, double noise_var, double spotWidth,
    double dH_lgs, double alt_lgs, int type_mat, int nlgs, double teldiam) {
  /* *** Covariance matrix per-element generation ***
   *   Arguments
   *   =========
   *	ipos:		Integer: global x-coordinate of the element w.r.t. the
   *entire matrix jpos:		Integer: global y-coordinate of the element
   *w.r.t. the entire matrix
   */

  // for now return a dummy value

  const double lambda2 = 0.00026942094446267851;
  // long Nsubap = Nsubap_wfs[0];
  // WFS m

  long Nsubapx = Nsubap_wfs[0];
  int m = 0;
  while ((ipos / (2 * Nsubapx)) >= 1) {
    m++;
    Nsubapx += Nsubap_wfs[m];
  }
  Nsubapx -= Nsubap_wfs[m];

  // int m = ipos / (2 * Nsubap);
  if (type_mat == 3) m = Nw - 1;
  // WFS n

  long Nsubapy = Nsubap_wfs[0];
  int n = 0;
  while ((jpos / (2 * Nsubapy)) >= 1) {
    n++;
    Nsubapy += Nsubap_wfs[n];
  }
  Nsubapy -= Nsubap_wfs[n];

  // int n = jpos / (2 * Nsubap);
  if (type_mat == 2) n = Nw - 1;
  // subap i
  // int i = ipos % (2 * Nsubap_wfs[m]);
  int i = ipos - 2 * Nsubapx;
  // subap j
  // int j = jpos % (2 * Nsubap_wfs[n]);
  int j = jpos - 2 * Nsubapy;
  // xy i
  int xy_i;
  // xy j
  int xy_j;
  if (i >= Nsubap_wfs[m]) {
    i -= Nsubap_wfs[m];
    xy_i = 1;
  } else
    xy_i = 0;
  if (j >= Nsubap_wfs[n]) {
    j -= Nsubap_wfs[n];
    xy_j = 1;
  } else
    xy_j = 0;

  const double sspSizem = teldiam / Nssp[m];
  const double sspSizen = teldiam / Nssp[n];

  const double kk = lambda2 / (sspSizem * sspSizen);

  int type = xy_i * 2 + xy_j;

  // Layer l
  double covar = 0.0;
#pragma unroll
  for (int l = 0; l < Nlayer; l++) {
    double sspSizeml = sspSizeL[m * Nlayer + l];
    double sspSizenl = sspSizeL[n * Nlayer + l];
    // test if the altitude layers is not higher than the LGS altitude
    if ((sspSizeml > 0) && (sspSizenl > 0)) {
      int pos1 = i + Nsubapx + l * Nx;
      int pos2 = j + Nsubapy + l * Nx;
      // if(threadIdx.x == 6 && threadIdx.y == 0 && blockIdx.x == 6 &&
      // blockIdx.y == 1) if((pos1 >= 6840) || (pos2 >= 6839))
      //{
      //	printf("================ pos1 = %d, pos2 = %d \n", pos1, pos2);
      //}
      //(6,0,0) in block (0,2,0);
      double du = u[pos1] - u[pos2];
      double dv = v[pos1] - v[pos2];

      double s1 = sspSizeml * 0.5;
      double s2 = sspSizenl * 0.5;

      double ac = s1 - s2;
      double ad = s1 + s2;
      double bc = -ad;  // initially -s1-s2;
      double bd = -ac;  // initially -s1+s2;

      if (type == 0)
        covar += 0.5 /* pasDPHI*/ *
                 cov_XX_gpu_gb(du, dv, ac, ad, bc, bd, tabDPHI, indexL0[l],
                               convert, Ndphi) *
                 kk * cn2[l];
      else if (type == 3)
        covar += 0.5 /* pasDPHI*/ *
                 cov_YY_gpu_gb(du, dv, ac, ad, bc, bd, tabDPHI, indexL0[l],
                               convert, Ndphi) *
                 kk * cn2[l];
      else {  // if ((type == 1) || (type == 2))
        double s0 =
            sqrt(s1 * s1 + s2 * s2);  // half size of the subaperture equivalent
                                      // to a convolution by s1 and s2
        double dd =
            (s1 > s2) ? 1. - s2 / s1 : 1. - s1 / s2;  // Nono's style ....
        covar +=
            0.25 /* pasDPHI*/ *
            cov_XY_gpu_gb(du, dv, s0, tabDPHI, indexL0[l], convert, Ndphi) *
            kk * cn2[l] * (1. - dd * dd);
      }
    }
  }
  // adding noise

  if (m == n) {
    if (m < nlgs) {
      if (i == j) {
        // lgs case
        const int pos1 = i + Nsubapx;
        double x = u[pos1];
        double y = v[pos1];
        double xwfs = alphaX[m] * 206265;
        double ywfs = alphaY[m] * 206265;
        double lltx = 0;
        double llty = 0;
        const double lltnorm = sqrtf(xwfs * xwfs + ywfs * ywfs);
        if (lltnorm != 0) {
          lltx = xwfs / lltnorm * teldiam / 2.0;
          llty = ywfs / lltnorm * teldiam / 2.0;
        }
        x -= lltx;
        y -= llty;
        x = 206265. * dH_lgs * x / alt_lgs /
            alt_lgs;  // extension at Fwhm, in arcsec
        y = 206265. * dH_lgs * y / alt_lgs /
            alt_lgs;                           // extension at Fwhm, in arcsec
        double lgsExt = sqrtf(x * x + y * y);  // lengh of the extension
        double lgsTheta = x != 0 ? atanf(y / x) : 0.0;  // angle of extension
        double totalExt = sqrtf(lgsExt * lgsExt + spotWidth * spotWidth);
        // lengh of the extension including seeing, laser size, ...
        double ratio = totalExt / spotWidth;
        double noiseLongAxis = noise_var * ratio * ratio;
        if (type == 0)
          covar += noiseLongAxis * cosf(lgsTheta) * cosf(lgsTheta) +
                   noise_var * sinf(lgsTheta) * sinf(lgsTheta);
        else if (type == 3)
          covar += noiseLongAxis * sinf(lgsTheta) * sinf(lgsTheta) +
                   noise_var * cosf(lgsTheta) * cosf(lgsTheta);
        else
          covar +=
              (noiseLongAxis - noise_var) * sinf(lgsTheta) * cosf(lgsTheta);
      }
      if ((type == 0) || (type == 3)) covar += lgs_cst;
    } else {
      // ngs case
      if (i == j) {
        if ((type == 0) || (type == 3)) {
          covar += noise_var;
        }
      }
    }
  }

  return (double)covar;
}

__device__ double compute_cphim_element(
    int ipos, int jpos, double convert, double *sspSizeL, long *Nssp, double *u,
    double *v, double *xact, double *yact, double xref, double yref, long npts,
    double *L0, double *cn2, int Ndphi, int Nw, int Ndm, int Nlayer,
    long *Nsubap, long Nx, long *Nactu_tot, int Nact, long *NlayerDM,
    long *indLayerDm, double *dx, double *alphaX, double *alphaY,
    double lgs_cst, double noise_var, double spotWidth, double dH_lgs,
    double alt_lgs, double *Hlayer, double *Hdm, double FoV, int nlgs,
    double teldiam, double *k2, double *tab_int_x, double *tab_int_y) {
  /* *** Covariance matrix per-element generation ***
   *   Arguments
   *   =========
   *	ipos:		Integer: global x-coordinate of the element w.r.t. the
   *entire matrix jpos:		Integer: global y-coordinate of the element
   *w.r.t. the entire matrix
   */

  // for now return a dummy value

  const double lambda2 =
      0.016414031750058719;  // RASC * 0.5 * 1e-6 / 2. / pi ie lambda = 0.5e-6

  // DM m
  int m = 0;
  long Nactux = Nactu_tot[0];
  while ((ipos / Nactux) >= 1) {
    m++;
    Nactux += Nactu_tot[m];
  }
  Nactux -= Nactu_tot[m];
  // WFS n
  long Nsubapx = Nsubap[0];
  int n = 0;  // jpos / (2 * Nsubap);
  while ((jpos / (2 * Nsubapx)) >= 1) {
    n++;
    Nsubapx += Nsubap[n];
  }
  Nsubapx -= Nsubap[n];
  // if (type_mat == 2) n = Nw-1;

  // Nact i
  int i = ipos - Nactux;
  // subap j
  int j = jpos - 2 * Nsubapx;
  int type = 0;
  if (j >= Nsubap[n]) {
    j -= Nsubap[n];
    type = 1;
  }
  /*
  //xy i
  int xy_i;
  //xy j
  int xy_j;
  if (i>=Nact/Ndm) {
    i-= Nact/Ndm;
    xy_i = 1;
  } else xy_i = 0;
  if (j>=Nsubap) {
    j-= Nsubap;
    xy_j = 1;
  } else xy_j = 0;
  */
  const double sspSizen = teldiam / Nssp[n];

  const double kk =
      lambda2 * k2[m] / sspSizen;  // k2 = y_wfs(1).lambda / 2. / pi /
                                   // y_dm(1).unitpervolt (Yorick computation)

  // Layer l
  double covar = 0.0;
  // long Nlayer4Dm = NlayerDM[m];
  long otherDM = 0;
  for (int ll = 0; ll < m; ll++) otherDM += NlayerDM[ll];

#pragma unroll
  for (int l = 0; l < Nlayer; l++) {
    long dolayer = 0;

    if (indLayerDm[l] == m) dolayer = 1;

    if (dolayer) {
      double sspSizenl = sspSizeL[n * Nlayer + l];
      // test if the altitude layers is not higher than the LGS altitude
      if ((sspSizenl > 0)) {
        int pos_act = i + Nactux;  // + l * Nact;
        int pos_ssp = j + Nsubapx + l * Nx;
        // int pos_act = ipos;
        // int pos_ssp = jpos;
        double deltah = abs(Hlayer[l] - Hdm[m]);
        // double pDiam = teldiam + 2 * FoV * Hdm[m];
        // double hproj = pDiam / FoV;
        // double xproj = xact[pos_act]/hproj;
        // double yproj = yact[ppos_act]/hproj;
        // double dX = xproj*deltah;
        // double dY = yproj*deltah;
        double du = xact[pos_act] - u[pos_ssp];
        double dv = yact[pos_act] - v[pos_ssp];

        double s2 = sspSizenl * 0.5;
        double fc = 0.5 / sqrt(dx[m] * dx[m] + FoV * FoV * deltah * deltah);

        // double ac = 0.;
        // double ad = 2*s2;
        // double bc = -ad;   // initially -s1-s2;
        // double bd = 0;   // initially -s1+s2;

        if (type == 0)
          covar += 0.5 *
                   cphim_XX(du, dv, u[pos_ssp], v[pos_ssp], xref, yref, s2,
                            L0[l], fc, npts, tab_int_x, tab_int_y) *
                   kk * cn2[l];
        else
          covar += 0.5 *
                   cphim_YY(du, dv, u[pos_ssp], v[pos_ssp], xref, yref, s2,
                            L0[l], fc, npts, tab_int_x, tab_int_y) *
                   kk * cn2[l];
      }
    }
  }

  return covar;
}

//------------------------------------------------------------------------------------------
__global__ void matcov_kernel_4(char uplo, char copy, float *data, int nrows,
                                int ncols, int xoffset, int yoffset, int lda,
                                double convert, double *sspSizeL, long *Nssp,
                                double *u, double *v, double pasDPHI,
                                double *tabDPHI, double *indexL0, double *cn2,
                                int Ndphi, int Nw, int Nlayer, long *Nsubap,
                                long Nx, double *alphaX, double *alphaY,
                                double lgs_cst, double noise_var,
                                double spotWidth, double dH_lgs, double alt_lgs,
                                int type_mat, int nlgs, double teldiam) {
  /* *** covariance matrix generation kernel ***
   *	The kernel generates the element values in a given matrix/submatrix
   *   The generation function can be any function, as long as each element
   *   can be computed both individually and independently
   *
   *	see argument description in the kernel driver
   */

  // local thread coordinates w.r.t. thread block
  const int tx_ = threadIdx.x;
  const int ty_ = threadIdx.y;

  // local thread block coordinates w.r.t. kernel grid
  const int bx_ = blockIdx.x;
  const int by_ = blockIdx.y;

  // local coordinates of the element w.r.t. submatrix
  int lx = bx_ * blockDim.x + tx_;
  int ly = by_ * blockDim.y + ty_;

  // global coordinates of the elemnt w.r.t. the entire matrix
  int gx = lx + xoffset;
  int gy = ly + yoffset;

  // out-of-bound threads should terminate
  if ((lx >= nrows) || (ly >= ncols)) return;

  // Advance the data pointer accordingly
  // data += ly * lda + lx;

  double value;
  if (uplo == 'l') {
    if (gy <= gx) {
      value = compute_element_4(
          gx, gy, convert, sspSizeL, Nssp, u, v, pasDPHI, tabDPHI, indexL0, cn2,
          Ndphi, Nw, Nlayer, Nsubap, Nx, alphaX, alphaY, lgs_cst, noise_var,
          spotWidth, dH_lgs, alt_lgs, type_mat, nlgs, teldiam);
      data[ly * lda + lx] = (float)value;
      if (copy == 'c') data[lx * lda + ly] = (float)value;
    }
  } else if (uplo == 'u') {  // upper
    if (gx <= gy) {
      value = compute_element_4(
          gx, gy, convert, sspSizeL, Nssp, u, v, pasDPHI, tabDPHI, indexL0, cn2,
          Ndphi, Nw, Nlayer, Nsubap, Nx, alphaX, alphaY, lgs_cst, noise_var,
          spotWidth, dH_lgs, alt_lgs, type_mat, nlgs, teldiam);
      data[ly * lda + lx] = (float)value;
      if (copy == 'c') data[lx * lda + ly] = (float)value;
    }
  } else {  // uplo = 'f' full generation
    value = compute_element_4(gx, gy, convert, sspSizeL, Nssp, u, v, pasDPHI,
                              tabDPHI, indexL0, cn2, Ndphi, Nw, Nlayer, Nsubap,
                              Nx, alphaX, alphaY, lgs_cst, noise_var, spotWidth,
                              dH_lgs, alt_lgs, type_mat, nlgs, teldiam);
    data[ly * lda + lx] = (float)value;
  }

  // if ((type_mat == 3) || (gx <= gy)) {
  //  // call the generation function
  //  data[0] = compute_element_4(gx, gy, convert, sspSizeL, Nssp, u, v,
  //  pasDPHI, tabDPHI, indexL0, cn2, Ndphi, Nw, Nlayer,
  //				Nsubap, alphaX, alphaY, lgs_cst, noise_var,
  // spotWidth, dH_lgs, alt_lgs, type_mat, nlgs, teldiam); printf("gx = %d, gy =
  // %d -----
  // %.2f \n", gx, gy, data[0]);
  //}
}

__global__ void CPHIM_kernel(
    float *data, int nrows, int ncols, int xoffset, int yoffset, int lda,
    double convert, double *sspSizeL, long *Nssp, double *u, double *v,
    double *xact, double *yact, double xref, double yref, long npts, double *L0,
    double *cn2, int Ndphi, int Nw, int Ndm, int Nlayer, long *Nsubap, long Nx,
    long *Nactu, int Nact, long *NlayerDM, long *indLayerDm, double *dx,
    double *alphaX, double *alphaY, double lgs_cst, double noise_var,
    double spotWidth, double dH_lgs, double alt_lgs, double *Hlayer,
    double *Hdm, double FoV, int nlgs, double teldiam, double *k2,
    double *tab_int_x, double *tab_int_y) {
  /* *** covariance matrix generation kernel ***
   *	The kernel generates the element values in a given matrix/submatrix
   *   The generation function can be any function, as long as each element
   *   can be computed both individually and independently
   *
   *	see argument description in the kernel driver
   */

  // local thread coordinates w.r.t. thread block
  const int tx_ = threadIdx.x;
  const int ty_ = threadIdx.y;

  // local thread block coordinates w.r.t. kernel grid
  const int bx_ = blockIdx.x;
  const int by_ = blockIdx.y;

  // local coordinates of the element w.r.t. submatrix
  int lx = bx_ * blockDim.x + tx_;
  int ly = by_ * blockDim.y + ty_;

  // global coordinates of the elemnt w.r.t. the entire matrix
  int gx = lx + xoffset;
  int gy = ly + yoffset;

  // out-of-bound threads should terminate
  if ((lx >= nrows) || (ly >= ncols)) return;

  // Advance the data pointer accordingly
  // data += ly * lda + lx;

  double value;

  value = compute_cphim_element(
      gx, gy, convert, sspSizeL, Nssp, u, v, xact, yact, xref, yref, npts, L0,
      cn2, Ndphi, Nw, Ndm, Nlayer, Nsubap, Nx, Nactu, Nact, NlayerDM,
      indLayerDm, dx, alphaX, alphaY, lgs_cst, noise_var, spotWidth, dH_lgs,
      alt_lgs, Hlayer, Hdm, FoV, nlgs, teldiam, k2, tab_int_x, tab_int_y);
  data[ly * lda + lx] = (float)value;
}
//============================================================================================
//============================= MATCOV 4 (NOISE)
//=============================================
//============================================================================================
void matcov_gpu_4(float *data, int nrows, int ncols, int xoffset, int yoffset,
                  int lda, struct gtomo_struct *tomo_gpu, sutra_atmos *atmos,
                  sutra_sensors *sensors, double *alphaX, double *alphaY) {
  /* *** matcov gpu kernel driver ***
   *  Arguments
   *  ==========
   *  data		double pointer: A pointer to the matrix/submatrix to be
   *generated. It should always point to the first element in a matrix/submatrix
   *
   *  nrows	integer: The number of rows of the matrix/submatrix to be
   *generated
   *
   *  ncols	integer: The number of columns of the matrix/submatrix to be
   *generated
   *
   *  xoffset	integer: The x-offset of the submatrix, must be zero if the
   *entire matrix is generated. Its the x-coordinate of the first element in the
   *matrix/submatrix
   *
   *  yoffset  integer: The y-offset of the submatrix, must be zero if the
   *entire matrix is generated. Its the y-coordinate of the first element in the
   *matrix/submatrix
   *
   *  lda		integer: The leading dimension of the matrix/submatrix
   */

  // hipError_t e;
  char uplo, copy;

  uplo = 'f';  // full generation is enabled by default
  copy = 'c';

  int type_mat = 1;

  if (type_mat == 1) {  // Caa matrix
    // check if a square diagonal tile is generated then we set uplo to 'l' or
    // 'u' and then enable the copy This also applies if the entire matrix will
    // be generated otherwise (off diagonal tile or non square submatrix) - full
    // generation is assumed
    if ((xoffset == yoffset) && (nrows == ncols)) {  // if sqaure & diagonal
      uplo = 'l';
      copy = 'c';
    } else {  // full generation, copy is ignored
      uplo = 'f';
    }
  }
  // else if(type_mat == 2) //
  else if (type_mat == 2 || type_mat == 3) {  // Cmaa matrix
    uplo = 'f';  // full generation, copy is ignored
  } else {
    printf("ERROR: unrecognized type_mat %d \n", type_mat);
    exit(1);
  }
  // uplo = 'f';
  // %%%%%%% Pre-computation of DPHI %%%%%%%%%%
  // Computes an array of DPHI (tabDPHI) for an array of subaperture distance rr
  // for each DIFFERENT L0
  double h[atmos->nscreens];
  int ii = 0;
  for (map<float, sutra_tscreen *>::iterator it = atmos->d_screens.begin();
       it != atmos->d_screens.end(); ++it) {
    h[ii] = (double)it->second->altitude;
    ii++;
  }

  double dmax = 0.0;
  double maxalt = h[atmos->nscreens - 1];
  int minssp = tomo_gpu->Nssp[0];
  for (int cc = 0; cc < tomo_gpu->Nw; cc++) {
    double tmp = sqrtf(alphaX[cc] * alphaX[cc] + alphaY[cc] * alphaY[cc]);
    if (tmp > dmax) dmax = tmp;
    if (tomo_gpu->Nssp[cc] < minssp) minssp = tomo_gpu->Nssp[cc];
  }
  const double crmax =
      dmax * 2 * maxalt + (1 + 1. / minssp) * tomo_gpu->DiamTel;

  const double pasDPHI = 1. / tomo_gpu->pasDPHI;  // inverse du pas de rr
  const long Ndphi = floor(crmax * pasDPHI) + 1;
  const double convert = (double)(Ndphi - 1) / (crmax + 1. / pasDPHI);

  int nbx = nrows / matcov_thread_x + (nrows % matcov_thread_x != 0);
  int nby = ncols / matcov_thread_y + (ncols % matcov_thread_y != 0);

  dim3 dimBlock(matcov_thread_x, matcov_thread_y);
  dim3 dimGrid(nbx, nby);
  const long Nsubap = sensors->d_wfs[0]->nvalid;  // tomo_gpu->Nx;
  /*
    int nb = (int)(atmos->nscreens*tomo_gpu->Nw);
    double *tmp;
    tmp=(double*)malloc((nb)*sizeof(double));
    carmaSafeCall(hipMemcpy(tmp, tomo_gpu->sspSizeL_d, sizeof(double) * nb,
                    hipMemcpyDeviceToHost));
    for (int ii = 0 ; ii < nb ; ii++){
          printf("%f \n",tmp[ii]);
    }
    //printf("convert : %9.9f\n",convert);
     */

  matcov_kernel_4<<<dimGrid, dimBlock, 0, tomo_gpu->matcov_stream>>>(
      uplo, copy, data, nrows, ncols, xoffset, yoffset, lda, convert,
      tomo_gpu->sspSizeL_d, tomo_gpu->Nssp_d, tomo_gpu->u_d, tomo_gpu->v_d,
      pasDPHI, tomo_gpu->tabDPHI_d, tomo_gpu->L0diff_d, tomo_gpu->cn2_d, Ndphi,
      tomo_gpu->Nw, atmos->nscreens, tomo_gpu->Nsubap_d, tomo_gpu->Nx,
      tomo_gpu->alphaX_d, tomo_gpu->alphaY_d, tomo_gpu->lgs_cst,
      (double)0.0 /*sensors->d_wfs[0]->noise*/, tomo_gpu->spot_width,
      tomo_gpu->lgs_depth, tomo_gpu->lgs_alt, type_mat, tomo_gpu->nlgs,
      tomo_gpu->DiamTel);
  carmaCheckMsg("matcov_kernel_4<<<>>> execution failed\n");
  hipStreamSynchronize(tomo_gpu->matcov_stream);
  /*
  int nb = (int)sensors->d_wfs[0]->nvalid * 2;
  nb = nb*nb;
    double *tmp;
    tmp=(double*)malloc((nb)*sizeof(double));
    carmaSafeCall(hipMemcpy(tmp, data, sizeof(double) * nb,
                    hipMemcpyDeviceToHost));
    for (int ii = 0 ; ii < nb ; ii++)
        std::cout << tmp[ii] << std::endl;
        */
  // if (type_mat == 1)
  // matcov_kernel_copy<<<dimGrid, dimBlock>>>(data, nrows, ncols, xoffset,
  // yoffset, lda);
}

//============================================================================================
//============================= CPHIM
//=============================================
//============================================================================================
void CPHIM(float *data, int nrows, int ncols, int xoffset, int yoffset, int lda,
           struct cphim_struct *cphim_struct, sutra_atmos *atmos,
           sutra_sensors *sensors, double *alphaX, double *alphaY,
           carma_device *device) {
  /* *** matcov gpu kernel driver ***
   *  Arguments
   *  ==========
   *  data		double pointer: A pointer to the matrix/submatrix to be
   *generated. It should always point to the first element in a matrix/submatrix
   *
   *  nrows	integer: The number of rows of the matrix/submatrix to be
   *generated
   *
   *  ncols	integer: The number of columns of the matrix/submatrix to be
   *generated
   *
   *  xoffset	integer: The x-offset of the submatrix, must be zero if the
   *entire matrix is generated. Its the x-coordinate of the first element in the
   *matrix/submatrix
   *
   *  yoffset  integer: The y-offset of the submatrix, must be zero if the
   *entire matrix is generated. Its the y-coordinate of the first element in the
   *matrix/submatrix
   *
   *  lda		integer: The leading dimension of the matrix/submatrix
   */

  // %%%%%%% Pre-computation of DPHI %%%%%%%%%%
  // Computes an array of DPHI (tabDPHI) for an array of subaperture distance rr
  // for each DIFFERENT L0

  double h[atmos->nscreens];
  int ii = 0;
  for (map<float, sutra_tscreen *>::iterator it = atmos->d_screens.begin();
       it != atmos->d_screens.end(); ++it) {
    h[ii] = (double)it->second->altitude;
    ii++;
  }

  double dmax = 0.0;
  double maxalt = h[atmos->nscreens - 1];
  int minssp = cphim_struct->Nssp[0];
  for (int cc = 0; cc < cphim_struct->Nw; cc++) {
    double tmp = sqrtf(alphaX[cc] * alphaX[cc] + alphaY[cc] * alphaY[cc]);
    if (tmp > dmax) dmax = tmp;
    if (cphim_struct->Nssp[cc] < minssp) minssp = cphim_struct->Nssp[cc];
  }
  const double crmax =
      dmax * 2 * maxalt + (1 + 1. / minssp) * cphim_struct->DiamTel;

  const double pasDPHI = 1. / cphim_struct->pasDPHI;  // inverse du pas de rr
  const long Ndphi = floor(crmax * pasDPHI) + 1;
  const double convert = (double)(Ndphi - 1) / (crmax + 1. / pasDPHI);

  int nbx = nrows / matcov_thread_x + (nrows % matcov_thread_x != 0);
  int nby = ncols / matcov_thread_y + (ncols % matcov_thread_y != 0);

  dim3 dimBlock(matcov_thread_x, matcov_thread_y);
  dim3 dimGrid(nbx, nby);
  const long Nsubap = sensors->d_wfs[0]->nvalid;

  /*
    int nb = (int)(1224);
    double *tmp;
    tmp=(double*)malloc((nb)*sizeof(double));
    carmaSafeCall(hipMemcpy(tmp, tomo_gpu->u_d, sizeof(double) * nb,
                    hipMemcpyDeviceToHost));
    for (int ii = 0 ; ii < nb ; ii++){
          printf("%5.20f \n",tmp[ii]);
    }
    printf("convert : %9.9f\n",convert);
  */

  CPHIM_kernel<<<dimGrid, dimBlock, 0, cphim_struct->cphim_stream>>>(
      data, nrows, ncols, xoffset, yoffset, lda, convert,
      cphim_struct->sspSizeL_d, cphim_struct->Nssp_d, cphim_struct->u_d,
      cphim_struct->v_d, cphim_struct->xact_d, cphim_struct->yact_d,
      cphim_struct->x0, cphim_struct->y0, cphim_struct->int_npts,
      cphim_struct->L0diff_d, cphim_struct->cn2_d, Ndphi, cphim_struct->Nw,
      cphim_struct->Ndm, atmos->nscreens, cphim_struct->Nsubap_d,
      cphim_struct->Nx, cphim_struct->Nactu_tot_d, cphim_struct->Nactu,
      cphim_struct->NlayerDM_d, cphim_struct->indLayerDm_d, cphim_struct->dx_d,
      cphim_struct->alphaX_d, cphim_struct->alphaY_d, cphim_struct->lgs_cst,
      (double)0.0, cphim_struct->spot_width, cphim_struct->lgs_depth,
      cphim_struct->lgs_alt, cphim_struct->h_d, cphim_struct->hDm_d,
      cphim_struct->FoV, cphim_struct->nlgs, cphim_struct->DiamTel,
      cphim_struct->k2_d, cphim_struct->tab_int_x, cphim_struct->tab_int_y);

  carmaCheckMsg("matcov_kernel_4<<<>>> execution failed\n");
  hipStreamSynchronize(cphim_struct->cphim_stream);
}

__device__ double unMoinsJ0(double x) {
  if (x < 0.1) {
    double x22 = (x * x) / 4.;
    return (1.0 - x22 / 4.) * x22;
  } else
    return (double)(1.0 - j0((double)x));
}

__global__ void compute_u831J0(double *x, double *y, int npts, double tmin,
                               double tmax, double dt) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  double t;
  while (tid < npts) {
    t = tmin + tid * dt;
    x[tid] = exp(t);
    y[tid] = exp(-t * (5 / 3.)) * unMoinsJ0(exp(t)) * dt;
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void cuda_zcen_krnl(double *idata, double *odata, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    odata[tid] = (idata[tid + 1] + idata[tid]) / 2.0;
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void intfrominftomin(double *data, double smallInt, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid < N) {
    data[tid] += smallInt;
    tid += blockDim.x * gridDim.x;
  }
}

void sub_pos_cphim(struct cphim_struct *cphim_struct, long Nlayer)
// void subap_position_gpu_gb(struct tomo_struct tomo, double ***u, double ***v)
/* DOCUMENT DOCUMENT         subap_position(tomo, u, v)
   <tomo>                : structure with all the needed information.
   <u> and <v>           : 3d arrays containing the sub-apertures projected
   coordinates onto all the layers. u[0][2][1] is the X-coordinate of the subap
   2 of the WFS 0 on the layer 1.

   Computes the projected coordinates of all subapertures  projected onto all
   the layer
 */
{
  int msize = Nlayer * cphim_struct->Nx;
  int nblocks = msize / tabDPHI_thread_x + ((msize % tabDPHI_thread_x) != 0);
  dim3 dimBlock(tabDPHI_thread_x, 1);
  dim3 dimGrid(nblocks, 1);
  /*
    int nb = (int)(msize);
        double *tmpp;
        tmpp=(double*)malloc((nb)*sizeof(double));
        carmaSafeCall(hipMemcpy(tmpp, cphim_struct->X_d, sizeof(double) * nb,
                            hipMemcpyDeviceToHost));
        for (int ii = 0 ; ii < nb ; ii++){
          printf("%5.5f \n",tmpp[ii]);
        }
        */
  // std::cout << "Nsubap : " << Nsubap << std::endl;

  subposition_gpu_gb_kernel<<<dimGrid, dimBlock, 0,
                              cphim_struct->cphim_stream>>>(
      cphim_struct->Nw, cphim_struct->Nx, cphim_struct->Nsubap_d, Nlayer,
      cphim_struct->alphaX_d, cphim_struct->alphaY_d, cphim_struct->h_d,
      cphim_struct->GsAlt_d, cphim_struct->Nssp_d, cphim_struct->diamPup_d,
      cphim_struct->thetaML_d, cphim_struct->ioff_d, cphim_struct->X_d,
      cphim_struct->Y_d, cphim_struct->XPup_d, cphim_struct->YPup_d,
      cphim_struct->u_d, cphim_struct->v_d);

  carmaCheckMsg("subposition_gpu_gb_kernel<<<>>> execution failed\n");
  /*
     int nb = (int)(184);
        double *tmpp;
        tmpp=(double*)malloc((nb)*sizeof(double));
        carmaSafeCall(hipMemcpy(tmpp, tomo_gpu->u_d, sizeof(double) * nb,
                            hipMemcpyDeviceToHost));
        for (int ii = 0 ; ii < nb ; ii++){
          printf("%5.5f \n",tmpp[ii]);
        }
  */
  // CudaCheckError();
}

void init_cphim_struct(struct cphim_struct *cphim_struct, sutra_atmos *atmos,
                       sutra_sensors *sensors, sutra_dms *dms, double diamTel) {
  hipError_t e;

  cphim_struct->DiamTel = diamTel;
  cphim_struct->Nw = sensors->nsensors();

  int Nactu = 0;
  int Ndm = 0;
  vector<sutra_dm *>::iterator p;
  p = dms->d_dms.begin();
  while (p != dms->d_dms.end()) {
    sutra_dm *dm = *p;
    if (dm->type != "tt") {
      Nactu += dm->ninflu;
      Ndm += 1;
    }
    p++;
  }

  cphim_struct->Nactu = Nactu;
  cphim_struct->Ndm = Ndm;
  cphim_struct->Nlayer = atmos->nscreens;
  cphim_struct->int_npts = 10000;
  cphim_struct->pasDu = 0.0001;
  cphim_struct->Nactu_tot = (long *)malloc(cphim_struct->Ndm * sizeof(long));
  cphim_struct->NlayerDM = (long *)malloc(cphim_struct->Ndm * sizeof(long));
  cphim_struct->indLayerDm = (long *)malloc(atmos->nscreens * sizeof(long));
  p = dms->d_dms.begin();
  int indx = 0;
  while (p != dms->d_dms.end()) {
    sutra_dm *dm = *p;
    if (dm->type != "tt") {
      cphim_struct->Nactu_tot[indx] = dm->ninflu;
      indx += 1;
    }
    p++;
  }

  cphim_struct->lgs_cst = 0.;
  cphim_struct->spot_width = 1.;
  cphim_struct->lgs_depth = 10000.;
  cphim_struct->lgs_alt = 90000.;
  cphim_struct->nlgs = 0;
  cphim_struct->pasDPHI = 0.0001;

  cphim_struct->Nx = 0;
  cphim_struct->Nssp = (long *)malloc(cphim_struct->Nw * sizeof(long));
  cphim_struct->Nsubap = (long *)malloc(cphim_struct->Nw * sizeof(long));
  cphim_struct->diamPup = (double *)malloc(cphim_struct->Nw * sizeof(double));
  cphim_struct->XPup = (double *)malloc(cphim_struct->Nw * sizeof(double));
  cphim_struct->YPup = (double *)malloc(cphim_struct->Nw * sizeof(double));
  cphim_struct->thetaML = (double *)malloc(cphim_struct->Nw * sizeof(double));
  cphim_struct->GsAlt = (double *)malloc(cphim_struct->Nw * sizeof(double));

  for (int i = 0; i < cphim_struct->Nw; i++) {
    cphim_struct->Nssp[i] = sensors->d_wfs[i]->nxsub;
    cphim_struct->Nsubap[i] = sensors->d_wfs[i]->nvalid;
    cphim_struct->diamPup[i] = (double)cphim_struct->Nssp[i];
    cphim_struct->XPup[i] = 0.;
    cphim_struct->YPup[i] = 0.;
    cphim_struct->thetaML[i] = 0.;
    if (sensors->d_wfs[i]->d_gs->lgs) {
      cphim_struct->nlgs += 1;
      cphim_struct->GsAlt[i] = 1.0 / cphim_struct->lgs_alt;
    } else
      cphim_struct->GsAlt[i] = 0.0;
    cphim_struct->GsAlt[i] = 0.;
    cphim_struct->Nx += sensors->d_wfs[i]->nvalid;
  }

  e = hipMalloc((void **)&(cphim_struct->Nactu_tot_d),
                 cphim_struct->Ndm * sizeof(long));
  process_err(e, "alloc gpu Nactu_tot_d");

  e = hipMalloc((void **)&(cphim_struct->NlayerDM_d),
                 cphim_struct->Ndm * sizeof(long));
  process_err(e, "alloc gpu NlayerDM_d");

  e = hipMalloc((void **)&(cphim_struct->indLayerDm_d),
                 atmos->nscreens * sizeof(long));
  process_err(e, "alloc gpu Nactu_tot_d");

  e = hipMalloc((void **)&(cphim_struct->indexL0_d),
                 atmos->nscreens * sizeof(long));
  process_err(e, "alloc gpu indexL0_d");

  e = hipMalloc((void **)&(cphim_struct->u_d),
                 atmos->nscreens * cphim_struct->Nx * sizeof(double));
  process_err(e, "alloc gpu u_d");
  // printf("size of u is %d\n",atmos->nscreens * sensors->d_wfs[0]->nvalid *
  // cphim_struct->Nw); printf("u_d = 0x%x \n", (cphim_struct->u_d) );

  e = hipMalloc((void **)&(cphim_struct->v_d),
                 atmos->nscreens * cphim_struct->Nx * sizeof(double));
  process_err(e, "alloc gpu v_d");
  // printf("size of v is %d\n", tomo.Nlayer*tomo.Nsubap[0]*tomo.Nw);
  // printf("v_d = 0x%x \n", (cphim_struct->v_d) );

  e = hipMalloc((void **)&(cphim_struct->sspSizeL_d),
                 cphim_struct->Nw * atmos->nscreens * sizeof(double));
  process_err(e, "alloc gpu sspSizeL_d");

  e = hipMalloc((void **)&(cphim_struct->cn2_d),
                 atmos->nscreens * sizeof(double));
  process_err(e, "alloc gpu cn2_d");

  e = hipMalloc((void **)&(cphim_struct->h_d),
                 atmos->nscreens * sizeof(double));
  process_err(e, "alloc gpu h_d");

  e = hipMalloc((void **)&(cphim_struct->hDm_d),
                 cphim_struct->Ndm * sizeof(double));
  process_err(e, "alloc gpu h_d");

  e = hipMalloc((void **)&(cphim_struct->Nssp_d),
                 cphim_struct->Nw * sizeof(long));
  process_err(e, "alloc gpu Nssp_d");

  e = hipMalloc((void **)&(cphim_struct->Nsubap_d),
                 cphim_struct->Nw * sizeof(long));
  process_err(e, "alloc gpu Nsubap_d");

  e = hipMalloc((void **)&(cphim_struct->ioff_d),
                 cphim_struct->Nw * sizeof(long));
  process_err(e, "alloc gpu ioff_d");

  e = hipMalloc((void **)&(cphim_struct->alphaX_d),
                 cphim_struct->Nw * sizeof(double));
  process_err(e, "alloc gpu alphaX_d");

  e = hipMalloc((void **)&(cphim_struct->alphaY_d),
                 cphim_struct->Nw * sizeof(double));
  process_err(e, "alloc gpu alphaY_d");

  e = hipMalloc((void **)&(cphim_struct->GsAlt_d),
                 cphim_struct->Nw * sizeof(double));
  process_err(e, "alloc gpu GsAlt_d");

  e = hipMalloc((void **)&(cphim_struct->diamPup_d),
                 cphim_struct->Nw * sizeof(double));
  process_err(e, "alloc gpu diamPup_d");

  e = hipMalloc((void **)&(cphim_struct->thetaML_d),
                 cphim_struct->Nw * sizeof(double));
  process_err(e, "alloc gpu thetaML_d");

  e = hipMalloc((void **)&(cphim_struct->k2_d),
                 cphim_struct->Ndm * sizeof(double));
  process_err(e, "alloc gpu k2_d");

  e = hipMalloc((void **)&(cphim_struct->X_d),
                 cphim_struct->Nx * sizeof(double));
  process_err(e, "alloc gpu X_d");
  // printf("size of X is %d\n", cphim_struct->Nx);
  // printf("X_d = 0x%x \n", (cphim_struct->X_d) );

  e = hipMalloc((void **)&(cphim_struct->Y_d),
                 cphim_struct->Nx * sizeof(double));
  process_err(e, "alloc gpu Y_d");
  // printf("size of X is %d\n", tomo.Nx);
  // printf("Y_d = 0x%x \n", (cphim_struct->Y_d) );
  e = hipMalloc((void **)&(cphim_struct->xact_d),
                 cphim_struct->Nactu * sizeof(double));
  process_err(e, "alloc gpu X_d");
  // printf("size of X is %d\n", cphim_struct->Nx);
  // printf("X_d = 0x%x \n", (cphim_struct->X_d) );

  e = hipMalloc((void **)&(cphim_struct->yact_d),
                 cphim_struct->Nactu * sizeof(double));
  process_err(e, "alloc gpu Y_d");

  e = hipMalloc((void **)&(cphim_struct->dx_d),
                 cphim_struct->Ndm * sizeof(double));
  process_err(e, "alloc gpu Y_d");

  e = hipMalloc((void **)&(cphim_struct->XPup_d),
                 cphim_struct->Nw * sizeof(double));
  process_err(e, "alloc gpu XPup_d");

  e = hipMalloc((void **)&(cphim_struct->YPup_d),
                 cphim_struct->Nw * sizeof(double));
  process_err(e, "alloc gpu YPup_d");

  cphim_struct->L0diff_d = NULL;
  cphim_struct->tabDPHI_d = NULL;

  e = hipMalloc((void **)&(cphim_struct->tab_int_x),
                 cphim_struct->int_npts * sizeof(double));
  process_err(e, "alloc gpu tab_int_x");

  e = hipMalloc((void **)&(cphim_struct->tab_int_y),
                 cphim_struct->int_npts * sizeof(double));
  process_err(e, "alloc gpu tab_int_y");

  e = hipStreamCreate(&(cphim_struct->cphim_stream));
  process_err(e, "create cphim stream");
}

void tab_u831J0(double *tab_int_x, double *tab_int_y, long npts) {
  // DEBUG_TRACE("tab_int !\n");

  double tmin = -4.;
  double tmax = 10.;
  hipError_t e;
  double *t = (double *)malloc(sizeof(double) * npts);
  e = hipMemcpy(t, tab_int_x, (npts) * sizeof(double), hipMemcpyDeviceToHost);
  process_err(e, "copy test");

  double *temp;
  temp = (double *)malloc((npts - 1) * sizeof(double));
  double *tab;
  tab = (double *)malloc((npts) * sizeof(double));
  double *temp_d;
  // int nblocks = 0 , nthreads = 0;
  double dt = (tmax - tmin) / (npts - 1);
  // getNumBlocksAndThreads(device, npts, nblocks, nthreads);
  /*
  int nblocks = msize / tabDPHI_thread_x + ( ( msize % tabDPHI_thread_x) != 0);
  dim3 dimBlock(tabDPHI_thread_x, 1);
  dim3 dimGrid(nblocks, 1);
  */
  int device;
  hipGetDevice(&device);
  struct hipDeviceProp_t props;
  hipGetDeviceProperties(&props, device);
  int nthreads = props.maxThreadsPerBlock;
  int nblocks = npts / nthreads + ((npts % nthreads) != 0);
  dim3 grid(nblocks), threads(nthreads);
  compute_u831J0<<<grid, threads>>>(tab_int_x, tab_int_y, npts, tmin, tmax, dt);
  carmaCheckMsg("compute_u831J0<<<>>> execution failed\n");
  // DEBUG_TRACE("tab_int !\n");
  e = hipMalloc((void **)&(temp_d), (npts - 1) * sizeof(double));
  process_err(e, "alloc gpu temp_d");
  nblocks = (npts - 1) / nthreads + (((npts - 1) % nthreads) != 0);
  dim3 grid2(nblocks);
  cuda_zcen_krnl<<<grid2, threads>>>(tab_int_y, temp_d, npts - 1);
  carmaCheckMsg("cuda_zcen_krnl<<<>>> execution failed\n");
  // cuda_zcen(tab_int_y,temp_d, npts-1, device);
  // DEBUG_TRACE("tab_int !\n");
  e = hipMemcpy(temp, temp_d, (npts - 1) * sizeof(double),
                 hipMemcpyDeviceToHost);
  process_err(e, "copy cpu temp");
  hipFree(temp_d);
  cumsum(tab, temp, npts);
  // DEBUG_TRACE("tab_int !\n");
  e = hipMemcpy(tab_int_y, tab, (npts) * sizeof(double),
                 hipMemcpyHostToDevice);
  process_err(e, "copy gpu tab");
  // DEBUG_TRACE("tab_int !\n");
  double smallx = exp(tmin);
  double smallInt = (0.75 * pow(smallx, 1 / 3.) * (1 - smallx * smallx / 112.));
  // DEBUG_TRACE("tab_int !\n");
  intfrominftomin<<<grid, threads>>>(tab_int_y, smallInt, npts);
  carmaCheckMsg("intfrominftomin<<<>>> execution failed\n");
}

void cumsum(double *odata, double *idata, int N) {
  odata[0] = 0;
  for (int i = 1; i < N; i++) {
    odata[i] = idata[i - 1] + odata[i - 1];
  }
}

void cuda_zcen(double *idata, double *odata, int N, carma_device *device) {
  int nblocks = 0, nthreads = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  cuda_zcen_krnl<<<grid, threads>>>(idata, odata, N);
  carmaCheckMsg("cuda_zcen_krnl<<<>>> execution failed\n");
}

void update_cphim_atm(struct cphim_struct *cphim_struct, sutra_sensors *sensors,
                      sutra_atmos *atmos, double *L0, double *cn2,
                      double *alphaX, double *alphaY) {
  hipError_t e;

  double h[atmos->nscreens];
  int ii = 0;
  for (map<float, sutra_tscreen *>::iterator it = atmos->d_screens.begin();
       it != atmos->d_screens.end(); ++it) {
    h[ii] = (double)it->second->altitude;
    ii++;
  }
  // DEBUG_TRACE("Here !\n");
  double dmax = 0.0;
  double maxalt = h[atmos->nscreens - 1];
  long minssp = cphim_struct->Nssp[0];
  for (int cc = 0; cc < cphim_struct->Nw; cc++) {
    double tmp = sqrtf(alphaX[cc] * alphaX[cc] + alphaY[cc] * alphaY[cc]);
    if (tmp > dmax) dmax = tmp;
    if (minssp > cphim_struct->Nssp[cc]) minssp = cphim_struct->Nssp[cc];
  }
  const double crmax =
      dmax * 2. * maxalt + (1 + 1. / minssp) * cphim_struct->DiamTel;
  const double pasDPHI = 1. / cphim_struct->pasDPHI;  // inverse du pas de rr
  const long Ndphi = floor(crmax * pasDPHI) + 1;
  cphim_struct->Ndphi = Ndphi;
  // const double convert = (double)(Ndphi-1)/(crmax+1./pasDPHI);
  // const double convert_int = (double)(cphim_struct->int_npts
  // -1)/(expf(10.0f)+expf(-(14./cphim_struct->int_npts))); const double
  // convert_int = 14./(cphim_struct->int_npts-1);
  // DEBUG_TRACE("Here !\n");
  e = hipMemcpyAsync(cphim_struct->h_d, h, atmos->nscreens * sizeof(double),
                      hipMemcpyHostToDevice, cphim_struct->cphim_stream);
  process_err(e, "copy gpu h_d");

  e = hipMemcpyAsync(cphim_struct->cn2_d, cn2,
                      atmos->nscreens * sizeof(double), hipMemcpyHostToDevice,
                      cphim_struct->cphim_stream);
  process_err(e, "copy gpu cn2_d");
  // DEBUG_TRACE("Here !\n");
  double *sspSizeL =
      (double *)malloc(sizeof(double) * cphim_struct->Nw * atmos->nscreens);
  for (int cc = 0; cc < cphim_struct->Nw * atmos->nscreens; cc++) {
    int n = cc / atmos->nscreens;
    int l = cc - n * atmos->nscreens;
    if (n >= sensors->nsensors()) n -= 1;
    sspSizeL[cc] =
        (((double)(cphim_struct->DiamTel / sensors->d_wfs[n]->nxsub)) *
         (1. -
          cphim_struct->GsAlt[n] *
              h[l]));  //+ 2*sqrt(alphaX[n]*alphaX[n]+alphaY[n]*alphaY[n])*h[l];
  }
  // DEBUG_TRACE("Here !\n");
  e = hipMemcpyAsync(cphim_struct->sspSizeL_d, sspSizeL,
                      cphim_struct->Nw * atmos->nscreens * sizeof(double),
                      hipMemcpyHostToDevice, cphim_struct->cphim_stream);
  process_err(e, "copy gpu sspSizeL_d");
  hipStreamSynchronize(cphim_struct->cphim_stream);
  // Search the different L0 and build indexL0
  const long Nlayer = atmos->nscreens;
  long i, j;
  int cpt = 1;
  double tmp[Nlayer];
  long indexL0[Nlayer];
  tmp[0] = L0[0];
  indexL0[0] = 0;

  for (i = 1; i < Nlayer; i++) {
    j = 0;
    const double l0 = L0[i];

    while ((j < cpt) && (tmp[j] != l0)) {
      j++;
    }

    indexL0[i] = j;

    if (j == cpt) {
      tmp[j] = l0;
      cpt++;
    }
  }
  e = hipMemcpyAsync((cphim_struct->indexL0_d), indexL0,
                      atmos->nscreens * sizeof(long), hipMemcpyHostToDevice,
                      cphim_struct->cphim_stream);
  process_err(e, "copy gpu indexL0_d");
  int Nl0 = cpt;
  /*
  double L0diff[Nl0];
  // DEBUG_TRACE("Here !\n");
  // allocate space for L0
  process_err(e, "alloc gpu L0diff_d");
  for (i = 0; i < Nl0; i++)  {
    L0diff[i] = tmp[i];
  }
  */
  if ((cphim_struct->L0diff_d) != NULL) {
    hipFree(cphim_struct->L0diff_d);
  }
  e = hipMalloc((void **)&(cphim_struct->L0diff_d),
                 cphim_struct->Nlayer * sizeof(double));
  // offload L0diff
  e = hipMemcpyAsync(cphim_struct->L0diff_d, L0,
                      cphim_struct->Nlayer * sizeof(double),
                      hipMemcpyHostToDevice, cphim_struct->cphim_stream);
  process_err(e, "offload L0diff");
  // précalcul de DPHI : que pour chaque différent L0
  if ((cphim_struct->tabDPHI_d) != NULL) {
    hipFree(cphim_struct->tabDPHI_d);
  }
  // printf("tabDPHI alloc \n");
  e = hipMalloc((void **)&(cphim_struct->tabDPHI_d),
                 Nl0 * Ndphi * cphim_struct->Ndm * sizeof(double));
  process_err(e, "alloc gpu tabDPHI_d");

  // DEBUG_TRACE("%5.5d %5.5d %5.5f\n",Ndphi,Nl0,convert);
  /*
  int nb = (int)(3);
  // FILE *f = fopen("tabDPHI_d.txt","w");
    double *tmpp;
    tmpp=(double*)malloc((nb)*sizeof(double));
    carmaSafeCall(hipMemcpy(tmpp, cphim_struct->h_d, sizeof(double) * nb,
                    hipMemcpyDeviceToHost));
    for (int ii = 0 ; ii < nb ; ii++){
        DEBUG_TRACE("%5.5f \n",tmpp[ii]);
    }
    */

  // tab_dphi_lowpass(cphim_struct->tabDPHI_d, cphim_struct, Ndphi,
  // cphim_struct->L0diff_d, Nl0,convert,convert_int);
  // carmaSafeCall(hipDeviceSynchronize());
  /*
  int nb = (int)(Ndphi);
   // FILE *f = fopen("tabDPHI_d.txt","w");
      double *tmpp;
      tmpp=(double*)malloc((nb)*sizeof(double));
      carmaSafeCall(hipMemcpy(tmpp, cphim_struct->tabDPHI_d, sizeof(double) *
  nb, hipMemcpyDeviceToHost)); for (int ii = 0 ; ii < nb ; ii++){ printf("%5.5f
  \n",tmpp[ii]);
      }
      */
  // %%%%%%% Computation of the sub-apertures positions and sizes %%%%%%%%%%%
  // u, v :arrays containing all the sub-apertures coordinates of all WFS, one
  // after the other u[0][1][3] is the X-coordinate of subap number 3 of wfs
  // number 0 at altitude 3

  // Computes  u and v
  // DEBUG_TRACE("Here %d %d %d!\n", (long)atmos->nscreens,
  // (long)cphim_struct->Nw, (long)sensors->d_wfs[0]->nvalid);
  sub_pos_cphim(cphim_struct, (long)atmos->nscreens);
  // carmaSafeCall(hipDeviceSynchronize());
  /*
    int nb = (int)(atmos->nscreens * sensors->d_wfs[0]->nvalid *
    cphim_struct->Nw); double *tmpp; tmpp=(double*)malloc((nb)*sizeof(double));
      carmaSafeCall(hipMemcpy(tmpp, cphim_struct->u_d, sizeof(double) * nb,
                    hipMemcpyDeviceToHost));
      for (int ii = 0 ; ii < nb ; ii++){
          printf("%5.5f \n",tmpp[ii]);
      }
  */
  if (sspSizeL) free(sspSizeL);
  // DEBUG_TRACE("Here !\n");
}

void update_cphim_sys(struct cphim_struct *cphim_struct, sutra_sensors *sensors,
                      double *alphaX, double *alphaY, double *xactu,
                      double *yactu, double *X, double *Y, long *NlayerDm,
                      long *indLayerDm, double *alt_dm, double *pitch,
                      double *k2, double FoV) {
  hipError_t e;

  long ioff[cphim_struct->Nw];
  ioff[0] = 0;
  for (int i = 1; i < cphim_struct->Nw; i++) {
    ioff[i] = ioff[i - 1] + sensors->d_wfs[i - 1]->nvalid;
  }

  cphim_struct->FoV = FoV;

  e = hipMemcpyAsync(cphim_struct->Nactu_tot_d, cphim_struct->Nactu_tot,
                      cphim_struct->Ndm * sizeof(long), hipMemcpyHostToDevice,
                      cphim_struct->cphim_stream);
  process_err(e, "copy gpu Nactu_tot_d");

  e = hipMemcpyAsync(cphim_struct->hDm_d, alt_dm,
                      cphim_struct->Ndm * sizeof(double),
                      hipMemcpyHostToDevice, cphim_struct->cphim_stream);
  process_err(e, "copy gpu hDm_d");

  e = hipMemcpyAsync(cphim_struct->NlayerDM_d, NlayerDm,
                      cphim_struct->Ndm * sizeof(long), hipMemcpyHostToDevice,
                      cphim_struct->cphim_stream);
  process_err(e, "copy gpu NlayerDM_d");

  e = hipMemcpyAsync(cphim_struct->indLayerDm_d, indLayerDm,
                      cphim_struct->Nlayer * sizeof(long),
                      hipMemcpyHostToDevice, cphim_struct->cphim_stream);
  process_err(e, "copy gpu indLayerDm_d");

  e = hipMemcpyAsync(cphim_struct->k2_d, k2,
                      cphim_struct->Ndm * sizeof(double),
                      hipMemcpyHostToDevice, cphim_struct->cphim_stream);
  process_err(e, "copy gpu k2_d");

  e = hipMemcpyAsync(cphim_struct->ioff_d, ioff,
                      cphim_struct->Nw * sizeof(long), hipMemcpyHostToDevice,
                      cphim_struct->cphim_stream);
  process_err(e, "copy gpu ioff_d");

  e = hipMemcpyAsync(cphim_struct->alphaX_d, alphaX,
                      cphim_struct->Nw * sizeof(double), hipMemcpyHostToDevice,
                      cphim_struct->cphim_stream);
  process_err(e, "copy gpu alphaX_d");

  e = hipMemcpyAsync(cphim_struct->alphaY_d, alphaY,
                      cphim_struct->Nw * sizeof(double), hipMemcpyHostToDevice,
                      cphim_struct->cphim_stream);
  process_err(e, "copy gpu alphaY_d");

  e = hipMemcpyAsync(cphim_struct->GsAlt_d, cphim_struct->GsAlt,
                      cphim_struct->Nw * sizeof(double), hipMemcpyHostToDevice,
                      cphim_struct->cphim_stream);
  process_err(e, "copy gpu GsAlt_d");

  e = hipMemcpyAsync(cphim_struct->Nssp_d, cphim_struct->Nssp,
                      cphim_struct->Nw * sizeof(long), hipMemcpyHostToDevice,
                      cphim_struct->cphim_stream);
  process_err(e, "copy gpu Nssp_d");

  e = hipMemcpyAsync(cphim_struct->Nsubap_d, cphim_struct->Nsubap,
                      cphim_struct->Nw * sizeof(long), hipMemcpyHostToDevice,
                      cphim_struct->cphim_stream);
  process_err(e, "copy gpu Nsubap_d");

  e = hipMemcpyAsync(cphim_struct->diamPup_d, cphim_struct->diamPup,
                      cphim_struct->Nw * sizeof(double), hipMemcpyHostToDevice,
                      cphim_struct->cphim_stream);
  process_err(e, "copy gpu diamPup_d");

  e = hipMemcpyAsync(cphim_struct->XPup_d, cphim_struct->XPup,
                      cphim_struct->Nw * sizeof(double), hipMemcpyHostToDevice,
                      cphim_struct->cphim_stream);
  process_err(e, "copy gpu XPup_d");

  e = hipMemcpyAsync(cphim_struct->YPup_d, cphim_struct->YPup,
                      cphim_struct->Nw * sizeof(double), hipMemcpyHostToDevice,
                      cphim_struct->cphim_stream);
  process_err(e, "copy gpu YPup_d");

  e = hipMemcpyAsync(cphim_struct->thetaML_d, cphim_struct->thetaML,
                      cphim_struct->Nw * sizeof(double), hipMemcpyHostToDevice,
                      cphim_struct->cphim_stream);
  process_err(e, "copy gpu thetaML_d");
  // DEBUG_TRACE("Update \n");
  /*
    double *X;
    double *Y;
    int *tmpX;
    int *tmpY;
    X=(double*)malloc((cphim_struct->Nx)*sizeof(double));
    Y=(double*)malloc((cphim_struct->Nx)*sizeof(double));
    tmpX=(int*)malloc((sensors->d_wfs[0]->nvalid)*sizeof(int));
    tmpY=(int*)malloc((sensors->d_wfs[0]->nvalid)*sizeof(int));
    int ind = 0;
    double p2m;
    for(int i=0 ; i<cphim_struct->Nw ; i++){
          if(i<sensors->nsensors()){
                  e =
    hipMemcpyAsync(tmpX,sensors->d_wfs[i]->d_validsubsx->getData() ,
    sizeof(int) * sensors->d_wfs[i]->nvalid,
                    hipMemcpyDeviceToHost,cphim_struct->cphim_stream);
                  process_err(e,"copy tmpX");
                  e =
    hipMemcpyAsync(tmpY,sensors->d_wfs[i]->d_validsubsy->getData() ,
    sizeof(int) * sensors->d_wfs[i]->nvalid,
                            hipMemcpyDeviceToHost,cphim_struct->cphim_stream);
                  process_err(e,"copy tmpY");
                  p2m =
    (cphim_struct->DiamTel/(double)sensors->d_wfs[i]->nxsub)/((double)(tmpX[1]-tmpX[0]));
          }
          else{
                  e =
    hipMemcpyAsync(tmpX,sensors->d_wfs[i-1]->d_validsubsx->getData() ,
    sizeof(int) * sensors->d_wfs[i-1]->nvalid,
                                    hipMemcpyDeviceToHost,cphim_struct->cphim_stream);
                  process_err(e,"copy tmpX");
                  e =
    hipMemcpyAsync(tmpY,sensors->d_wfs[i-1]->d_validsubsy->getData() ,
    sizeof(int) * sensors->d_wfs[i-1]->nvalid,
                                            hipMemcpyDeviceToHost,cphim_struct->cphim_stream);
                  process_err(e,"copy tmpY");
                  p2m =
    (cphim_struct->DiamTel/(double)sensors->d_wfs[i-1]->nxsub)/((double)(tmpX[1]-tmpX[0]));
          }

          for(int j=0 ; j<sensors->d_wfs[0]->nvalid ; j++){
                  if(i<sensors->nsensors()){
                          X[ind + j] = ((double)tmpX[j] * p2m)-
    (double)((cphim_struct->DiamTel/2.)*(1.-1./(double)sensors->d_wfs[i]->nxsub));
                          Y[ind + j] = ((double)tmpY[j] * p2m) -
    (double)((cphim_struct->DiamTel/2.)*(1.-1./(double)sensors->d_wfs[i]->nxsub));
                  }
                  else{
                          X[ind + j] = ((double)tmpX[j] * p2m)-
    (double)((cphim_struct->DiamTel/2.)*(1.-1./(double)sensors->d_wfs[i-1]->nxsub));
                          Y[ind + j] = ((double)tmpY[j] * p2m) -
    (double)((cphim_struct->DiamTel/2.)*(1.-1./(double)sensors->d_wfs[i-1]->nxsub));
                  }
          }
          if(i<sensors->nsensors())
          ind += sensors->d_wfs[i]->nvalid;
          else ind += sensors->d_wfs[i-1]->nvalid;
    }
    /*
    for (int ii = 0; ii<cphim_struct->Nx ; ii++){
          std::cout << "X : " << X[ii] << std::endl;
    }
    for (int jj = 0; jj<cphim_struct->Nx ; jj++){
          std::cout << "Y : " << Y[jj] << std::endl;
    }
    */
  // generateXY(cphim_struct,sensors);

  e = hipMemcpyAsync(cphim_struct->X_d, X, cphim_struct->Nx * sizeof(double),
                      hipMemcpyHostToDevice, cphim_struct->cphim_stream);
  process_err(e, "copy gpu X_d");
  e = hipMemcpyAsync(cphim_struct->Y_d, Y, cphim_struct->Nx * sizeof(double),
                      hipMemcpyHostToDevice, cphim_struct->cphim_stream);
  process_err(e, "copy gpu Y_d");

  e = hipMemcpyAsync(cphim_struct->xact_d, xactu,
                      cphim_struct->Nactu * sizeof(double),
                      hipMemcpyHostToDevice, cphim_struct->cphim_stream);
  process_err(e, "copy gpu xact_d");
  e = hipMemcpyAsync(cphim_struct->yact_d, yactu,
                      cphim_struct->Nactu * sizeof(double),
                      hipMemcpyHostToDevice, cphim_struct->cphim_stream);
  process_err(e, "copy gpu yact_d");
  e = hipMemcpyAsync(cphim_struct->dx_d, pitch,
                      cphim_struct->Ndm * sizeof(double),
                      hipMemcpyHostToDevice, cphim_struct->cphim_stream);
  process_err(e, "copy gpu dx_d");

  std::cout << "   computing tabulated integral...";
  tab_u831J0(cphim_struct->tab_int_x, cphim_struct->tab_int_y,
             cphim_struct->int_npts);
  std::cout << " done" << std::endl;
  cphim_struct->x0 = xactu[cphim_struct->Nactu / 2 + 1];
  cphim_struct->y0 = yactu[cphim_struct->Nactu / 2 + 1];
  // cphim_struct->dx = (xactu[1] - xactu[0]) * 0.5;
  // hipStreamSynchronize(cphim_struct->cphim_stream);
  // DEBUG_TRACE("Update \n");
  hipStreamSynchronize(cphim_struct->cphim_stream);
  /*
    int nb = (int)(cphim_struct->Nactu);
    double *tmp;
    tmp=(double*)malloc((nb)*sizeof(double));
    carmaSafeCall(hipMemcpy(tmp, cphim_struct->yact_d, sizeof(double) * nb,
                    hipMemcpyDeviceToHost));
    for (int ii = 0 ; ii < nb ; ii++){
          printf("%5.5f \n",tmp[ii]);
    }
  */
}

void free_cphim_struct(struct cphim_struct *cphim_struct) {
  hipError_t e;

  if ((cphim_struct->dx_d)) e = hipFree(cphim_struct->dx_d);
  process_err(e, "free gpu dx_d");

  if ((cphim_struct->hDm_d)) e = hipFree(cphim_struct->hDm_d);
  process_err(e, "free gpu hDm_d");

  if ((cphim_struct->NlayerDM_d)) e = hipFree(cphim_struct->NlayerDM_d);
  process_err(e, "free gpu NlayerDM_d");

  if ((cphim_struct->indLayerDm_d)) e = hipFree(cphim_struct->indLayerDm_d);
  process_err(e, "free gpu indLayerDm_d");

  if ((cphim_struct->Nactu_tot_d)) e = hipFree(cphim_struct->Nactu_tot_d);
  process_err(e, "free gpu Nactu_tot_d");

  if ((cphim_struct->u_d)) e = hipFree(cphim_struct->u_d);
  process_err(e, "free gpu u_d");

  if (cphim_struct->v_d) e = hipFree(cphim_struct->v_d);
  process_err(e, "free gpu v_d");

  if (cphim_struct->sspSizeL_d) e = hipFree(cphim_struct->sspSizeL_d);
  process_err(e, "free gpu sspSizeL_d");

  if (cphim_struct->cn2_d) e = hipFree(cphim_struct->cn2_d);
  process_err(e, "free gpu cn2_d");

  if (cphim_struct->h_d) e = hipFree(cphim_struct->h_d);
  process_err(e, "free gpu h_d");

  if (cphim_struct->indexL0_d) e = hipFree(cphim_struct->indexL0_d);
  process_err(e, "free gpu indexL0_d");

  if (cphim_struct->Nssp_d) e = hipFree(cphim_struct->Nssp_d);
  process_err(e, "free gpu Nssp_d");

  if (cphim_struct->Nsubap_d) e = hipFree(cphim_struct->Nsubap_d);
  process_err(e, "free gpu Nsubap_d");

  if (cphim_struct->ioff_d) e = hipFree(cphim_struct->ioff_d);
  process_err(e, "free gpu ioff_d");

  if (cphim_struct->alphaX_d) e = hipFree(cphim_struct->alphaX_d);
  process_err(e, "free gpu alphaX_d");

  if (cphim_struct->alphaY_d) e = hipFree(cphim_struct->alphaY_d);
  process_err(e, "free gpu alphaY_d");

  if (cphim_struct->GsAlt_d) e = hipFree(cphim_struct->GsAlt_d);
  process_err(e, "free gpu GsAlt_d");

  if (cphim_struct->diamPup_d) e = hipFree(cphim_struct->diamPup_d);
  process_err(e, "free gpu diamPup_d");

  if (cphim_struct->thetaML_d) e = hipFree(cphim_struct->thetaML_d);
  process_err(e, "free gpu thetaML_d");

  if (cphim_struct->X_d) e = hipFree(cphim_struct->X_d);
  process_err(e, "free gpu X_d");

  if (cphim_struct->Y_d) e = hipFree(cphim_struct->Y_d);
  process_err(e, "free gpu Y_d");

  if (cphim_struct->tab_int_x) e = hipFree(cphim_struct->tab_int_x);
  process_err(e, "free gpu tab_int_x");

  if (cphim_struct->tab_int_y) e = hipFree(cphim_struct->tab_int_y);
  process_err(e, "free gpu tab_int_y");

  if (cphim_struct->xact_d) e = hipFree(cphim_struct->xact_d);
  process_err(e, "free gpu xact_d");

  if (cphim_struct->yact_d) e = hipFree(cphim_struct->yact_d);
  process_err(e, "free gpu yact_d");

  if (cphim_struct->XPup_d) e = hipFree(cphim_struct->XPup_d);
  process_err(e, "free gpu XPup_d");

  if (cphim_struct->YPup_d) e = hipFree(cphim_struct->YPup_d);
  process_err(e, "free gpu YPup_d");

  /*
  if (cphim_struct->Cmm_d) e = hipFree(cphim_struct->Cmm_d);
  process_err(e, "free gpu YPup_d");

  if (cphim_struct->Cpm_d) e = hipFree(cphim_struct->Cpm_d);
  process_err(e, "free gpu YPup_d");

  if (cphim_struct->R_d) e = hipFree(cphim_struct->R_d);
  process_err(e, "free gpu YPup_d");
  */

  if ((cphim_struct->tabDPHI_d) != NULL) e = hipFree(cphim_struct->tabDPHI_d);
  process_err(e, "free gpu tabDPHI_d");

  if ((cphim_struct->L0diff_d) != NULL) e = hipFree(cphim_struct->L0diff_d);
  process_err(e, "free gpu L0diff_d");

  // destroy matcov stream
  e = hipStreamDestroy(cphim_struct->cphim_stream);
  process_err(e, "destroy matcov stream");
}

__global__ void test_dphi_highpass_krnl(double *odata, double *r, double *tabx,
                                        double *taby, double d, long N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < N) {
    double fc = 1 / (2 * d);
    odata[tid] = DPHI_highpass_gb(r[tid], fc, tabx, taby, N);
  }
}

void test_DPHI_highpass(double R, double x0, long npts, carma_device *device) {
  // DEBUG_TRACE("tab_int !\n");
  hipError_t e;
  double *tabx_d;
  double *taby_d;
  e = hipMalloc((void **)&(tabx_d), (npts) * sizeof(double));
  process_err(e, "copy tabx_d");
  e = hipMalloc((void **)&(taby_d), (npts) * sizeof(double));
  process_err(e, "copy taby_d");
  tab_u831J0(tabx_d, taby_d, npts);

  double *tabx = (double *)malloc(npts * sizeof(double));
  double *taby = (double *)malloc(npts * sizeof(double));
  e = hipMemcpy(tabx, tabx_d, (npts) * sizeof(double), hipMemcpyDeviceToHost);
  process_err(e, "copy tabx");
  e = hipMemcpy(taby, taby_d, (npts) * sizeof(double), hipMemcpyDeviceToHost);
  process_err(e, "copy taby");
  FILE *myfile;
  myfile = fopen("tabulated_int.txt", "w");
  for (int i = 0; i < npts; i++) {
    fprintf(myfile, "%9.9f %9.9f\n", tabx[i], taby[i]);
  }
  fclose(myfile);

  double *odata;
  e = hipMalloc((void **)&(odata), (npts) * sizeof(double));
  double *r;
  r = (double *)malloc(npts * sizeof(double));
  double dr = R / (npts - 1);
  for (int i = 0; i < npts; i++) {
    r[i] = i * dr;
  }
  double *r_d;
  e = hipMalloc((void **)&(r_d), (npts) * sizeof(double));
  e = hipMemcpy(r_d, r, (npts) * sizeof(double), hipMemcpyHostToDevice);
  process_err(e, "copy cpu temp");
  process_err(e, "alloc gpu temp_d");
  int nblocks = 0;
  int nthreads = 0;
  getNumBlocksAndThreads(device, npts, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);
  test_dphi_highpass_krnl<<<grid, threads>>>(odata, r_d, tabx_d, taby_d, x0,
                                             npts);

  e = hipMemcpy(tabx, r_d, (npts) * sizeof(double), hipMemcpyDeviceToHost);
  process_err(e, "copy tabx");
  e = hipMemcpy(taby, odata, (npts) * sizeof(double), hipMemcpyDeviceToHost);
  process_err(e, "copy taby");
  myfile = fopen("DPHI_highpass.txt", "w");
  for (int i = 0; i < npts; i++) {
    fprintf(myfile, "%9.9f %9.9f\n", tabx[i], taby[i]);
  }
  fclose(myfile);

  hipFree(tabx_d);
  hipFree(taby_d);
  hipFree(odata);
  hipFree(r_d);
  free(r);
  free(tabx);
  free(taby);
}
