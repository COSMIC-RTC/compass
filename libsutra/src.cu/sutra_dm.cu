#include "hip/hip_runtime.h"
#include <sutra_dm.h>

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template<class T>
struct SharedMemory {
  __device__
  inline operator T*() {
    extern __shared__ int __smem[];
    return (T*) __smem;
  }

  __device__
  inline operator const T*() const {
    extern __shared__ int __smem[];
    return (T*) __smem;
  }
};

// specialize for double to avoid unaligned memory 
// access compile errors
template<>
struct SharedMemory<double> {
  __device__
  inline operator double*() {
    extern __shared__ double __smem_d[];
    return (double*) __smem_d;
  }

  __device__
  inline operator const double*() const {
    extern __shared__ double __smem_d[];
    return (double*) __smem_d;
  }
};

template<class T>
__global__ void dmshape_krnl(T *g_idata, T *g_odata, int *pos, int *istart,
    int *npts, T *comm, unsigned int n, int N) {
  T *sdata = SharedMemory<T>();

  // load shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < N) {
    int local_istart = istart[i];
    int local_npts = npts[i];

    sdata[tid] = 0;

    if (local_npts > 0) {
      for (int cc = 0; cc < local_npts; cc++) {
        int lpos = pos[local_istart + cc];
        int ninflu = lpos / n;
        sdata[tid] += comm[ninflu] * g_idata[lpos];
      }
    }
  }

  __syncthreads();

  if (i < N) {
    // write result for this block to global mem
    g_odata[i] = sdata[tid];
  }
}

template<class T>
void comp_dmshape(int threads, int blocks, T *d_idata, T *d_odata, int *pos,
    int *istart, int *npts, T *comm, unsigned int n, int N) {
  dim3 dimBlock(threads, 1, 1);
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps 
  // worth of shared memory so that we don't index shared memory out of bounds
  int smemSize =
      (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);
  dmshape_krnl<T> <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, pos,
      istart, npts, comm, n, N);

  cutilCheckMsg("dmshape_kernel<<<>>> execution failed\n");

}

template void
comp_dmshape<float>(int threads, int blocks, float *d_idata, float *d_odata,
    int *pos, int *istart, int *npts, float *comm, unsigned int n, int N);

template void
comp_dmshape<double>(int threads, int blocks, double *d_idata, double *d_odata,
    int *pos, int *istart, int *npts, double *comm, unsigned int n, int N);

template<class T>
__global__ void oneactu_krnl(T *g_idata, T *g_odata, int nactu, T ampli,
    int *xoff, int *yoff, int dim_im, int dim_influ, int N) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < N) {
    int iy = i / dim_im;
    int ix = i - iy * dim_im;
    int ixactu = ix - xoff[nactu];
    int iyactu = iy - yoff[nactu];

    // write result for this block to global mem
    if ((ixactu > -1) && (ixactu < dim_influ) && (iyactu > -1)
        && (iyactu < dim_influ)) {
      int tid = ixactu + iyactu * dim_influ + nactu * dim_influ * dim_influ;
      g_odata[i] = ampli * g_idata[tid];
    }
  }
}

template<class T>
__global__ void oneactu_krnl_fast(T *g_idata, T *g_odata, int nactu, T ampli,
    int *xoff, int *yoff, int dim_im, int dim_influ, int N) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < N) {
    int iy = i / dim_influ;
    int ix = i - iy * dim_influ;
    int ixactu = ix + xoff[nactu];
    int iyactu = iy + yoff[nactu];

    // write result for this block to global mem
    if ((ixactu > -1) && (ixactu < dim_im) && (iyactu > -1)
        && (iyactu < dim_im)) {
      int tid = ixactu + iyactu * dim_im;
      g_odata[tid] = ampli * g_idata[i + nactu * dim_influ * dim_influ];
    }
  }
}

template<class T>
void oneactu(int threads, int blocks, T *d_idata, T *d_odata, int nactu,
    T ampli, int *xoff, int *yoff, int dim_im, int dim_influ, int N) {
  dim3 dimBlock(threads, 1, 1);
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps 
  // worth of shared memory so that we don't index shared memory out of bounds
  //oneactu_krnl<T><<< dimGrid, dimBlock >>>(d_idata,d_odata, nactu, ampli, xoff, yoff, dim_im, dim_influ, N);
  oneactu_krnl_fast<T> <<<dimGrid, dimBlock>>>(d_idata, d_odata, nactu, ampli,
      xoff, yoff, dim_im, dim_influ, N);

  cutilCheckMsg("oneactu_kernel<<<>>> execution failed\n");

}

template void
oneactu<float>(int threads, int blocks, float *d_idata, float *d_odata,
    int nactu, float ampli, int *xoff, int *yoff, int dim_im, int dim_influ,
    int N);

template void
oneactu<double>(int threads, int blocks, double *d_idata, double *d_odata,
    int nactu, double ampli, int *xoff, int *yoff, int dim_im, int dim_influ,
    int N);

template<class T>
__global__ void oneactu_krnl_fast(T *g_idata, T *g_odata, int nactu, T ampli,
    int dim_im, int dim_influ, int N) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < N) {
    int iy = i / dim_influ;
    int ix = i - iy * dim_influ;

    // write result for this block to global mem
    if ((ix > -1) && (ix < dim_im) && (iy > -1) && (iy < dim_im)) {
      int tid = ix + iy * dim_im;
      g_odata[tid] = ampli * g_idata[i + nactu * dim_influ * dim_influ];
    }
  }
}

template<class T>
void oneactu(int threads, int blocks, T *d_idata, T *d_odata, int nactu,
    T ampli, int dim_im, int dim_influ, int N) {
  dim3 dimBlock(threads, 1, 1);
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps 
  // worth of shared memory so that we don't index shared memory out of bounds
  //oneactu_krnl<T><<< dimGrid, dimBlock >>>(d_idata,d_odata, nactu, ampli, xoff, yoff, dim_im, dim_influ, N);
  oneactu_krnl_fast<T> <<<dimGrid, dimBlock>>>(d_idata, d_odata, nactu, ampli,
      dim_im, dim_influ, N);

  cutilCheckMsg("oneactu_kernel<<<>>> execution failed\n");

}

template void
oneactu<float>(int threads, int blocks, float *d_idata, float *d_odata,
    int nactu, float ampli, int dim_im, int dim_influ, int N);

template void
oneactu<double>(int threads, int blocks, double *d_idata, double *d_odata,
    int nactu, double ampli, int dim_im, int dim_influ, int N);

template<class T>
__global__ void fulldmshape_krnl(T *g_idata, T *g_odata, int ninflu,
    int diminflu, T *comm, int N) {
  T *sdata = SharedMemory<T>();

  // load shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < N) {
    sdata[tid] = 0;

    for (int cc = 0; cc < ninflu; cc++) {
      sdata[tid] += comm[cc] * g_idata[i + cc * diminflu];
    }
  }
  __syncthreads();

  if (i < N) {
    // write result for this block to global mem
    g_odata[i] = sdata[tid];
  }
}

template<class T>
void comp_fulldmshape(int threads, int blocks, T *d_idata, T *d_odata,
    int ninflu, int diminflu, T *comm, int N) {
  dim3 dimBlock(threads, 1, 1);
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps 
  // worth of shared memory so that we don't index shared memory out of bounds
  int smemSize =
      (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);
  fulldmshape_krnl<T> <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata,
      ninflu, diminflu, comm, N);

  cutilCheckMsg("fulldmshape_kernel<<<>>> execution failed\n");

}

template void
comp_fulldmshape<float>(int threads, int blocks, float *d_idata, float *d_odata,
    int ninflu, int diminflu, float *comm, int N);

template void
comp_fulldmshape<double>(int threads, int blocks, double *d_idata,
    double *d_odata, int ninflu, int diminflu, double *comm, int N);

