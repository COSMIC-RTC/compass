#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
//  This file is part of COMPASS <https://anr-compass.github.io/compass/>
//
//  Copyright (C) 2011-2023 COMPASS Team <https://github.com/ANR-COMPASS>
//  All rights reserved.

// -----------------------------------------------------------------------------

//! \file      sutra_coronagraph.cu
//! \ingroup   libsutra
//! \class     SutraCoronagraph
//! \brief     this class provides the coronagraph features to COMPASS
//! \author    COMPASS Team <https://github.com/ANR-COMPASS>
//! \version   5.5.0
//! \date      2022/01/24

#include <sutra_coronagraph.h>
#include <carma_utils.cuh>

__global__ void compute_electric_field_krnl(hipFloatComplex *ef, float* opd, float scale,
                            float* amplitude, float* mask, int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        float A = amplitude[tid];
        float _opd = opd[tid];
        float _mask = mask[tid];
        ef[tid].x = A * cosf(scale * _opd) * _mask;
        ef[tid].y = A * sinf(scale * _opd) * _mask;
        tid += blockDim.x * gridDim.x;
    }
}

int compute_electric_field(hipFloatComplex *electric_field, float* phase_opd, float scale,
                            float* amplitude, float* mask, int dimx, int dimy, CarmaDevice *device) {
    int nBlocks, nThreads;
    get_num_blocks_and_threads(device, dimx*dimy, nBlocks, nThreads);
    dim3 grid(nBlocks), threads(nThreads);

    compute_electric_field_krnl<<<grid, threads>>>(electric_field, phase_opd, scale,
                                                    amplitude, mask, dimx*dimy);
    return EXIT_SUCCESS;
}

__global__ void remove_complex_avg_krnl(hipFloatComplex *ef, hipFloatComplex sum,
                                    float* mask, int Nvalid, int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N) {
        float _mask = mask[tid];
        ef[tid].x -= (sum.x / Nvalid * _mask);
        ef[tid].y -= (sum.y / Nvalid * _mask);
        tid += blockDim.x * gridDim.x;
    }
}
int remove_complex_avg(hipFloatComplex *electric_field, hipFloatComplex sum, float* mask, int Nvalid,
                        int dimx, int dimy, CarmaDevice *device) {

    int nBlocks, nThreads;
    get_num_blocks_and_threads(device, dimx*dimy, nBlocks, nThreads);
    dim3 grid(nBlocks), threads(nThreads);

    remove_complex_avg_krnl<<<grid, threads>>>(electric_field, sum, mask, Nvalid, dimx*dimy);
    return EXIT_SUCCESS;
}

__global__ void accumulate_abs2_krnl(hipFloatComplex *img, float* abs2img, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid < N) {
    hipFloatComplex cache = img[tid];
    abs2img[tid] += (cache.x * cache.x + cache.y * cache.y);
    tid += blockDim.x * gridDim.x;
  }
}

int accumulate_abs2(hipFloatComplex *img, float* abs2img, int N, CarmaDevice *device) {
  int nBlocks, nThreads;
  get_num_blocks_and_threads(device, N, nBlocks, nThreads);
  dim3 grid(nBlocks), threads(nThreads);

  accumulate_abs2_krnl<<<grid, threads>>>(img, abs2img, N);

  return EXIT_SUCCESS;
}

__global__ void apply_mask_krnl(hipFloatComplex *electric_field, float* mask, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid < N) {
    float cache = mask[tid];
    electric_field[tid].x *= cache;
    electric_field[tid].y *= cache;
    tid += blockDim.x * gridDim.x;
  }
}

int apply_mask(hipFloatComplex *electric_field, float* mask, int N, CarmaDevice *device) {
  int nBlocks, nThreads;
  get_num_blocks_and_threads(device, N, nBlocks, nThreads);
  dim3 grid(nBlocks), threads(nThreads);

  apply_mask_krnl<<<grid, threads>>>(electric_field, mask, N);

  return EXIT_SUCCESS;

}
