#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
//  This file is part of COMPASS <https://anr-compass.github.io/compass/>
//
//  Copyright (C) 2011-2023 COMPASS Team <https://github.com/ANR-COMPASS>
//  All rights reserved.

// -----------------------------------------------------------------------------

//! \file      sutra_centroider_pyr.cu
//! \ingroup   libsutra
//! \class     SutraCentroiderPyr
//! \brief     this class provides the centroider_pyr features to COMPASS
//! \author    COMPASS Team <https://github.com/ANR-COMPASS>
//! \version   5.4.2
//! \date      2022/01/24

#include <carma_utils.cuh>
#include "sutra_centroider_utils.cuh"
#include <sutra_centroider_pyr.h>

template <class T>
__global__ void pyrslopes_krnl(T *g_odata, T *g_idata, int *subindx,
                               int *subindy, float *intensities,
                               sutra::SlopesIndex si,
                               unsigned int ns, unsigned int nvalid,
                               unsigned int nim) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < nvalid) {
        int i1 = subindx[i] + subindy[i] * ns;
        int i2 = subindx[i + nvalid] + subindy[i + nvalid] * ns;
        int i3 = subindx[i + 2 * nvalid] + subindy[i + 2 * nvalid] * ns;
        int i4 = subindx[i + 3 * nvalid] + subindy[i + 3 * nvalid] * ns;

        g_odata[si.x(i)] = ((g_idata[i2] + g_idata[i4]) - (g_idata[i1] + g_idata[i3])) / intensities[i];
        g_odata[si.y(i)] = ((g_idata[i3] + g_idata[i4]) - (g_idata[i1] + g_idata[i2])) / intensities[i];
    }
}

template <class T>
void pyr_slopes(T *d_odata, T *d_idata, int *subindx, int *subindy,
                float *intensities, int ns, int nvalid, int nim,
                SlopeOrder slope_order, CarmaDevice *device) {
  // cout << "hello cu" << endl;

    int nb_blocks, nb_threads;
    get_num_blocks_and_threads(device, nvalid, nb_blocks, nb_threads);
    dim3 grid(nb_blocks), threads(nb_threads);

    sutra::SlopesIndex si{nvalid, slope_order};


    pyrslopes_krnl<T><<<grid, threads>>>(d_odata, d_idata, subindx, subindy,
                                         intensities, si, ns, nvalid, nim);

    carma_check_msg("pyrslopes_kernel<<<>>> execution failed\n");
}

template void pyr_slopes<float> (float *d_odata, float *d_idata, int *subindx,
                                 int *subindy, float *intensities, int ns,
                                 int nvalid, int nim, SlopeOrder slope_order,
                                 CarmaDevice *device);
template void pyr_slopes<double>(double *d_odata, double *d_idata, int *subindx,
                                 int *subindy, float *intensities, int ns,
                                 int nvalid, int nim, SlopeOrder slope_order,
                                 CarmaDevice *device);

template <class T, T fct_sin(T)>
__global__ void pyr2slopes_krnl(T *g_odata, T *ref, T *g_idata, int *subindx,
                                int *subindy, float *intensities,
                                sutra::SlopesIndex si,
                                unsigned int ns, unsigned int nvalid,
                                float scale, T valid_thresh, int do_sin) {

  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  T tmp;
  const T cmin(-1);
  const T cmax(1);
  while (i < nvalid) {
    const int iq1 = subindx[i] + subindy[i] * ns;
    const int iq2 = subindx[i + nvalid] + subindy[i + nvalid] * ns;
    const int iq3 = subindx[i + 2 * nvalid] + subindy[i + 2 * nvalid] * ns;
    const int iq4 = subindx[i + 3 * nvalid] + subindy[i + 3 * nvalid] * ns;

    if (intensities[i] < valid_thresh) { // flux too low -> set slopes to 9
        g_odata[si.x(i)] = 0;
        g_odata[si.y(i)] = 0;
    } else {
        tmp = ((g_idata[iq1] + g_idata[iq4]) - (g_idata[iq2] + g_idata[iq3])) /
              intensities[i];
        //tmp = carma_clip(tmp, cmin, cmax); // clip unexpected values
        if (do_sin) {
            g_odata[si.x(i)] = scale * fct_sin( tmp / 2. ); // fct_sin calculates the sine of the input argument × π .
        } else {
            g_odata[si.x(i)] = scale * tmp;
        }
        tmp = ((g_idata[iq1] + g_idata[iq3]) - (g_idata[iq2] + g_idata[iq4])) /
              intensities[i];
        //tmp = carma_clip(tmp, cmin, cmax); // clip unexpected values
        if (do_sin) {
            g_odata[si.y(i)] = scale * fct_sin( tmp / 2. ); // fct_sin calculates the sine of the input argument × π .
        } else {
            g_odata[si.y(i)] = scale * tmp;
        }
    }
    g_odata[si.x(i)] -= ref[si.x(i)];
    g_odata[si.y(i)] -= ref[si.y(i)];
    i += blockDim.x * gridDim.x;
  }
}

template <class T, T fct_sin(T)>
void pyr2_slopes_full(T *d_odata, T *ref, T *d_idata, int *subindx,
                      int *subindy, float *intensities, int ns, int nvalid,
                      float scale, T valid_thresh, int do_sin,
                      SlopeOrder slope_order, CarmaDevice *device) {
  int nb_blocks, nb_threads;
  get_num_blocks_and_threads(device, nvalid, nb_blocks, nb_threads);
  dim3 grid(nb_blocks), threads(nb_threads);

  sutra::SlopesIndex si{nvalid, slope_order};

  pyr2slopes_krnl<T, fct_sin>
      <<<grid, threads>>>(d_odata, ref, d_idata, subindx, subindy, intensities,
                          si,
                          ns, nvalid, scale, valid_thresh, do_sin);

  carma_check_msg("pyrslopes_kernel<<<>>> execution failed\n");
}

template <>
void pyr2_slopes<float>(float *d_odata, float *ref, float *d_idata,
                        int *subindx, int *subindy, float *intensities, int ns,
                        int nvalid, float scale, float valid_thresh, int do_sin,
                        SlopeOrder slope_order, CarmaDevice *device) {
  pyr2_slopes_full<float, sinpif>(d_odata, ref, d_idata, subindx, subindy,
                                  intensities, ns, nvalid, scale, valid_thresh,
                                  do_sin, slope_order, device);
}
template <>
void pyr2_slopes<double>(double *d_odata, double *ref, double *d_idata,
                         int *subindx, int *subindy, float *intensities, int ns,
                         int nvalid, float scale, double valid_thresh, int do_sin,
                         SlopeOrder slope_order, CarmaDevice *device) {
  pyr2_slopes_full<double, sinpi>(d_odata, ref, d_idata, subindx, subindy,
                                  intensities, ns, nvalid, scale, valid_thresh,
                                  do_sin, slope_order, device);
}
