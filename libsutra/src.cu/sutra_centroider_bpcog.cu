#include "hip/hip_runtime.h"
#include <sutra_centroider_bpcog.h>
#include <carma_utils.cuh>

template <typename T, int BLOCK_THREADS>
__launch_bounds__(BLOCK_THREADS) __global__
    void centroids(T *d_img, T *d_centroids, T *ref, int *validx, int *validy,
                   T *d_intensities, int nbpix, unsigned int npix,
                   unsigned int size, T scale, T offset,
                   unsigned int nelem_thread) {
  // Specialize BlockRadixSort for a 1D block of BLOCK_THREADS threads owning 1
  // item each
  typedef hipcub::BlockRadixSort<T, BLOCK_THREADS, 1> BlockRadixSortT;
  typedef hipcub::BlockReduce<T, BLOCK_THREADS> BlockReduce;

  // Allocate shared memory for BlockRadixSort
  __shared__ typename BlockRadixSortT::TempStorage temp_storageSort;
  __shared__ typename BlockReduce::TempStorage temp_storageSum;
  __shared__ T threshold;

  T idata = 0;
  T xdata = 0;
  T ydata = 0;

  unsigned int tid = threadIdx.x;
  unsigned int xvalid = validx[blockIdx.x];
  unsigned int yvalid = validy[blockIdx.x];
  unsigned int x = tid % npix;
  unsigned int y = tid / npix;
  int idim = (x + xvalid) + (y + yvalid) * size;

  T items[1];
  items[0] = d_img[idim];

  __syncthreads();
  BlockRadixSortT(temp_storageSort).SortDescending(items);

  if (tid == nbpix) threshold = items[0];

  __syncthreads();

  if (idim < size * size) {
    const T data_thresh = d_img[idim] - threshold;
    idata += (data_thresh > 0) ? data_thresh : 0;
    xdata += (data_thresh > 0) ? data_thresh * x : 0;
    ydata += (data_thresh > 0) ? data_thresh * y : 0;
    d_img[idim] = data_thresh;
  }

  __syncthreads();
  T intensity = BlockReduce(temp_storageSum).Sum(idata, blockDim.x);
  T slopex = BlockReduce(temp_storageSum).Sum(xdata, blockDim.x);
  T slopey = BlockReduce(temp_storageSum).Sum(ydata, blockDim.x);
  if (tid == 0) {
    d_centroids[blockIdx.x] =
        ((slopex * 1.0 / (intensity + 1.e-6)) - offset) * scale -
        ref[blockIdx.x];
    d_centroids[blockIdx.x + gridDim.x] =
        ((slopey * 1.0 / (intensity + 1.e-6)) - offset) * scale -
        ref[blockIdx.x + gridDim.x];
    d_intensities[blockIdx.x] = intensity;
  }
}

template <class T>
void get_centroids(int size, int threads, int blocks, int npix, T *d_img,
                   T *d_centroids, T *ref, int *validx, int *validy,
                   T *intensities, int nbpix, T scale, T offset,
                   carma_device *device) {
  int maxThreads = device->get_properties().maxThreadsPerBlock;
  unsigned int nelem_thread = 1;
  while ((threads / nelem_thread > maxThreads) ||
         (threads % nelem_thread != 0)) {
    nelem_thread++;
  }

  threads /= nelem_thread;
  dim3 dimBlock(threads, 1, 1);
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  if (threads <= 16)
    centroids<T, 16><<<dimGrid, dimBlock>>>(d_img, d_centroids, ref, validx,
                                            validy, intensities, nbpix, npix,
                                            size, scale, offset, nelem_thread);
  else if (threads <= 32)
    centroids<T, 32><<<dimGrid, dimBlock>>>(d_img, d_centroids, ref, validx,
                                            validy, intensities, nbpix, npix,
                                            size, scale, offset, nelem_thread);

  else if (threads <= 64)
    centroids<T, 64><<<dimGrid, dimBlock>>>(d_img, d_centroids, ref, validx,
                                            validy, intensities, nbpix, npix,
                                            size, scale, offset, nelem_thread);
  else if (threads <= 128)
    centroids<T, 128><<<dimGrid, dimBlock>>>(d_img, d_centroids, ref, validx,
                                             validy, intensities, nbpix, npix,
                                             size, scale, offset, nelem_thread);
  else if (threads <= 256)
    centroids<T, 256><<<dimGrid, dimBlock>>>(d_img, d_centroids, ref, validx,
                                             validy, intensities, nbpix, npix,
                                             size, scale, offset, nelem_thread);
  else if (threads <= 512)
    centroids<T, 512><<<dimGrid, dimBlock>>>(d_img, d_centroids, ref, validx,
                                             validy, intensities, nbpix, npix,
                                             size, scale, offset, nelem_thread);
  else
    printf("SH way too big !!!\n");

  carmaCheckMsg("centroids_kernel<<<>>> execution failed\n");
}

template void get_centroids<float>(int size, int threads, int blocks, int npix,
                                   float *d_img, float *d_centroids, float *ref,
                                   int *validx, int *validy, float *intensities,
                                   int nbpix, float scale, float offset,
                                   carma_device *device);

template void get_centroids<double>(int size, int threads, int blocks, int npix,
                                    double *d_img, double *d_centroids,
                                    double *ref, int *validx, int *validy,
                                    double *intensities, int nbpix,
                                    double scale, double offset,
                                    carma_device *device);

template <class T>
__device__ inline void sortmax_krnl(T *sdata, unsigned int *values, int size,
                                    int n) {
  if (!((size & (size - 1)) == 0)) {
    unsigned int s;
    if ((size & 1) != 0)
      s = size / 2 + 1;  //(size&1)==size%2
    else
      s = size / 2;
    unsigned int s_old = size;
    while (s > 0) {
      if ((n < s) && (n + s < s_old)) {
        if (sdata[n] < sdata[n + s]) {
          mswap(values[n], values[n + s]);
          mswap(sdata[n], sdata[n + s]);
        }
      }
      __syncthreads();
      s_old = s;
      s /= 2;
      if ((2 * s < s_old) && (s != 0)) s += 1;
    }
  } else {
    // do reduction in shared mem
    for (unsigned int s = size / 2; s > 0; s >>= 1) {
      if (n < s) {
        if (sdata[n] < sdata[n + s]) {
          mswap(values[n], values[n + s]);
          mswap(sdata[n], sdata[n + s]);
        }
      }
      __syncthreads();
    }
  }
}

template <class T>
__global__ void sortmax(T *g_idata, T *g_odata, unsigned int *values, int nmax,
                        int Npix, int size, int nelem_thread) {
  extern __shared__ uint svalues[];
  T *sdata = (T *)&svalues[Npix];
  /*
    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    svalues[tid] = tid;
    sdata[tid] = g_idata[i];
  */
  unsigned int tid = threadIdx.x;
  // unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  // unsigned int y = (tid / n) + 1;
  int idim;
  int sdim;

  for (int cc = 0; cc < nelem_thread; cc++) {
    idim = tid * nelem_thread + cc + (blockDim.x * nelem_thread) * blockIdx.x;
    sdim = tid * nelem_thread + cc;
    if (idim < size) {
      sdata[sdim] = g_idata[idim];
      svalues[sdim] = sdim;
    }
  }

  __syncthreads();

  for (int cc = 0; cc < nmax; cc++) {
    for (int cpt = 0; cpt < nelem_thread; cpt++) {
      sdim = tid * nelem_thread + cpt;
      if (sdim >= cc)
        sortmax_krnl(&(sdata[cc]), &(svalues[cc]), Npix - cc, sdim - cc);
      __syncthreads();
    }
  }
  for (int cpt = 0; cpt < nelem_thread; cpt++) {
    sdim = tid * nelem_thread + cpt;
    if (sdim < nmax) {
      g_odata[nmax * blockIdx.x + sdim] = sdata[sdim];  // - sdata[nmax - 1];
      values[nmax * blockIdx.x + sdim] = svalues[sdim];
    }
  }
  /*
   __syncthreads();
   if ((blockIdx.x == 0) && (tid < nmax))
   printf("tid %d sdata %f \n",tid,g_odata[tid]);
   */
}

template <class T>
void subap_sortmax(int threads, int blocks, T *d_idata, T *d_odata,
                   unsigned int *values, int nmax, carma_device *device) {
  int maxThreads = device->get_properties().maxThreadsPerBlock;
  unsigned int nelem_thread = 1;
  while ((threads / nelem_thread > maxThreads) ||
         (threads % nelem_thread != 0)) {
    nelem_thread++;
  }

  dim3 dimBlock(threads / nelem_thread, 1, 1);
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  size_t smemSize = threads * (sizeof(T) + sizeof(uint));
  sortmax<T><<<dimGrid, dimBlock, smemSize>>>(
      d_idata, d_odata, values, nmax, threads, threads * blocks, nelem_thread);

  carmaCheckMsg("sortmax_kernel<<<>>> execution failed\n");
}
template void subap_sortmax<float>(int threads, int blocks, float *d_idata,
                                   float *d_odata, unsigned int *values,
                                   int nmax, carma_device *device);
template void subap_sortmax<double>(int threads, int blocks, double *d_idata,
                                    double *d_odata, unsigned int *values,
                                    int nmax, carma_device *device);

template <class T>
__global__ void centroid_bpix(int nsub, int n, T *g_idata, unsigned int *values,
                              T *g_odata, T scale, T offset) {
  extern __shared__ uint svalues[];
  T *sdata = (T *)&svalues[blockDim.x];
  T intensities;
  // T minimum;

  // load shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  svalues[tid] = values[i];
  sdata[tid] = g_idata[i] - g_idata[blockIdx.x * blockDim.x + blockDim.x - 1];

  __syncthreads();

  reduce_krnl(sdata, blockDim.x, tid);

  __syncthreads();
  // get the sum per subap
  if (tid == 0) intensities = (abs(sdata[tid]) > 1.e-6 ? sdata[tid] : 0.0f);

  __syncthreads();

  // Reload sdata
  sdata[tid] = g_idata[i] - g_idata[blockIdx.x * blockDim.x + blockDim.x - 1];

  __syncthreads();

  // compute the centroid on the first part of the array
  sdata[tid] *= ((svalues[tid] % n));
  // x centroid
  __syncthreads();
  reduce_krnl(sdata, blockDim.x, tid);
  //__syncthreads();
  if (tid == 0)
    g_odata[blockIdx.x] =
        (intensities != 0.0f ? ((sdata[tid] / intensities) - offset) * scale
                             : 0.0f);
  __syncthreads();
  sdata[tid] = g_idata[i] - g_idata[blockIdx.x * blockDim.x + blockDim.x - 1];

  __syncthreads();

  // compute the centroid on the first part of the array
  sdata[tid] *= (svalues[tid] / n);
  // y centroid
  __syncthreads();

  reduce_krnl(sdata, blockDim.x, tid);
  //__syncthreads();
  if (tid == 0)
    g_odata[blockIdx.x + nsub] =
        (intensities != 0.0f ? ((sdata[tid] / intensities) - offset) * scale
                             : 0.0f);
}

template <class T>
void subap_bpcentro(int threads, int blocks, int npix, T *d_idata,
                    unsigned int *values, T *d_odata, T scale, T offset) {
  dim3 dimBlock(threads, 1, 1);
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  int smemSize = threads * (sizeof(T) + sizeof(uint));

  centroid_bpix<T><<<dimGrid, dimBlock, smemSize>>>(
      blocks, npix, d_idata, values, d_odata, scale, offset);

  carmaCheckMsg("centroid_bpix<<<>>> execution failed\n");
}
template void subap_bpcentro<float>(int threads, int blocks, int npix,
                                    float *d_idata, unsigned int *values,
                                    float *d_odata, float scale, float offset);
template void subap_bpcentro<double>(int threads, int blocks, int npix,
                                     double *d_idata, unsigned int *values,
                                     double *d_odata, double scale,
                                     double offset);
