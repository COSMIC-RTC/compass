#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
//  This file is part of COMPASS <https://anr-compass.github.io/compass/>
//
//  Copyright (C) 2011-2023 COMPASS Team <https://github.com/ANR-COMPASS>
//  All rights reserved.

// -----------------------------------------------------------------------------

//! \file      sutra_centroider_pbcog.cu
//! \ingroup   libsutra
//! \class     sutra_centroider_pbcog
//! \brief     this class provides the centroider_pbcog features to COMPASS
//! \author    COMPASS Team <https://github.com/ANR-COMPASS>
//! \version   5.4.2
//! \date      2022/01/24

#include <sutra_centroider_bpcog.h>
#include <sutra_centroider_utils.cuh>
#include <carma_utils.cuh>

template <int BLOCK_THREADS, typename T>
__launch_bounds__(BLOCK_THREADS) __global__
    void centroids(float *d_img, T *d_centroids, T *ref, int *validx,
                   int *validy, float *d_intensities, int nbpix,
                   unsigned int npix, sutra::SlopesIndex si, unsigned int size, T scale, T offset,
                   unsigned int nelem_thread) {
  // Specialize BlockRadixSort for a 1D block of BLOCK_THREADS threads owning 1
  // item each
  typedef hipcub::BlockRadixSort<float, BLOCK_THREADS, 1> BlockRadixSortT;
  typedef hipcub::BlockReduce<float, BLOCK_THREADS> BlockReduce;

  // Allocate shared memory for BlockRadixSort
  __shared__ typename BlockRadixSortT::TempStorage temp_storageSort;
  __shared__ typename BlockReduce::TempStorage temp_storageSum;
  __shared__ float threshold;

  float idata = 0;
  float xdata = 0;
  float ydata = 0;

  unsigned int tid = threadIdx.x;
  unsigned int xvalid = validx[blockIdx.x];
  unsigned int yvalid = validy[blockIdx.x];
  unsigned int x = tid % npix;
  unsigned int y = tid / npix;
  int idim = (x + xvalid) + (y + yvalid) * size;

  float items[1];
  items[0] = ((idim < size * size) && (tid < npix * npix)) ? d_img[idim] : 0.f;

  __syncthreads();
  BlockRadixSortT(temp_storageSort).SortDescending(items);

  if (tid == nbpix) threshold = items[0];

  __syncthreads();
  if ((idim < size * size) && (tid < npix * npix)) {
    float data_thresh =
        (d_img[idim] > threshold) ? d_img[idim] - threshold : 0.f;
    idata += data_thresh;
    xdata += data_thresh * x;
    ydata += data_thresh * y;
    d_img[idim] = data_thresh;
  }

  __syncthreads();
  float intensity = BlockReduce(temp_storageSum).Sum(idata, npix * npix);
  __syncthreads();
  float slopex = BlockReduce(temp_storageSum).Sum(xdata, npix * npix);
  __syncthreads();
  float slopey = BlockReduce(temp_storageSum).Sum(ydata, npix * npix);

  if (tid == 0) {
    d_centroids[si.x(blockIdx.x)] = (T(slopex / (intensity + 1.e-6)) - offset) * scale - ref[si.x(blockIdx.x)];
    d_centroids[si.y(blockIdx.x)] = (T(slopey / (intensity + 1.e-6)) - offset) * scale - ref[si.y(blockIdx.x)];
    d_intensities[blockIdx.x] = intensity;
  }
}

template <class T>
void get_centroids(int size, int threads, int blocks, int npix, float *d_img,
                   T *d_centroids, T *ref, int *validx, int *validy,
                   float *intensities, int nbpix, float scale, float offset,
                   SlopeOrder slope_order,
                   CarmaDevice *device) {
  int maxThreads = device->get_properties().maxThreadsPerBlock;
  unsigned int nelem_thread = 1;
  while ((threads / nelem_thread > maxThreads) ||
         (threads % nelem_thread != 0)) {
    nelem_thread++;
  }

  sutra::SlopesIndex si{blocks, slope_order};

  threads /= nelem_thread;
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  if (threads <= 16)
    centroids<  16><<<dimGrid, threads>>>(d_img, d_centroids, ref, validx, validy,
                                          intensities, nbpix, npix, si, size,
                                          T(scale), T(offset), nelem_thread);
  else if (threads <= 36)
    centroids<  36><<<dimGrid, threads>>>(d_img, d_centroids, ref, validx, validy,
                                          intensities, nbpix, npix, si, size,
                                          T(scale), T(offset), nelem_thread);
  else if (threads <= 64)
    centroids<  64><<<dimGrid, threads>>>(d_img, d_centroids, ref, validx, validy,
                                          intensities, nbpix, npix, si, size,
                                          T(scale), T(offset), nelem_thread);
  else if (threads <= 100)
    centroids< 100><<<dimGrid, threads>>>(d_img, d_centroids, ref, validx, validy,
                                          intensities, nbpix, npix, si, size,
                                          T(scale), T(offset), nelem_thread);
  else if (threads <= 144)
    centroids< 144><<<dimGrid, threads>>>(d_img, d_centroids, ref, validx, validy,
                                          intensities, nbpix, npix, si, size,
                                          T(scale), T(offset), nelem_thread);
  else if (threads <= 256)
    centroids< 256><<<dimGrid, threads>>>(d_img, d_centroids, ref, validx, validy,
                                          intensities, nbpix, npix, si, size,
                                          T(scale), T(offset), nelem_thread);
  else if (threads <= 512)
    centroids< 512><<<dimGrid, threads>>>(d_img, d_centroids, ref, validx, validy,
                                          intensities, nbpix, npix, si, size,
                                          T(scale), T(offset), nelem_thread);
  else if (threads <= 1024)
    centroids<1024><<<dimGrid, threads>>>(d_img, d_centroids, ref, validx, validy,
                                          intensities, nbpix, npix, si, size,
                                          T(scale), T(offset), nelem_thread);
  else
    printf("SH way too big !!!\n");

  carma_check_msg("centroids_kernel<<<>>> execution failed\n");
}

template void get_centroids<float>(int size, int threads, int blocks, int npix,
                                   float *d_img, float *d_centroids, float *ref,
                                   int *validx, int *validy, float *intensities,
                                   int nbpix, float scale, float offset,
                                   SlopeOrder slope_order,
                                   CarmaDevice *device);

template void get_centroids<double>(int size, int threads, int blocks, int npix,
                                    float *d_img, double *d_centroids,
                                    double *ref, int *validx, int *validy,
                                    float *intensities, int nbpix, float scale,
                                    float offset, SlopeOrder slope_order,
                                    CarmaDevice *device);
#ifdef CAN_DO_HALF
template void get_centroids<half>(int size, int threads, int blocks, int npix,
                                  float *d_img, half *d_centroids, half *ref,
                                  int *validx, int *validy, float *intensities,
                                  int nbpix, float scale, float offset,
                                  SlopeOrder slope_order,
                                  CarmaDevice *device);
#endif

template <class T>
__device__ inline void sortmax_krnl(T *sdata, unsigned int *values, int size,
                                    int n) {
  if (!((size & (size - 1)) == 0)) {
    unsigned int s;
    if ((size & 1) != 0)
      s = size / 2 + 1;  //(size&1)==size%2
    else
      s = size / 2;
    unsigned int s_old = size;
    while (s > 0) {
      if ((n < s) && (n + s < s_old)) {
        if (sdata[n] < sdata[n + s]) {
          mswap(values[n], values[n + s]);
          mswap(sdata[n], sdata[n + s]);
        }
      }
      __syncthreads();
      s_old = s;
      s /= 2;
      if ((2 * s < s_old) && (s != 0)) s += 1;
    }
  } else {
    // do reduction in shared mem
    for (unsigned int s = size / 2; s > 0; s >>= 1) {
      if (n < s) {
        if (sdata[n] < sdata[n + s]) {
          mswap(values[n], values[n + s]);
          mswap(sdata[n], sdata[n + s]);
        }
      }
      __syncthreads();
    }
  }
}

template <class T>
__global__ void sortmax(T *g_idata, T *g_odata, unsigned int *values, int nmax,
                        int Npix, int size, int nelem_thread) {
  extern __shared__ uint svalues[];
  T *sdata = (T *)&svalues[Npix];
  /*
    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    svalues[tid] = tid;
    sdata[tid] = g_idata[i];
  */
  unsigned int tid = threadIdx.x;
  // unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  // unsigned int y = (tid / n) + 1;
  int idim;
  int sdim;

  for (int cc = 0; cc < nelem_thread; cc++) {
    idim = tid * nelem_thread + cc + (blockDim.x * nelem_thread) * blockIdx.x;
    sdim = tid * nelem_thread + cc;
    if (idim < size) {
      sdata[sdim] = g_idata[idim];
      svalues[sdim] = sdim;
    }
  }

  __syncthreads();

  for (int cc = 0; cc < nmax; cc++) {
    for (int cpt = 0; cpt < nelem_thread; cpt++) {
      sdim = tid * nelem_thread + cpt;
      if (sdim >= cc)
        sortmax_krnl(&(sdata[cc]), &(svalues[cc]), Npix - cc, sdim - cc);
      __syncthreads();
    }
  }
  for (int cpt = 0; cpt < nelem_thread; cpt++) {
    sdim = tid * nelem_thread + cpt;
    if (sdim < nmax) {
      g_odata[nmax * blockIdx.x + sdim] = sdata[sdim];  // - sdata[nmax - 1];
      values[nmax * blockIdx.x + sdim] = svalues[sdim];
    }
  }
  /*
   __syncthreads();
   if ((blockIdx.x == 0) && (tid < nmax))
   printf("tid %d sdata %f \n",tid,g_odata[tid]);
   */
}

template <class T>
void subap_sortmax(int threads, int blocks, T *d_idata, T *d_odata,
                   unsigned int *values, int nmax, CarmaDevice *device) {
  int maxThreads = device->get_properties().maxThreadsPerBlock;
  unsigned int nelem_thread = 1;
  while ((threads / nelem_thread > maxThreads) ||
         (threads % nelem_thread != 0)) {
    nelem_thread++;
  }

  dim3 dimBlock(threads / nelem_thread, 1, 1);
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  size_t smemSize = threads * (sizeof(T) + sizeof(uint));
  sortmax<T><<<dimGrid, dimBlock, smemSize>>>(
      d_idata, d_odata, values, nmax, threads, threads * blocks, nelem_thread);

  carma_check_msg("sortmax_kernel<<<>>> execution failed\n");
}
template void subap_sortmax<float>(int threads, int blocks, float *d_idata,
                                   float *d_odata, unsigned int *values,
                                   int nmax, CarmaDevice *device);
template void subap_sortmax<double>(int threads, int blocks, double *d_idata,
                                    double *d_odata, unsigned int *values,
                                    int nmax, CarmaDevice *device);

template <class T>
__global__ void centroid_bpix(int nsub, int n, T *g_idata, unsigned int *values,
                              T *g_odata, float scale, float offset) {
  extern __shared__ uint svalues[];
  T *sdata = (T *)&svalues[blockDim.x];
  T intensities;
  // T minimum;

  // load shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  svalues[tid] = values[i];
  sdata[tid] = g_idata[i] - g_idata[blockIdx.x * blockDim.x + blockDim.x - 1];

  __syncthreads();

  reduce_krnl(sdata, blockDim.x, tid);

  __syncthreads();
  // get the sum per subap
  if (tid == 0) intensities = (abs(sdata[tid]) > 1.e-6 ? sdata[tid] : 0.0f);

  __syncthreads();

  // Reload sdata
  sdata[tid] = g_idata[i] - g_idata[blockIdx.x * blockDim.x + blockDim.x - 1];

  __syncthreads();

  // compute the centroid on the first part of the array
  sdata[tid] *= ((svalues[tid] % n));
  // x centroid
  __syncthreads();
  reduce_krnl(sdata, blockDim.x, tid);
  //__syncthreads();
  if (tid == 0)
    g_odata[blockIdx.x] =
        (intensities != 0.0f ? ((sdata[tid] / intensities) - offset) * scale
                             : 0.0f);
  __syncthreads();
  sdata[tid] = g_idata[i] - g_idata[blockIdx.x * blockDim.x + blockDim.x - 1];

  __syncthreads();

  // compute the centroid on the first part of the array
  sdata[tid] *= (svalues[tid] / n);
  // y centroid
  __syncthreads();

  reduce_krnl(sdata, blockDim.x, tid);
  //__syncthreads();
  if (tid == 0)
    g_odata[blockIdx.x + nsub] =
        (intensities != 0.0f ? ((sdata[tid] / intensities) - offset) * scale
                             : 0.0f);
}

template <class T>
void subap_bpcentro(int threads, int blocks, int npix, T *d_idata,
                    unsigned int *values, T *d_odata, float scale,
                    float offset) {
  dim3 dimBlock(threads, 1, 1);
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  int smemSize = threads * (sizeof(T) + sizeof(uint));

  centroid_bpix<T><<<dimGrid, dimBlock, smemSize>>>(
      blocks, npix, d_idata, values, d_odata, scale, offset);

  carma_check_msg("centroid_bpix<<<>>> execution failed\n");
}
template void subap_bpcentro<float>(int threads, int blocks, int npix,
                                    float *d_idata, unsigned int *values,
                                    float *d_odata, float scale, float offset);
template void subap_bpcentro<double>(int threads, int blocks, int npix,
                                     double *d_idata, unsigned int *values,
                                     double *d_odata, float scale,
                                     float offset);
