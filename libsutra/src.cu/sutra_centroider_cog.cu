#include "hip/hip_runtime.h"
#include <sutra_centroider_cog.h>
#include <carma_utils.cuh>

template <class T, int Nthreads>
__global__ void centroids(T *d_img, T *d_centroids, T *ref, int *validx,
                          int *validy, T *d_intensities, unsigned int npix,
                          unsigned int size, T scale, T offset,
                          unsigned int nelem_thread) {
  if (blockDim.x > Nthreads) {
    if (threadIdx.x == 0) printf("Wrong size argument\n");
    return;
  }
  // Specialize BlockReduce for a 1D block of 128 threads on type int
  typedef hipcub::BlockReduce<T, Nthreads> BlockReduce;
  // Allocate shared memory for BlockReduce
  __shared__ typename BlockReduce::TempStorage temp_storage;

  T idata = 0;
  T xdata = 0;
  T ydata = 0;
  // load shared mem
  unsigned int tid = threadIdx.x;
  unsigned int xvalid = validx[blockIdx.x];
  unsigned int yvalid = validy[blockIdx.x];
  unsigned int x, y;
  int idim;

  for (int cc = 0; cc < nelem_thread; cc++) {
    x = ((tid * nelem_thread + cc) % npix);
    y = ((tid * nelem_thread + cc) / npix);
    // idim = tid * nelem_thread + cc + (blockDim.x * nelem_thread) *
    // blockIdx.x;
    idim = (x + xvalid) + (y + yvalid) * size;
    if (idim < size * size) {
      idata += d_img[idim];
      xdata += d_img[idim] * (T)x;
      ydata += d_img[idim] * (T)y;
    }
  }

  // sdata[tid] = (i < N) ? g_idata[i] * x : 0;
  __syncthreads();

  T intensity = BlockReduce(temp_storage).Sum(idata, blockDim.x);
  T slopex = BlockReduce(temp_storage).Sum(xdata, blockDim.x);
  T slopey = BlockReduce(temp_storage).Sum(ydata, blockDim.x);
  // write result for this block to global mem
  if (tid == 0) {
    d_centroids[blockIdx.x] =
        ((slopex * (T)1.0 / (intensity + (T)1.e-6)) - offset) * scale -
        ref[blockIdx.x];
    d_centroids[blockIdx.x + gridDim.x] =
        ((slopey * (T)1.0 / (intensity + (T)1.e-6)) - offset) * scale -
        ref[blockIdx.x + gridDim.x];
    d_intensities[blockIdx.x] = intensity;
  }
}

template <class T>
void get_centroids(int size, int threads, int blocks, int npix, T *d_img,
                   T *d_centroids, T *ref, int *validx, int *validy,
                   T *intensities, T scale, T offset, carma_device *device) {
  int maxThreads = device->get_properties().maxThreadsPerBlock;
  unsigned int nelem_thread = 1;
  while ((threads / nelem_thread > maxThreads) ||
         (threads % nelem_thread != 0)) {
    nelem_thread++;
  }

  threads /= nelem_thread;
  dim3 dimBlock(threads, 1, 1);
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  if (threads <= 16)
    centroids<T, 16><<<dimGrid, dimBlock>>>(d_img, d_centroids, ref, validx,
                                            validy, intensities, npix, size,
                                            scale, offset, nelem_thread);
  else if (threads <= 32)
    centroids<T, 32><<<dimGrid, dimBlock>>>(d_img, d_centroids, ref, validx,
                                            validy, intensities, npix, size,
                                            scale, offset, nelem_thread);

  else if (threads <= 64)
    centroids<T, 64><<<dimGrid, dimBlock>>>(d_img, d_centroids, ref, validx,
                                            validy, intensities, npix, size,
                                            scale, offset, nelem_thread);
  else if (threads <= 128)
    centroids<T, 128><<<dimGrid, dimBlock>>>(d_img, d_centroids, ref, validx,
                                             validy, intensities, npix, size,
                                             scale, offset, nelem_thread);
  else if (threads <= 256)
    centroids<T, 256><<<dimGrid, dimBlock>>>(d_img, d_centroids, ref, validx,
                                             validy, intensities, npix, size,
                                             scale, offset, nelem_thread);
  else if (threads <= 512)
    centroids<T, 512><<<dimGrid, dimBlock>>>(d_img, d_centroids, ref, validx,
                                             validy, intensities, npix, size,
                                             scale, offset, nelem_thread);
  else
    printf("SH way too big !!!\n");

  carmaCheckMsg("centroids_kernel<<<>>> execution failed\n");

  //   centroidy<T><<<dimGrid, dimBlock, smemSize>>>(
  //       d_idata, &(d_odata[blocks]), alpha, n, size, scale, offset,
  //       nelem_thread);

  //   carmaCheckMsg("centroidy_kernel<<<>>> execution failed\n");
}

template void get_centroids<float>(int size, int threads, int blocks, int npix,
                                   float *d_img, float *d_centroids, float *ref,
                                   int *validx, int *validy, float *intensities,
                                   float scale, float offset,
                                   carma_device *device);

template void get_centroids<double>(int size, int threads, int blocks, int npix,
                                    double *d_img, double *d_centroids,
                                    double *ref, int *validx, int *validy,
                                    double *intensities, double scale,
                                    double offset, carma_device *device);

#ifdef CAN_DO_HALF
template void get_centroids<half>(int size, int threads, int blocks, int npix,
                                  half *d_img, half *d_centroids, half *ref,
                                  int *validx, int *validy, half *intensities,
                                  half scale, half offset,
                                  carma_device *device);
#endif

// template <class T>
// __global__ void centroidx(T *g_idata, T *g_odata, T *alpha, unsigned int n,
//                           unsigned int N, T scale, T offset,
//                           unsigned int nelem_thread) {
//   T *sdata = SharedMemory<T>();

//   // load shared mem
//   unsigned int tid = threadIdx.x;
//   // unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
//   // unsigned int x = (tid % n) + 1;
//   unsigned int x;
//   int idim;
//   sdata[tid] = 0;
//   for (int cc = 0; cc < nelem_thread; cc++) {
//     x = ((tid * nelem_thread + cc) % n);
//     idim = tid * nelem_thread + cc + (blockDim.x * nelem_thread) *
//     blockIdx.x; if (idim < N)
//       sdata[tid] += g_idata[idim] * x;
//     else
//       sdata[tid] += 0;
//   }

//   // sdata[tid] = (i < N) ? g_idata[i] * x : 0;

//   __syncthreads();

//   reduce_krnl(sdata, blockDim.x, tid);

//   // write result for this block to global mem
//   if (tid == 0)
//     g_odata[blockIdx.x] =
//         ((sdata[0] * 1.0 / (alpha[blockIdx.x] + 1.e-6)) - offset) * scale;
// }

// template <class T>
// __global__ void centroidy(T *g_idata, T *g_odata, T *alpha, unsigned int n,
//                           unsigned int N, T scale, T offset,
//                           unsigned int nelem_thread) {
//   T *sdata = SharedMemory<T>();

//   // load shared mem
//   unsigned int tid = threadIdx.x;
//   // unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
//   // unsigned int y = (tid / n) + 1;
//   unsigned int y;
//   int idim;
//   sdata[tid] = 0;
//   for (int cc = 0; cc < nelem_thread; cc++) {
//     y = ((tid * nelem_thread + cc) / n);
//     idim = tid * nelem_thread + cc + (blockDim.x * nelem_thread) *
//     blockIdx.x; if (idim < N)
//       sdata[tid] += g_idata[idim] * y;
//     else
//       sdata[tid] += 0;
//   }

//   // sdata[tid] = (i < N) ? g_idata[i] * y : 0;

//   __syncthreads();

//   reduce_krnl(sdata, blockDim.x, tid);

//   // write result for this block to global mem
//   if (tid == 0)
//     g_odata[blockIdx.x] =
//         ((sdata[0] * 1.0 / (alpha[blockIdx.x] + 1.e-6)) - offset) * scale;
// }

// template <class T>
// __global__ void centroidx_async(T *g_idata, T *g_odata, T *alpha,
//                                 unsigned int n, unsigned int N, T scale,
//                                 T offset, int stream_offset) {
//   T *sdata = SharedMemory<T>();

//   // load shared mem
//   unsigned int tid = threadIdx.x;
//   unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
//   unsigned int x = (tid % n) + 1;
//   i += stream_offset * blockDim.x;

//   sdata[tid] = (i < N) ? g_idata[i] * x : 0;

//   __syncthreads();

//   reduce_krnl(sdata, blockDim.x, tid);

//   // write result for this block to global mem
//   if (tid == 0)
//     g_odata[blockIdx.x + stream_offset] =
//         ((sdata[0] * 1.0 / (alpha[blockIdx.x + stream_offset] + 1.e-6)) -
//          offset) *
//         scale;
// }

// template <class T>
// __global__ void centroidy_async(T *g_idata, T *g_odata, T *alpha,
//                                 unsigned int n, unsigned int N, T scale,
//                                 T offset, int stream_offset) {
//   T *sdata = SharedMemory<T>();

//   // load shared mem
//   unsigned int tid = threadIdx.x;
//   unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
//   unsigned int y = (tid / n) + 1;
//   i += stream_offset * blockDim.x;

//   sdata[tid] = (i < N) ? g_idata[i] * y : 0;

//   __syncthreads();

//   reduce_krnl(sdata, blockDim.x, tid);

//   // write result for this block to global mem
//   if (tid == 0)
//     g_odata[blockIdx.x + stream_offset] =
//         ((sdata[0] * 1.0 / (alpha[blockIdx.x + stream_offset] + 1.e-6)) -
//          offset) *
//         scale;
// }

// template <class T>
// void get_centroids_async(int threads, int blocks, int n, carma_streams
// *streams,
//                          T *d_idata, T *d_odata, T *alpha, T scale, T offset)
//                          {
//   int nstreams = streams->get_nbStreams();
//   int nbelem = threads * blocks;

//   dim3 dimBlock(threads);
//   dim3 dimGrid(blocks / nstreams);

//   // when there is only one warp per block, we need to allocate two warps
//   // worth of shared memory so that we don't index shared memory out of
//   bounds int smemSize =
//       (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);
//   for (int i = 0; i < nstreams; i++) {
//     centroidx_async<T><<<dimGrid, dimBlock, smemSize,
//     streams->get_stream(i)>>>(
//         d_idata, d_odata, alpha, n, nbelem, scale, offset,
//         i * blocks / nstreams);

//     carmaCheckMsg("centroidx_kernel<<<>>> execution failed\n");

//     centroidy_async<T><<<dimGrid, dimBlock, smemSize,
//     streams->get_stream(i)>>>(
//         d_idata, &(d_odata[blocks]), alpha, n, nbelem, scale, offset,
//         i * blocks / nstreams);

//     carmaCheckMsg("centroidy_kernel<<<>>> execution failed\n");
//   }
// }

// template void get_centroids_async<float>(int threads, int blocks, int n,
//                                          carma_streams *streams, float
//                                          *d_idata, float *d_odata, float
//                                          *alpha, float scale, float offset);
// template void get_centroids_async<double>(int threads, int blocks, int n,
//                                           carma_streams *streams,
//                                           double *d_idata, double *d_odata,
//                                           double *alpha, double scale,
//                                           double offset);
