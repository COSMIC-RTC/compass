#include "hip/hip_runtime.h"
#include <sutra_ao_utils.h>

unsigned int
nextPow2(unsigned int x) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}

__global__ void
cfillrealp_krnl(hipFloatComplex *odata, float *idata, int N) {

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    odata[tid].x = idata[tid];
    tid += blockDim.x * gridDim.x;
  }
}

int
cfillrealp(hipFloatComplex *d_odata, float *d_idata, int N, int device) {

  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);

  dim3 grid(nblocks), threads(nthreads);

  cfillrealp_krnl<<<grid, threads>>>(d_odata, d_idata, N);

  cutilCheckMsg("cfillrealp_kernel<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}

__global__ void
cgetrealp_krnl(float *odata, hipFloatComplex *idata, int N) {

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    odata[tid] = idata[tid].x;
    tid += blockDim.x * gridDim.x;
  }
}

int
cgetrealp(float *d_odata, hipFloatComplex *d_idata, int N, int device) {

  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);

  dim3 grid(nblocks), threads(nthreads);

  cgetrealp_krnl<<<grid, threads>>>(d_odata, d_idata, N);

  cutilCheckMsg("cgetrealp_kernel<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}

__global__ void
abs2_krnl(float *odata, hipFloatComplex *idata, int N) {
  __shared__ hipFloatComplex cache;

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if (tid < N) {
    cache = idata[tid];
    odata[tid] = cache.x * cache.x + cache.y * cache.y;
  }
}

int
abs2(float *d_odata, hipFloatComplex *d_idata, int N, int device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);

  dim3 grid(nblocks), threads(nthreads);

  abs2_krnl<<<grid, threads>>>(d_odata, d_idata, N);
  cutilCheckMsg("abs2_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void
abs2c_krnl(hipFloatComplex *odata, hipFloatComplex *idata, int N) {
  __shared__ hipFloatComplex cache;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  if (tid < N) {
    cache = idata[tid];
    odata[tid].x = cache.x * cache.x + cache.y * cache.y;
    odata[tid].y = 0.0;
  }
}

int
abs2c(hipFloatComplex *d_odata, hipFloatComplex *d_idata, int N, int device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);

  dim3 grid(nblocks), threads(nthreads);

  abs2c_krnl<<<grid, threads>>>(d_odata, d_idata, N);
  cutilCheckMsg("abs2c_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void
subapnorm_krnl(float *odata, float *idata, float *fact, float *norm,
    float nphot, int n, int N) {

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    odata[tid] = idata[tid] * fact[tid / n] / norm[tid / n] * nphot;
    tid += blockDim.x * gridDim.x;
  }
}

int
subap_norm(float *d_odata, float *d_idata, float *fact, float *norm,
    float nphot, int n, int N, int device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);

  dim3 grid(nblocks), threads(nthreads);

  subapnorm_krnl<<<grid, threads>>>(d_odata, d_idata, fact, norm, nphot, n, N);
  cutilCheckMsg("subapnorm_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void
subapnormasync_krnl(float *odata, float *idata, float *fact, float *norm,
    float nphot, int n, int N, int istart) {

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  tid += istart;
  if (tid < N) {
    odata[tid] = idata[tid] * fact[tid / n] / norm[tid / n] * nphot;
  }
}

int
subap_norm_async(float *d_odata, float *d_idata, float *fact, float *norm,
    float nphot, int n, int N, carma_streams *streams, int device) {
  int nthreads = 0, nblocks = 0;
  int nstreams = streams->get_nbStreams();
  getNumBlocksAndThreads(device, N / nstreams, nblocks, nthreads);

  dim3 grid(nblocks), threads(nthreads);

  for (int i = 0; i < nstreams; i++) {
    subapnormasync_krnl<<<grid, threads, 0, streams->get_stream(i)>>>(d_odata,
        d_idata, fact, norm, nphot, n, N, i * nblocks * nthreads);
    cutilCheckMsg("subapnormasync_kernel<<<>>> execution failed\n");
  }

  return EXIT_SUCCESS;
}

__global__ void
krnl_fillindx(float *odata, float *idata, int *indx, float alpha, float beta,
    int N) {

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    odata[tid] = (alpha * idata[indx[tid]]) + beta;
    tid += blockDim.x * gridDim.x;
  }
}

int
fillindx(float *d_odata, float *d_idata, int *indx, float alpha, float beta,
    int N, int device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);

  dim3 grid(nblocks), threads(nthreads);

  krnl_fillindx<<<grid, threads>>>(d_odata, d_idata, indx, alpha, beta, N);

  cutilCheckMsg("fillindx_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

int
fillindx(float *d_odata, float *d_idata, int *indx, int N, int device) {
  return fillindx(d_odata, d_idata, indx, 1.0f, 0.0f, N, device);

}
int
fillindx(float *d_odata, float *d_idata, int *indx, float alpha, int N,
    int device) {
  return fillindx(d_odata, d_idata, indx, alpha, 0.0f, N, device);

}
__global__ void
fillarr2d_krnl(float *odata, float *idata, int tidx0, int Ncol, int NC, int N) {

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int tidB;

  while (tid < N) {
    if (Ncol > 1)
      tidB = tidx0 + (tid / Ncol) * NC + (tid % Ncol);
    else
      tidB = tidx0 + tid * NC;
    odata[tidB] = idata[tid];
    tid += blockDim.x * gridDim.x;
  }
}

int
fillarr2d(float *d_odata, float *d_idata, int x0, int Ncol, int NC, int N,
    int device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);

  dim3 grid(nblocks), threads(nthreads);

  fillarr2d_krnl<<<grid, threads>>>(d_odata, d_idata, x0, Ncol, NC, N);

  cutilCheckMsg("fillarr2d_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void
getarr2d_krnl(float *odata, float *idata, int tidx0, int Ncol, int NC, int N) {

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int tidB;

  while (tid < N) {
    if (Ncol > 1)
      tidB = tidx0 + (tid / Ncol) * NC + (tid % Ncol);
    else
      tidB = tidx0 + tid * NC;
    odata[tid] = idata[tidB];
    tid += blockDim.x * gridDim.x;
  }
}

int
getarr2d(float *d_odata, float *d_idata, int x0, int Ncol, int NC, int N,
    int device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);

  dim3 grid(nblocks), threads(nthreads);

  getarr2d_krnl<<<grid, threads>>>(d_odata, d_idata, x0, Ncol, NC, N);

  cutilCheckMsg("getarr2d_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void
addai_krnl(float *odata, float* idata, int i, int sgn, int N) {

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    if (sgn == 1)
      odata[tid] += idata[i];
    else
      odata[tid] -= idata[i];
    tid += blockDim.x * gridDim.x;
  }
}

int
addai(float *d_odata, float *i_data, int i, int sgn, int N, int device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);

  dim3 grid(nblocks), threads(nthreads);

  addai_krnl<<<grid, threads>>>(d_odata, i_data, i, sgn, N);

  cutilCheckMsg("plusai_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

template<class T>
  __global__ void
  roll_krnl(T *idata, int N, int M, int Ntot) {

    int tidt = threadIdx.x + blockIdx.x * blockDim.x;
    int nim = tidt / Ntot;

    int tid = tidt - nim * Ntot;

    while (tid < Ntot) {

      int x = tid % N;
      int y = tid / N;

      int xx = (x + N / 2) % N;
      int yy = (y + M / 2) % M;
      int tid2 = xx + yy * N;

      __shared__ T tmp;
      tmp = idata[tid + nim * (N * M)];
      idata[tid + nim * (N * M)] = idata[tid2 + nim * (N * M)];
      idata[tid2 + nim * (N * M)] = tmp;

      tid += blockDim.x * gridDim.x;
    }
  }

template<class T>
  int
  roll(T *idata, int N, int M, int nim) {

    struct hipDeviceProp_t deviceProperties;
    hipGetDeviceProperties(&deviceProperties, 0);

    long Ntot = N * M * nim;
    int maxThreads = deviceProperties.maxThreadsPerBlock;
    int nBlocks = deviceProperties.multiProcessorCount * 8;
    int nThreads = (Ntot / 2 + nBlocks - 1) / nBlocks;

    if (nThreads > maxThreads) {
      nThreads = maxThreads;
      nBlocks = (Ntot / 2 + nThreads - 1) / nThreads;
    }

    dim3 grid(nBlocks), threads(nThreads);

    roll_krnl<<<grid, threads>>>(idata, N, M, Ntot / 2);

    cutilCheckMsg("roll_kernel<<<>>> execution failed\n");
    return EXIT_SUCCESS;

  }

template int
roll<float>(float *idata, int N, int M, int nim);

template int
roll<double>(double *idata, int N, int M, int nim);

template int
roll<hipFloatComplex>(hipFloatComplex *idata, int N, int M, int nim);

template<class T>
  __global__ void
  roll_krnl(T *idata, int N, int M) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < (N * M / 2)) {
      int x = tid % N;
      int y = tid / N;

      int xx = (x + N / 2) % N;
      int yy = (y + M / 2) % M;
      int tid2 = xx + yy * N;

      __shared__ T tmp;
      tmp = idata[tid];
      idata[tid] = idata[tid2];
      idata[tid2] = tmp;

      tid += blockDim.x * gridDim.x;
    }
  }

template<class T>
  int
  roll(T *idata, int N, int M) {

    struct hipDeviceProp_t deviceProperties;
    hipGetDeviceProperties(&deviceProperties, 0);

    long Ntot = N * M;
    int maxThreads = deviceProperties.maxThreadsPerBlock;
    int nBlocks = deviceProperties.multiProcessorCount * 8;
    int nThreads = (Ntot / 2 + nBlocks - 1) / nBlocks;

    if (nThreads > maxThreads) {
      nThreads = maxThreads;
      nBlocks = (Ntot / 2 + nThreads - 1) / nThreads;
    }

    dim3 grid(nBlocks), threads(nThreads);

    roll_krnl<<<grid, threads>>>(idata, N, M);

    cutilCheckMsg("roll_kernel<<<>>> execution failed\n");
    return EXIT_SUCCESS;

  }

template int
roll<float>(float *idata, int N, int M);

template int
roll<double>(double *idata, int N, int M);

template int
roll<hipFloatComplex>(hipFloatComplex *idata, int N, int M);

