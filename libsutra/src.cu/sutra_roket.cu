#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
//  This file is part of COMPASS <https://anr-compass.github.io/compass/>
//
//  Copyright (C) 2011-2023 COMPASS Team <https://github.com/ANR-COMPASS>
//  All rights reserved.

// -----------------------------------------------------------------------------

//! \file      sutra_roket.cu
//! \ingroup   libsutra
//! \class     SutraRoket
//! \brief     this class provides the roket features to COMPASS
//! \author    COMPASS Team <https://github.com/ANR-COMPASS>
//! \version   5.5.0
//! \date      2022/01/24

#include <sutra_roket.h>

__global__ void separate_modes_krnl(float *modes, float *filtmodes, int nmodes,
                                    int nfilt) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int bornemin = nmodes - nfilt - 2;
  int bornemax = nmodes - 2;
  while (tid < nmodes) {
    if (tid >= bornemin && tid < bornemax) {
      filtmodes[tid] = modes[tid];
      modes[tid] = 0.0f;
    } else
      filtmodes[tid] = 0.0f;

    tid += blockDim.x * gridDim.x;
  }
}

int separate_modes(float *modes, float *filtmodes, int nmodes, int nfilt,
                   CarmaDevice *device) {
  int nb_threads = 0, nb_blocks = 0;
  get_num_blocks_and_threads(device, nmodes, nb_blocks, nb_threads);
  dim3 grid(nb_blocks), threads(nb_threads);

  separate_modes_krnl<<<grid, threads>>>(modes, filtmodes, nmodes, nfilt);
  carma_check_msg("separate_modes_krnl<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}
