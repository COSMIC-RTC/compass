#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
//  This file is part of COMPASS <https://anr-compass.github.io/compass/>
//
//  Copyright (C) 2011-2019 COMPASS Team <https://github.com/ANR-COMPASS>
//  All rights reserved.
//  Distributed under GNU - LGPL
//
//  COMPASS is free software: you can redistribute it and/or modify it under the terms of the GNU Lesser
//  General Public License as published by the Free Software Foundation, either version 3 of the License,
//  or any later version.
//
//  COMPASS: End-to-end AO simulation tool using GPU acceleration
//  The COMPASS platform was designed to meet the need of high-performance for the simulation of AO systems.
//
//  The final product includes a software package for simulating all the critical subcomponents of AO,
//  particularly in the context of the ELT and a real-time core based on several control approaches,
//  with performances consistent with its integration into an instrument. Taking advantage of the specific
//  hardware architecture of the GPU, the COMPASS tool allows to achieve adequate execution speeds to
//  conduct large simulation campaigns called to the ELT.
//
//  The COMPASS platform can be used to carry a wide variety of simulations to both testspecific components
//  of AO of the E-ELT (such as wavefront analysis device with a pyramid or elongated Laser star), and
//  various systems configurations such as multi-conjugate AO.
//
//  COMPASS is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the
//  implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
//  See the GNU Lesser General Public License for more details.
//
//  You should have received a copy of the GNU Lesser General Public License along with COMPASS.
//  If not, see <https://www.gnu.org/licenses/lgpl-3.0.txt>.
// -----------------------------------------------------------------------------

//! \file      sutra_roket.cu
//! \ingroup   libsutra
//! \class     sutra_roket
//! \brief     this class provides the roket features to COMPASS
//! \author    COMPASS Team <https://github.com/ANR-COMPASS>
//! \version   4.4.0
//! \date      2011/01/28
//! \copyright GNU Lesser General Public License

#include <sutra_roket.h>

__global__ void separate_modes_krnl(float *modes, float *filtmodes, int nmodes,
                                    int nfilt) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int bornemin = nmodes - nfilt - 2;
  int bornemax = nmodes - 2;
  while (tid < nmodes) {
    if (tid >= bornemin && tid < bornemax) {
      filtmodes[tid] = modes[tid];
      modes[tid] = 0.0f;
    } else
      filtmodes[tid] = 0.0f;

    tid += blockDim.x * gridDim.x;
  }
}

int separate_modes(float *modes, float *filtmodes, int nmodes, int nfilt,
                   carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, nmodes, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  separate_modes_krnl<<<grid, threads>>>(modes, filtmodes, nmodes, nfilt);
  carmaCheckMsg("separate_modes_krnl<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}
