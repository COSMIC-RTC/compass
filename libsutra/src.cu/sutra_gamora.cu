#include "hip/hip_runtime.h"
#include <sutra_gamora.h>

__global__ void fillamplikrnl(hipFloatComplex *amplipup, float *phase, int *wherephase,
                              float scale, int Npts, int nx, int Nx, int puponly) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int nim;
  int nline;
  int ncol;

  while (tid < Npts) {
    nim = wherephase[tid];
    nline = nim / nx;
    ncol = nim - nline*nx;
    nim = ncol + nline*Nx;
    if(puponly == 1) {
      amplipup[nim].x = 1.0f;
      amplipup[nim].y = 0.0f;
    } else if(puponly == 0) {
      amplipup[nim].x = cosf(-scale * phase[tid]);
      amplipup[nim].y = sinf(-scale * phase[tid]);
    } else if(puponly == 2) {
      amplipup[nim].x = phase[tid];
      amplipup[nim].y = 0.0f;

    }
    tid += blockDim.x * gridDim.x;
  }
}

int fill_amplipup(hipFloatComplex *amplipup, float *phase, int *wherephase,
                  float scale, int Npts, int nx, int Nx, int puponly, carma_device *device) {

  int nBlocks,nThreads;
  getNumBlocksAndThreads(device, Npts, nBlocks, nThreads);
  dim3 grid(nBlocks), threads(nThreads);

  fillamplikrnl<<<grid, threads>>>(amplipup, phase, wherephase, scale, Npts, nx, Nx, puponly);
  carmaCheckMsg("fillamplikrnl<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void cumulpsf_krnl(float *odata, hipFloatComplex *idata, int N) {
  hipFloatComplex cache;

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    cache = idata[tid];
    odata[tid] += (cache.x * cache.x + cache.y * cache.y);
    tid += blockDim.x * gridDim.x;
  }
}

int cumulpsf(float *d_odata, hipFloatComplex *d_idata, int N, carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  cumulpsf_krnl<<<grid, threads>>>(d_odata, d_idata, N);
  carmaCheckMsg("cumulpsf_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void abs2complex_krnl(hipFloatComplex *d_odata, hipFloatComplex *d_idata, int N) {
  hipFloatComplex idata;
  hipFloatComplex odata;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while(tid < N) {
    idata = d_idata[tid];
    odata.x = idata.x * idata.x + idata.y * idata.y;
    odata.y = 0.0f;
    d_odata[tid] = odata;
    tid += blockDim.x * gridDim.x;
  }
}
int abs2complex(hipFloatComplex *d_odata, hipFloatComplex *d_idata, int N, carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  abs2complex_krnl<<<grid, threads>>>(d_odata, d_idata, N);
  carmaCheckMsg("abs2complex_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void modulus2_krnl(float *d_odata, hipFloatComplex *d_idata, int N) {
  hipFloatComplex idata;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while(tid < N) {
    idata = d_idata[tid];
    d_odata[tid] = idata.x * idata.x + idata.y * idata.y;
    tid += blockDim.x * gridDim.x;
  }
}
int modulus2(float *d_odata, hipFloatComplex *d_idata, int N, carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  modulus2_krnl<<<grid, threads>>>(d_odata, d_idata, N);
  carmaCheckMsg("modulus2_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void real_krnl(float *d_odata, hipFloatComplex *d_idata, int N) {
  hipFloatComplex cache;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while(tid < N) {
    cache = d_idata[tid];
    d_odata[tid] = cache.x;
    tid += blockDim.x * gridDim.x;
  }
}

int real(float *d_odata, hipFloatComplex *d_idata, int N, carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  real_krnl<<<grid, threads>>>(d_odata, d_idata, N);
  carmaCheckMsg("real_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void fillmask_krnl(float *d_odata, float *d_idata, int N, int norm) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while(tid < N) {
    d_odata[tid] = d_idata[tid] < (1e-5*norm) ? 0.0f : 1.0f;
    tid += blockDim.x * gridDim.x;
  }
}

int fill_mask(float *d_odata, float *d_idata, int N, int norm, carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  fillmask_krnl<<<grid, threads>>>(d_odata, d_idata, N, norm);
  carmaCheckMsg("fillmask_krnl<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void pow2_krnl(hipFloatComplex *d_odata, hipFloatComplex *d_idata, int N) {
  hipFloatComplex idata;
  hipFloatComplex odata;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while(tid < N) {
    idata = d_idata[tid];
    odata.x = idata.x * idata.x;
    odata.y = 0.0f;
    d_odata[tid] = odata;
    tid += blockDim.x * gridDim.x;
  }
}
int pow2(hipFloatComplex *d_odata, hipFloatComplex *d_idata, int N, carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  pow2_krnl<<<grid, threads>>>(d_odata, d_idata, N);
  carmaCheckMsg("pow2_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void fillterm1_krnl(float *d_odata, hipFloatComplex *d_idata,  hipFloatComplex *d_pupfft, int N) {
  hipFloatComplex idata;
  hipFloatComplex pupfft;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while(tid < N) {
    idata = d_idata[tid];
    pupfft = d_pupfft[tid];
    d_odata[tid] = idata.x * pupfft.x + idata.y * pupfft.y;
    tid += blockDim.x * gridDim.x;
  }
}

int fill_term1(float *d_odata, hipFloatComplex *d_idata, hipFloatComplex *d_pupfft, int N, carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  fillterm1_krnl<<<grid, threads>>>(d_odata, d_idata, d_pupfft, N);
  carmaCheckMsg("fillterm1_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void add2Dphi_krnl(hipFloatComplex *d_odata, float *d_term1,  float *d_term2, float e, int N) {
  hipFloatComplex cache;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while(tid < N) {
    cache = d_odata[tid];
    cache.x += 2*((d_term1[tid] - d_term2[tid]) * e);
    cache.y = 0.0f;
    d_odata[tid] = cache;
    tid += blockDim.x * gridDim.x;
  }
}

int add2Dphi(hipFloatComplex *d_odata, float *d_term1, float *d_term2, float e, int N, carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  add2Dphi_krnl<<<grid, threads>>>(d_odata, d_term1, d_term2, e, N);
  carmaCheckMsg("add2Dphi_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void computeOTFvii_krnl(float *d_otfVii, hipFloatComplex * d_Dphi, float *d_otftel,
                                   float *d_mask, float scale, int N) {

  hipFloatComplex Dphi;
  float tmp;
  float den;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while(tid < N) {
    Dphi = d_Dphi[tid];
    den = d_otftel[tid] > 1e-9 ? 1.0f/d_otftel[tid] : 0.0f;
    tmp = Dphi.x * den * d_mask[tid] * scale * scale;
    d_otfVii[tid] = expf(-0.5 * tmp) * d_mask[tid];

    tid += blockDim.x * gridDim.x;
  }
}
int computeOTFvii(float *d_otfVii, hipFloatComplex * d_Dphi, float *d_otftel, float *d_mask,
                  float scale, int N, carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  computeOTFvii_krnl<<<grid, threads>>>(d_otfVii, d_Dphi, d_otftel, d_mask,scale, N);
  carmaCheckMsg("computeOTFvii_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void ifftscale_krnl(hipFloatComplex *d_odata, float scale, int N) {
  hipFloatComplex cache;
  int tid = threadIdx.x + blockIdx.x*blockDim.x;

  while(tid < N) {
    cache = d_odata[tid];
    cache.x = cache.x * scale;
    cache.y = cache.y * scale;
    d_odata[tid] = cache;
    tid += blockDim.x * gridDim.x;
  }
}
int ifftscale(hipFloatComplex *d_odata, float scale, int N, carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  ifftscale_krnl<<<grid, threads>>>(d_odata,scale, N);
  carmaCheckMsg("iffscale_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}
