#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
//  This file is part of COMPASS <https://anr-compass.github.io/compass/>
//
//  Copyright (C) 2011-2019 COMPASS Team <https://github.com/ANR-COMPASS>
//  All rights reserved.
//  Distributed under GNU - LGPL
//
//  COMPASS is free software: you can redistribute it and/or modify it under the terms of the GNU Lesser 
//  General Public License as published by the Free Software Foundation, either version 3 of the License, 
//  or any later version.
//
//  COMPASS: End-to-end AO simulation tool using GPU acceleration 
//  The COMPASS platform was designed to meet the need of high-performance for the simulation of AO systems. 
//  
//  The final product includes a software package for simulating all the critical subcomponents of AO, 
//  particularly in the context of the ELT and a real-time core based on several control approaches, 
//  with performances consistent with its integration into an instrument. Taking advantage of the specific 
//  hardware architecture of the GPU, the COMPASS tool allows to achieve adequate execution speeds to
//  conduct large simulation campaigns called to the ELT. 
//  
//  The COMPASS platform can be used to carry a wide variety of simulations to both testspecific components 
//  of AO of the E-ELT (such as wavefront analysis device with a pyramid or elongated Laser star), and 
//  various systems configurations such as multi-conjugate AO.
//
//  COMPASS is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the 
//  implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  
//  See the GNU Lesser General Public License for more details.
//
//  You should have received a copy of the GNU Lesser General Public License along with COMPASS. 
//  If not, see <https://www.gnu.org/licenses/lgpl-3.0.txt>.
// -----------------------------------------------------------------------------

//! \file      sutra_gamora.cu
//! \ingroup   libsutra
//! \class     sutra_gamora
//! \brief     this class provides the gamora features to COMPASS
//! \author    COMPASS Team <https://github.com/ANR-COMPASS>
//! \version   4.3.1
//! \date      2011/01/28
//! \copyright GNU Lesser General Public License

#include <sutra_gamora.h>

__global__ void fillamplikrnl(hipFloatComplex *amplipup, float *phase,
                              int *wherephase, float scale, int Npts, int nx,
                              int Nx, int puponly) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int nim;
  int nline;
  int ncol;

  while (tid < Npts) {
    nim = wherephase[tid];
    nline = nim / nx;
    ncol = nim - nline * nx;
    nim = ncol + nline * Nx;
    if (puponly == 1) {
      amplipup[nim].x = 1.0f;
      amplipup[nim].y = 0.0f;
    } else if (puponly == 0) {
      amplipup[nim].x = cosf(scale * phase[tid]);
      amplipup[nim].y = sinf(scale * phase[tid]);
    } else if (puponly == 2) {
      amplipup[nim].x = phase[tid];
      amplipup[nim].y = 0.0f;
    }
    tid += blockDim.x * gridDim.x;
  }
}

int fill_amplipup(hipFloatComplex *amplipup, float *phase, int *wherephase,
                  float scale, int Npts, int nx, int Nx, int puponly,
                  carma_device *device) {
  int nBlocks, nThreads;
  getNumBlocksAndThreads(device, Npts, nBlocks, nThreads);
  dim3 grid(nBlocks), threads(nThreads);

  fillamplikrnl<<<grid, threads>>>(amplipup, phase, wherephase, scale, Npts, nx,
                                   Nx, puponly);
  carmaCheckMsg("fillamplikrnl<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void cumulpsf_krnl(float *odata, hipFloatComplex *idata, int N) {
  hipFloatComplex cache;

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    cache = idata[tid];
    odata[tid] += (cache.x * cache.x + cache.y * cache.y);
    tid += blockDim.x * gridDim.x;
  }
}

int cumulpsf(float *d_odata, hipFloatComplex *d_idata, int N,
             carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  cumulpsf_krnl<<<grid, threads>>>(d_odata, d_idata, N);
  carmaCheckMsg("cumulpsf_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void abs2complex_krnl(hipFloatComplex *d_odata,
                                 hipFloatComplex *d_idata, int N) {
  hipFloatComplex idata;
  hipFloatComplex odata;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    idata = d_idata[tid];
    odata.x = idata.x * idata.x + idata.y * idata.y;
    odata.y = 0.0f;
    d_odata[tid] = odata;
    tid += blockDim.x * gridDim.x;
  }
}
int abs2complex(hipFloatComplex *d_odata, hipFloatComplex *d_idata, int N,
                carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  abs2complex_krnl<<<grid, threads>>>(d_odata, d_idata, N);
  carmaCheckMsg("abs2complex_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void modulus2_krnl(float *d_odata, hipFloatComplex *d_idata, int N) {
  hipFloatComplex idata;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    idata = d_idata[tid];
    d_odata[tid] = idata.x * idata.x + idata.y * idata.y;
    tid += blockDim.x * gridDim.x;
  }
}
int modulus2(float *d_odata, hipFloatComplex *d_idata, int N,
             carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  modulus2_krnl<<<grid, threads>>>(d_odata, d_idata, N);
  carmaCheckMsg("modulus2_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void real_krnl(float *d_odata, hipFloatComplex *d_idata, int N) {
  hipFloatComplex cache;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    cache = d_idata[tid];
    d_odata[tid] = cache.x;
    tid += blockDim.x * gridDim.x;
  }
}

int real(float *d_odata, hipFloatComplex *d_idata, int N, carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  real_krnl<<<grid, threads>>>(d_odata, d_idata, N);
  carmaCheckMsg("real_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void fillmask_krnl(float *d_odata, float *d_idata, int N, int norm) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    d_odata[tid] = d_idata[tid] < (1e-5 * norm) ? 0.0f : 1.0f;
    tid += blockDim.x * gridDim.x;
  }
}

int fill_mask(float *d_odata, float *d_idata, int N, int norm,
              carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  fillmask_krnl<<<grid, threads>>>(d_odata, d_idata, N, norm);
  carmaCheckMsg("fillmask_krnl<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void pow2_krnl(hipFloatComplex *d_odata, hipFloatComplex *d_idata,
                          int N) {
  hipFloatComplex idata;
  hipFloatComplex odata;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    idata = d_idata[tid];
    odata.x = idata.x * idata.x;
    odata.y = 0.0f;
    d_odata[tid] = odata;
    tid += blockDim.x * gridDim.x;
  }
}
int pow2(hipFloatComplex *d_odata, hipFloatComplex *d_idata, int N,
         carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  pow2_krnl<<<grid, threads>>>(d_odata, d_idata, N);
  carmaCheckMsg("pow2_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void fillterm1_krnl(float *d_odata, hipFloatComplex *d_idata,
                               hipFloatComplex *d_pupfft, int N) {
  hipFloatComplex idata;
  hipFloatComplex pupfft;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    idata = d_idata[tid];
    pupfft = d_pupfft[tid];
    d_odata[tid] = idata.x * pupfft.x + idata.y * pupfft.y;
    tid += blockDim.x * gridDim.x;
  }
}

int fill_term1(float *d_odata, hipFloatComplex *d_idata,
               hipFloatComplex *d_pupfft, int N, carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  fillterm1_krnl<<<grid, threads>>>(d_odata, d_idata, d_pupfft, N);
  carmaCheckMsg("fillterm1_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void add2Dphi_krnl(hipFloatComplex *d_odata, float *d_term1,
                              float *d_term2, float e, int N) {
  hipFloatComplex cache;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    cache = d_odata[tid];
    cache.x += 2 * ((d_term1[tid] - d_term2[tid]) * e);
    cache.y = 0.0f;
    d_odata[tid] = cache;
    tid += blockDim.x * gridDim.x;
  }
}

int add2Dphi(hipFloatComplex *d_odata, float *d_term1, float *d_term2, float e,
             int N, carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  add2Dphi_krnl<<<grid, threads>>>(d_odata, d_term1, d_term2, e, N);
  carmaCheckMsg("add2Dphi_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void computeOTFvii_krnl(float *d_otfVii, hipFloatComplex *d_Dphi,
                                   float *d_otftel, float *d_mask, float scale,
                                   int N) {
  hipFloatComplex Dphi;
  float tmp;
  float den;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    Dphi = d_Dphi[tid];
    den = d_otftel[tid] > 1e-9 ? 1.0f / d_otftel[tid] : 0.0f;
    tmp = Dphi.x * den * d_mask[tid] * scale * scale;
    d_otfVii[tid] = expf(-0.5 * tmp) * d_mask[tid];

    tid += blockDim.x * gridDim.x;
  }
}
int computeOTFvii(float *d_otfVii, hipFloatComplex *d_Dphi, float *d_otftel,
                  float *d_mask, float scale, int N, carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  computeOTFvii_krnl<<<grid, threads>>>(d_otfVii, d_Dphi, d_otftel, d_mask,
                                        scale, N);
  carmaCheckMsg("computeOTFvii_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void ifftscale_krnl(hipFloatComplex *d_odata, float scale, int N) {
  hipFloatComplex cache;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    cache = d_odata[tid];
    cache.x = cache.x * scale;
    cache.y = cache.y * scale;
    d_odata[tid] = cache;
    tid += blockDim.x * gridDim.x;
  }
}
int ifftscale(hipFloatComplex *d_odata, float scale, int N,
              carma_device *device) {
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, N, nblocks, nthreads);
  dim3 grid(nblocks), threads(nthreads);

  ifftscale_krnl<<<grid, threads>>>(d_odata, scale, N);
  carmaCheckMsg("iffscale_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}
