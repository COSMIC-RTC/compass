#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
//  This file is part of COMPASS <https://anr-compass.github.io/compass/>
//
//  Copyright (C) 2011-2019 COMPASS Team <https://github.com/ANR-COMPASS>
//  All rights reserved.
//  Distributed under GNU - LGPL
//
//  COMPASS is free software: you can redistribute it and/or modify it under the terms of the GNU Lesser 
//  General Public License as published by the Free Software Foundation, either version 3 of the License, 
//  or any later version.
//
//  COMPASS: End-to-end AO simulation tool using GPU acceleration 
//  The COMPASS platform was designed to meet the need of high-performance for the simulation of AO systems. 
//  
//  The final product includes a software package for simulating all the critical subcomponents of AO, 
//  particularly in the context of the ELT and a real-time core based on several control approaches, 
//  with performances consistent with its integration into an instrument. Taking advantage of the specific 
//  hardware architecture of the GPU, the COMPASS tool allows to achieve adequate execution speeds to
//  conduct large simulation campaigns called to the ELT. 
//  
//  The COMPASS platform can be used to carry a wide variety of simulations to both testspecific components 
//  of AO of the E-ELT (such as wavefront analysis device with a pyramid or elongated Laser star), and 
//  various systems configurations such as multi-conjugate AO.
//
//  COMPASS is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the 
//  implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  
//  See the GNU Lesser General Public License for more details.
//
//  You should have received a copy of the GNU Lesser General Public License along with COMPASS. 
//  If not, see <https://www.gnu.org/licenses/lgpl-3.0.txt>.
// -----------------------------------------------------------------------------

//! \file      sutra_kl.cu
//! \ingroup   libsutra
//! \class     sutra_kl
//! \brief     this class provides the kl features to COMPASS
//! \author    COMPASS Team <https://github.com/ANR-COMPASS>
//! \version   4.3.1
//! \date      2011/01/28
//! \copyright GNU Lesser General Public License

#include <sutra_kl.h>

__device__ float kl_sfi(float *rabas, float *azbas, int npix, int nrow) {
  return rabas[npix] * azbas[nrow];
}

__device__ void kl_interp(float alpha, float ampli, float *odata, float *rabas,
                          float *azbas, float xbi, float ybi, int nr, int np,
                          int tido) {
  if ((xbi >= 0) && (xbi <= nr - 1) && (ybi >= 0) && (ybi <= np - 1)) {
    int i0, i1, j0, j1;
    long ibi = (long)xbi;
    long jbi = (long)ybi;

    i0 = ibi;
    if (ibi < nr - 1)
      i1 = ibi + 1;
    else
      i1 = nr - 1;

    j0 = jbi;
    if (jbi < np - 1)
      j1 = jbi + 1;
    else
      j1 = np - 1;

    float wi, wj, w00, w01, w10, w11;

    wi = 1.0f - (xbi - ibi);
    wj = 1.0f - (ybi - jbi);

    w00 = wi * wj;
    w10 = (1 - wi) * wj;
    w01 = wi * (1 - wj);
    w11 = (1 - wi) * (1 - wj);

    odata[tido] =
        alpha * odata[tido] + ampli * (w00 * kl_sfi(rabas, azbas, i0, j0) +
                                       w10 * kl_sfi(rabas, azbas, i1, j0) +
                                       w01 * kl_sfi(rabas, azbas, i0, j1) +
                                       w11 * kl_sfi(rabas, azbas, i1, j1));
  } else
    odata[tido] = 0.0f;
}

__device__ void kl_interp(float alpha, float *ampli, int nkl, float *odata,
                          float *rabas, float *azbas, float xbi, float ybi,
                          int nr, int np, int tido) {
  if ((xbi >= 0) && (xbi <= nr - 1) && (ybi >= 0) && (ybi <= np - 1)) {
    int i0, i1, j0, j1;
    long ibi = (long)xbi;
    long jbi = (long)ybi;

    i0 = ibi;
    if (ibi < nr - 1)
      i1 = ibi + 1;
    else
      i1 = nr - 1;

    j0 = jbi;
    if (jbi < np - 1)
      j1 = jbi + 1;
    else
      j1 = np - 1;

    float wi, wj, w00, w01, w10, w11;

    wi = 1.0f - (xbi - ibi);
    wj = 1.0f - (ybi - jbi);

    w00 = wi * wj;
    w10 = (1 - wi) * wj;
    w01 = wi * (1 - wj);
    w11 = (1 - wi) * (1 - wj);

    odata[tido] =
        alpha * odata[tido] + ampli[nkl] * (w00 * kl_sfi(rabas, azbas, i0, j0) +
                                            w10 * kl_sfi(rabas, azbas, i1, j0) +
                                            w01 * kl_sfi(rabas, azbas, i0, j1) +
                                            w11 * kl_sfi(rabas, azbas, i1, j1));
  } else
    odata[tido] = 0.0f;
}

__global__ void getkl_krnl(float alpha, float ampli, float *odata, float *rabas,
                           float *azbas, float *cr, float *cp, int nr, int np,
                           int nx, int Nx, int xoff, int yoff) {
  int xid = threadIdx.x + blockIdx.x * blockDim.x;
  int yid = threadIdx.y + blockIdx.y * blockDim.y;

  if ((xid < nx) && (yid < nx)) {
    int xref = xid + xoff;
    int yref = yid + yoff;
    int tido = xref + yref * Nx;

    int tid = xid + yid * nx;
    float xbi = cr[tid];
    float ybi = cp[tid];

    kl_interp(alpha, ampli, odata, rabas, azbas, xbi, ybi, nr, np, tido);
  }
}

__global__ void combikl_krnl(float *com, int nkl, float *odata, float *rabas,
                             int *d_ord, float *azbas, float *cr, float *cp,
                             int nr, int np, int nx, int Nx, int xoff,
                             int yoff) {
  int xid = threadIdx.x + blockIdx.x * blockDim.x;
  int yid = threadIdx.y + blockIdx.y * blockDim.y;

  if ((xid < nx) && (yid < nx)) {
    int xref = xid + xoff;
    int yref = yid + yoff;
    int tido = xref + yref * Nx;

    int tid = xid + yid * nx;
    float xbi = cr[tid];
    float ybi = cp[tid];

    odata[tido] = 0.0f;
    int tmp;
    float *rabas_cc;
    float *azbas_cc;
    // int cc=10;
    for (int cc = 0; cc < nkl; cc++) {
      tmp = d_ord[cc] - 1;
      rabas_cc = &(rabas[cc * nr]);
      azbas_cc = &(azbas[tmp * np]);
      kl_interp(1.0f, com, cc, odata, rabas_cc, azbas_cc, xbi, ybi, nr, np,
                tido);
    }
    __syncthreads();
  }
}

int getkl(float alpha, float ampli, float *d_odata, float *rabas, float *azbas,
          float *cr, float *cp, int nr, int np, int nx, int Nx, int xoff,
          int yoff) {
  int block_size = 8;
  int nnx =
      nx + block_size - nx % block_size;  // find next multiple of BLOCK_SZ
  int nny = nx + block_size - nx % block_size;
  dim3 blocks(nnx / block_size, nny / block_size),
      threads(block_size, block_size);

  // int smemSize = (block_size +1) * (block_size +1) * sizeof(float);

  getkl_krnl<<<blocks, threads>>>(alpha, ampli, d_odata, rabas, azbas, cr, cp,
                                  nr, np, nx, Nx, xoff, yoff);

  carmaCheckMsg("get_kernel<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}

int getkl(float ampli, float *d_odata, float *rabas, float *azbas, float *cr,
          float *cp, int nr, int np, int nx, int Nx, int xoff, int yoff) {
  return getkl(0.0f, ampli, d_odata, rabas, azbas, cr, cp, nr, np, nx, Nx, xoff,
               yoff);
}

int getkl(float *d_odata, float *rabas, float *azbas, float *cr, float *cp,
          int nr, int np, int nx, int Nx, int xoff, int yoff) {
  return getkl(0.0f, 1.0f, d_odata, rabas, azbas, cr, cp, nr, np, nx, Nx, xoff,
               yoff);
}

int combikl(float *com, int nkl, float *d_odata, float *rabas, int *d_ord,
            float *azbas, float *cr, float *cp, int nr, int np, int nx, int Nx,
            int xoff, int yoff) {
  int block_size = 8;
  int nnx =
      nx + block_size - nx % block_size;  // find next multiple of BLOCK_SZ
  int nny = nx + block_size - nx % block_size;
  dim3 blocks(nnx / block_size, nny / block_size),
      threads(block_size, block_size);

  // int smemSize = (block_size +1) * (block_size +1) * sizeof(float);

  // for (int cc=0;cc<nkl;cc++)
  combikl_krnl<<<blocks, threads>>>(com, nkl, d_odata, rabas, d_ord, azbas, cr,
                                    cp, nr, np, nx, Nx, xoff, yoff);

  carmaCheckMsg("get_kernel<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}

// Florian features
__global__ void flokrnl(long dim, float *bas) {
  int tid = blockIdx.x;
  if (tid < dim) bas[tid * dim + tid] = tid;
}

int cget_flokl(long nkl, long dim, float *covmat, float *filter, float *bas) {
  // int i;
  printf("flag CUDA \n");
  // for (i=0;i<dim;i++) bas[i] = i;
  flokrnl<<<dim, 1>>>(dim, bas);
  printf("flag CUDA done \n");
  carmaCheckMsg("get_kernel<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}
