#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
//  This file is part of COMPASS <https://anr-compass.github.io/compass/>
//
//  Copyright (C) 2011-2023 COMPASS Team <https://github.com/ANR-COMPASS>
//  All rights reserved.

// -----------------------------------------------------------------------------

//! \file      sutra_kl.cu
//! \ingroup   libsutra
//! \class     SutraKL
//! \brief     this class provides the kl features to COMPASS
//! \author    COMPASS Team <https://github.com/ANR-COMPASS>
//! \version   5.5.0
//! \date      2022/01/24

#include <sutra_kl.h>

__device__ float kl_sfi(float *rabas, float *azbas, int npix, int nrow) {
  return rabas[npix] * azbas[nrow];
}

__device__ void kl_interp(float alpha, float ampli, float *odata, float *rabas,
                          float *azbas, float xbi, float ybi, int nr, int np,
                          int tido) {
  if ((xbi >= 0) && (xbi <= nr - 1) && (ybi >= 0) && (ybi <= np - 1)) {
    int i0, i1, j0, j1;
    long ibi = (long)xbi;
    long jbi = (long)ybi;

    i0 = ibi;
    if (ibi < nr - 1)
      i1 = ibi + 1;
    else
      i1 = nr - 1;

    j0 = jbi;
    if (jbi < np - 1)
      j1 = jbi + 1;
    else
      j1 = np - 1;

    float wi, wj, w00, w01, w10, w11;

    wi = 1.0f - (xbi - ibi);
    wj = 1.0f - (ybi - jbi);

    w00 = wi * wj;
    w10 = (1 - wi) * wj;
    w01 = wi * (1 - wj);
    w11 = (1 - wi) * (1 - wj);

    odata[tido] =
        alpha * odata[tido] + ampli * (w00 * kl_sfi(rabas, azbas, i0, j0) +
                                       w10 * kl_sfi(rabas, azbas, i1, j0) +
                                       w01 * kl_sfi(rabas, azbas, i0, j1) +
                                       w11 * kl_sfi(rabas, azbas, i1, j1));
  } else
    odata[tido] = 0.0f;
}

__device__ void kl_interp(float alpha, float *ampli, int nkl, float *odata,
                          float *rabas, float *azbas, float xbi, float ybi,
                          int nr, int np, int tido) {
  if ((xbi >= 0) && (xbi <= nr - 1) && (ybi >= 0) && (ybi <= np - 1)) {
    int i0, i1, j0, j1;
    long ibi = (long)xbi;
    long jbi = (long)ybi;

    i0 = ibi;
    if (ibi < nr - 1)
      i1 = ibi + 1;
    else
      i1 = nr - 1;

    j0 = jbi;
    if (jbi < np - 1)
      j1 = jbi + 1;
    else
      j1 = np - 1;

    float wi, wj, w00, w01, w10, w11;

    wi = 1.0f - (xbi - ibi);
    wj = 1.0f - (ybi - jbi);

    w00 = wi * wj;
    w10 = (1 - wi) * wj;
    w01 = wi * (1 - wj);
    w11 = (1 - wi) * (1 - wj);

    odata[tido] =
        alpha * odata[tido] + ampli[nkl] * (w00 * kl_sfi(rabas, azbas, i0, j0) +
                                            w10 * kl_sfi(rabas, azbas, i1, j0) +
                                            w01 * kl_sfi(rabas, azbas, i0, j1) +
                                            w11 * kl_sfi(rabas, azbas, i1, j1));
  } else
    odata[tido] = 0.0f;
}

__global__ void getkl_krnl(float alpha, float ampli, float *odata, float *rabas,
                           float *azbas, float *cr, float *cp, int nr, int np,
                           int nx, int Nx, int xoff, int yoff) {
  int xid = threadIdx.x + blockIdx.x * blockDim.x;
  int yid = threadIdx.y + blockIdx.y * blockDim.y;

  if ((xid < nx) && (yid < nx)) {
    int xref = xid + xoff;
    int yref = yid + yoff;
    int tido = xref + yref * Nx;

    int tid = xid + yid * nx;
    float xbi = cr[tid];
    float ybi = cp[tid];

    kl_interp(alpha, ampli, odata, rabas, azbas, xbi, ybi, nr, np, tido);
  }
}

__global__ void combikl_krnl(float *com, int nkl, float *odata, float *rabas,
                             int *d_ord, float *azbas, float *cr, float *cp,
                             int nr, int np, int nx, int Nx, int xoff,
                             int yoff) {
  int xid = threadIdx.x + blockIdx.x * blockDim.x;
  int yid = threadIdx.y + blockIdx.y * blockDim.y;

  if ((xid < nx) && (yid < nx)) {
    int xref = xid + xoff;
    int yref = yid + yoff;
    int tido = xref + yref * Nx;

    int tid = xid + yid * nx;
    float xbi = cr[tid];
    float ybi = cp[tid];

    odata[tido] = 0.0f;
    int tmp;
    float *rabas_cc;
    float *azbas_cc;
    // int cc=10;
    for (int cc = 0; cc < nkl; cc++) {
      tmp = d_ord[cc] - 1;
      rabas_cc = &(rabas[cc * nr]);
      azbas_cc = &(azbas[tmp * np]);
      kl_interp(1.0f, com, cc, odata, rabas_cc, azbas_cc, xbi, ybi, nr, np,
                tido);
    }
    __syncthreads();
  }
}

int getkl(float alpha, float ampli, float *d_odata, float *rabas, float *azbas,
          float *cr, float *cp, int nr, int np, int nx, int Nx, int xoff,
          int yoff) {
  int block_size = 8;
  int nnx =
      nx + block_size - nx % block_size;  // find next multiple of BLOCK_SZ
  int nny = nx + block_size - nx % block_size;
  dim3 blocks(nnx / block_size, nny / block_size),
      threads(block_size, block_size);

  // int smemSize = (block_size +1) * (block_size +1) * sizeof(float);

  getkl_krnl<<<blocks, threads>>>(alpha, ampli, d_odata, rabas, azbas, cr, cp,
                                  nr, np, nx, Nx, xoff, yoff);

  carma_check_msg("get_kernel<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}

int getkl(float ampli, float *d_odata, float *rabas, float *azbas, float *cr,
          float *cp, int nr, int np, int nx, int Nx, int xoff, int yoff) {
  return getkl(0.0f, ampli, d_odata, rabas, azbas, cr, cp, nr, np, nx, Nx, xoff,
               yoff);
}

int getkl(float *d_odata, float *rabas, float *azbas, float *cr, float *cp,
          int nr, int np, int nx, int Nx, int xoff, int yoff) {
  return getkl(0.0f, 1.0f, d_odata, rabas, azbas, cr, cp, nr, np, nx, Nx, xoff,
               yoff);
}

int combikl(float *com, int nkl, float *d_odata, float *rabas, int *d_ord,
            float *azbas, float *cr, float *cp, int nr, int np, int nx, int Nx,
            int xoff, int yoff) {
  int block_size = 8;
  int nnx =
      nx + block_size - nx % block_size;  // find next multiple of BLOCK_SZ
  int nny = nx + block_size - nx % block_size;
  dim3 blocks(nnx / block_size, nny / block_size),
      threads(block_size, block_size);

  // int smemSize = (block_size +1) * (block_size +1) * sizeof(float);

  // for (int cc=0;cc<nkl;cc++)
  combikl_krnl<<<blocks, threads>>>(com, nkl, d_odata, rabas, d_ord, azbas, cr,
                                    cp, nr, np, nx, Nx, xoff, yoff);

  carma_check_msg("get_kernel<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}

// Florian features
__global__ void flokrnl(long dim, float *bas) {
  int tid = blockIdx.x;
  if (tid < dim) bas[tid * dim + tid] = tid;
}

int cget_flokl(long nkl, long dim, float *covmat, float *filter, float *bas) {
  // int i;
  printf("flag CUDA \n");
  // for (i=0;i<dim;i++) bas[i] = i;
  flokrnl<<<dim, 1>>>(dim, bas);
  printf("flag CUDA done \n");
  carma_check_msg("get_kernel<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}
