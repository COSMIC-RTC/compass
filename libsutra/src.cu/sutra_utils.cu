#include "hip/hip_runtime.h"
// This file is part of COMPASS <https://github.com/COSMIC-RTC/compass>
//
// COMPASS is free software: you can redistribute it and/or modify it under the terms of the GNU Lesser
// General Public License as published by the Free Software Foundation, either version 3 of the 
// License, or any later version.
//
// COMPASS is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; 
// without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. 
// See the GNU Lesser General Public License for more details.
//
// You should have received a copy of the GNU Lesser General Public License along with COMPASS. 
// If not, see <https://www.gnu.org/licenses/>
//
//  Copyright (C) 2011-2024 COSMIC Team <https://github.com/COSMIC-RTC/compass>

//! \file      sutra_utils.cu
//! \ingroup   libsutra
//! \class     sutra_utils
//! \brief     this file provides utilities to COMPASS
//! \author    COSMIC Team <https://github.com/COSMIC-RTC/compass>
//! \date      2022/01/24

#include <sutra_utils.hpp>
#include "carma_utils.cuh"

int32_t compute_nmaxhr(int64_t nvalid) {
  // this is the big array => we use nmaxhr and treat it sequentially

  int32_t mnmax = 500;
  int32_t nmaxhr = mnmax;
  if (nvalid > 2 * mnmax) {
    int32_t tmp0 = nvalid % mnmax;
    int32_t tmp = 0;
    for (int32_t cc = 1; cc < mnmax / 5; cc++) {
      tmp = nvalid % (mnmax + cc);
      if ((tmp > tmp0) || (tmp == 0)) {
        if (tmp == 0)
          tmp0 = 2 * mnmax;
        else
          tmp = tmp0;

        nmaxhr = mnmax + cc;
      }
    }
    return nmaxhr;
  }
  return nvalid;
}

__global__ void cfillrealp_krnl(hipFloatComplex *odata, float *idata, int32_t N) {
  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    odata[tid].x = idata[tid];
    tid += blockDim.x * gridDim.x;
  }
}

int32_t cfillrealp(hipFloatComplex *d_odata, float *d_idata, int32_t N,
               CarmaDevice *device) {
  int32_t nb_threads = 0, nb_blocks = 0;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);

  dim3 grid(nb_blocks), threads(nb_threads);

  cfillrealp_krnl<<<grid, threads>>>(d_odata, d_idata, N);

  carma_check_msg("cfillrealp_kernel<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}

__global__ void cgetrealp_krnl(float *odata, hipFloatComplex *idata, int32_t N) {
  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    odata[tid] = idata[tid].x;
    tid += blockDim.x * gridDim.x;
  }
}

int32_t cgetrealp(float *d_odata, hipFloatComplex *d_idata, int32_t N,
              CarmaDevice *device) {
  int32_t nb_threads = 0, nb_blocks = 0;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);

  dim3 grid(nb_blocks), threads(nb_threads);

  cgetrealp_krnl<<<grid, threads>>>(d_odata, d_idata, N);

  carma_check_msg("cgetrealp_kernel<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}

__global__ void abs2_krnl(float *odata, hipFloatComplex *idata, int32_t N) {
  hipFloatComplex cache;

  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    cache = idata[tid];
    odata[tid] = cache.x * cache.x + cache.y * cache.y;
    tid += blockDim.x * gridDim.x;
  }
}

int32_t abs2(float *d_odata, hipFloatComplex *d_idata, int32_t N, CarmaDevice *device) {
  int32_t nb_threads = 0, nb_blocks = 0;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);

  dim3 grid(nb_blocks), threads(nb_threads);

  abs2_krnl<<<grid, threads>>>(d_odata, d_idata, N);
  carma_check_msg("abs2_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void abs2_krnl(float *odata, hipFloatComplex *idata, int32_t N,
                          float fact) {
  hipFloatComplex cache;

  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    cache = idata[tid];
    odata[tid] += (fact * (cache.x * cache.x + cache.y * cache.y));
    tid += blockDim.x * gridDim.x;
  }
}

int32_t abs2(float *d_odata, hipFloatComplex *d_idata, int32_t N, float fact,
         CarmaDevice *device) {
  int32_t nb_threads = 0, nb_blocks = 0;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);

  dim3 grid(nb_blocks), threads(nb_threads);

  abs2_krnl<<<grid, threads>>>(d_odata, d_idata, N, fact);
  carma_check_msg("abs2_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void abs2c_krnl(hipFloatComplex *odata, hipFloatComplex *idata,
                           int32_t N) {
  hipFloatComplex cache;
  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    cache = idata[tid];
    odata[tid].x = cache.x * cache.x + cache.y * cache.y;
    odata[tid].y = 0.0;
    tid += blockDim.x * gridDim.x;
  }
}

int32_t abs2c(hipFloatComplex *d_odata, hipFloatComplex *d_idata, int32_t N,
          CarmaDevice *device) {
  int32_t nb_threads = 0, nb_blocks = 0;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);

  dim3 grid(nb_blocks), threads(nb_threads);
  // DEBUG_TRACE("N = %d, nb_threads = %d, nb_blocks = %d;",N , nb_threads, nb_blocks);
  abs2c_krnl<<<grid, threads>>>(d_odata, d_idata, N);
  carma_check_msg("abs2c_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void subapnorm_krnl(float *odata, float *idata, float *fact,
                               float *norm, float nphot, int32_t n, int32_t N) {
  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    if (norm[tid / n] != 0) {
      odata[tid] = idata[tid] * fact[tid / n] / norm[tid / n] * nphot;
    }
    tid += blockDim.x * gridDim.x;
  }
}

int32_t subap_norm(float *d_odata, float *d_idata, float *fact, float *norm,
               float nphot, int32_t n, int32_t N, CarmaDevice *device) {
  int32_t nb_threads = 0, nb_blocks = 0;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);

  dim3 grid(nb_blocks), threads(nb_threads);

  subapnorm_krnl<<<grid, threads>>>(d_odata, d_idata, fact, norm, nphot, n, N);
  carma_check_msg("subapnorm_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void subapnormasync_krnl(float *odata, float *idata, float *fact,
                                    float *norm, float nphot, int32_t n, int32_t N,
                                    int32_t istart) {
  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  tid += istart;
  while (tid < N) {
    odata[tid] = idata[tid] * fact[tid / n] / norm[tid / n] * nphot;
    tid += blockDim.x * gridDim.x;
  }
}

int32_t subap_norm_async(float *d_odata, float *d_idata, float *fact, float *norm,
                     float nphot, int32_t n, int32_t N, CarmaStreams *streams,
                     CarmaDevice *device) {
  int32_t nb_threads = 0, nb_blocks = 0;
  int32_t nstreams = streams->get_nb_streams();
  get_num_blocks_and_threads(device, N / nstreams, nb_blocks, nb_threads);

  dim3 grid(nb_blocks), threads(nb_threads);

  for (int32_t i = 0; i < nstreams; i++) {
    subapnormasync_krnl<<<grid, threads, 0, streams->get_stream(i)>>>(
        d_odata, d_idata, fact, norm, nphot, n, N, i * nb_blocks * nb_threads);
    carma_check_msg("subapnormasync_kernel<<<>>> execution failed\n");
  }

  return EXIT_SUCCESS;
}

__global__ void krnl_fillindx(float *odata, float *idata, int32_t *indx,
                              float alpha, float beta, int32_t N) {
  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    odata[tid] = (alpha * idata[indx[tid]]) + beta;
    tid += blockDim.x * gridDim.x;
  }
}

int32_t fillindx(float *d_odata, float *d_idata, int32_t *indx, float alpha, float beta,
             int32_t N, CarmaDevice *device) {
  int32_t nb_threads = 0, nb_blocks = 0;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);

  dim3 grid(nb_blocks), threads(nb_threads);

  krnl_fillindx<<<grid, threads>>>(d_odata, d_idata, indx, alpha, beta, N);

  carma_check_msg("fillindx_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

int32_t fillindx(float *d_odata, float *d_idata, int32_t *indx, int32_t N,
             CarmaDevice *device) {
  return fillindx(d_odata, d_idata, indx, 1.0f, 0.0f, N, device);
}
int32_t fillindx(float *d_odata, float *d_idata, int32_t *indx, float alpha, int32_t N,
             CarmaDevice *device) {
  return fillindx(d_odata, d_idata, indx, alpha, 0.0f, N, device);
}
__global__ void fillarr2d_krnl(float *odata, float *idata, int32_t tidx0, int32_t Ncol,
                               int32_t NC, int32_t N, int32_t dir) {
  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  int32_t tidB;

  while (tid < N) {
    if (Ncol > 1)
      tidB = tidx0 + (tid / Ncol) * NC + (tid % Ncol);
    else
      tidB = tidx0 + tid * NC;
    if (dir > 0)
      odata[tidB] = idata[tid];
    else
      odata[tidB] = idata[N - 1 - tid];
    tid += blockDim.x * gridDim.x;
  }
}

int32_t fillarr2d(float *d_odata, float *d_idata, int32_t x0, int32_t Ncol, int32_t NC, int32_t N,
              int32_t dir, CarmaDevice *device) {
  int32_t nb_threads = 0, nb_blocks = 0;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);

  dim3 grid(nb_blocks), threads(nb_threads);

  fillarr2d_krnl<<<grid, threads>>>(d_odata, d_idata, x0, Ncol, NC, N, dir);

  carma_check_msg("fillarr2d_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}
int32_t fillarr2d(float *d_odata, float *d_idata, int32_t x0, int32_t Ncol, int32_t NC, int32_t N,
              CarmaDevice *device) {
  return fillarr2d(d_odata, d_idata, x0, Ncol, NC, N, 1, device);
}
__global__ void getarr2d_krnl(float *odata, float *idata, int32_t tidx0, int32_t Ncol,
                              int32_t NC, int32_t N) {
  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  int32_t tidB;

  while (tid < N) {
    if (Ncol > 1)
      tidB = tidx0 + (tid / Ncol) * NC + (tid % Ncol);
    else
      tidB = tidx0 + tid * NC;
    odata[tid] = idata[tidB];
    tid += blockDim.x * gridDim.x;
  }
}

int32_t getarr2d(float *d_odata, float *d_idata, int32_t x0, int32_t Ncol, int32_t NC, int32_t N,
             CarmaDevice *device) {
  int32_t nb_threads = 0, nb_blocks = 0;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);

  dim3 grid(nb_blocks), threads(nb_threads);

  getarr2d_krnl<<<grid, threads>>>(d_odata, d_idata, x0, Ncol, NC, N);

  carma_check_msg("getarr2d_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

template <class T>
__global__ void addai_krnl(T *odata, T *idata, int32_t i, int32_t sgn, int32_t N) {
  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    if (sgn == 1)
      odata[tid] += idata[i];
    else
      odata[tid] -= idata[i];
    tid += blockDim.x * gridDim.x;
  }
}

template <class T>
int32_t addai(T *d_odata, T *i_data, int32_t i, int32_t sgn, int32_t N, CarmaDevice *device) {
  int32_t nb_threads = 0, nb_blocks = 0;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);

  dim3 grid(nb_blocks), threads(nb_threads);

  addai_krnl<T><<<grid, threads>>>(d_odata, i_data, i, sgn, N);

  carma_check_msg("plusai_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}
template int32_t addai<float>(float *d_odata, float *i_data, int32_t i, int32_t sgn, int32_t N,
                          CarmaDevice *device);
template int32_t addai<double>(double *d_odata, double *i_data, int32_t i, int32_t sgn,
                           int32_t N, CarmaDevice *device);

template <class T>
__global__ void roll_krnl(T *idata, int32_t N, int32_t M, int32_t Nim) {
  T tmp;

  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < (N * M / 2)) {
    int32_t x = tid % N;
    int32_t y = tid / N;

    int32_t xx = (x + N / 2) % N;
    int32_t yy = (y + M / 2) % M;
    int32_t tid2 = xx + yy * N;

    for (int32_t ii = 0; ii < Nim; ii++) {
      tmp = idata[tid + ii * N * M];
      idata[tid + ii * N * M] = idata[tid2 + ii * N * M];
      idata[tid2 + ii * N * M] = tmp;
    }

    tid += blockDim.x * gridDim.x;
  }
}

template <class T>
int32_t roll(T *idata, int32_t N, int32_t M, int32_t nim, CarmaDevice *device) {
  int64_t Ntot = N * M;
  int32_t nb_blocks, nb_threads;
  get_num_blocks_and_threads(device, Ntot / 2, nb_blocks, nb_threads);

  dim3 grid(nb_blocks), threads(nb_threads);

  roll_krnl<T><<<grid, threads>>>(idata, N, M, nim);

  carma_check_msg("roll_kernel<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}

template int32_t roll<float>(float *idata, int32_t N, int32_t M, int32_t nim,
                         CarmaDevice *device);

template int32_t roll<double>(double *idata, int32_t N, int32_t M, int32_t nim,
                          CarmaDevice *device);

template int32_t roll<hipFloatComplex>(hipFloatComplex *idata, int32_t N, int32_t M, int32_t nim,
                                  CarmaDevice *device);

template <class T>
__global__ void roll_krnl(T *idata, int32_t N, int32_t M) {
  T tmp;

  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < (N * M / 2)) {
    int32_t x = tid % N;
    int32_t y = tid / N;

    int32_t xx = (x + N / 2) % N;
    int32_t yy = (y + M / 2) % M;
    int32_t tid2 = xx + yy * N;

    tmp = idata[tid];
    idata[tid] = idata[tid2];
    idata[tid2] = tmp;

    tid += blockDim.x * gridDim.x;
  }
}

template <class T>
int32_t roll(T *idata, int32_t N, int32_t M, CarmaDevice *device) {
  int64_t Ntot = N * M;
  int32_t nb_blocks, nb_threads;
  get_num_blocks_and_threads(device, Ntot / 2, nb_blocks, nb_threads);

  dim3 grid(nb_blocks), threads(nb_threads);

  roll_krnl<T><<<grid, threads>>>(idata, N, M);

  carma_check_msg("roll_kernel<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}

template int32_t roll<float>(float *idata, int32_t N, int32_t M, CarmaDevice *device);

template int32_t roll<double>(double *idata, int32_t N, int32_t M, CarmaDevice *device);

template int32_t roll<hipFloatComplex>(hipFloatComplex *idata, int32_t N, int32_t M,
                                  CarmaDevice *device);

template <class T>
__global__ void roll_mult_krnl(T *odata, T *idata, int32_t N, int32_t M, T alpha) {
  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < (N * M / 2)) {
    int32_t x = tid % N;
    int32_t y = tid / N;

    int32_t xx = (x + N / 2) % N;
    int32_t yy = (y + M / 2) % M;
    int32_t tid2 = xx + yy * N;

    odata[tid] = alpha * idata[tid2];
    odata[tid2] = alpha * idata[tid];

    tid += blockDim.x * gridDim.x;
  }
}

template <class T>
int32_t roll_mult(T *odata, T *idata, int32_t N, int32_t M, T alpha, CarmaDevice *device) {
  int64_t Ntot = N * M;
  int32_t nb_blocks, nb_threads;
  get_num_blocks_and_threads(device, Ntot / 2, nb_blocks, nb_threads);

  dim3 grid(nb_blocks), threads(nb_threads);

  roll_mult_krnl<T><<<grid, threads>>>(odata, idata, N, M, alpha);

  carma_check_msg("roll_kernel<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}

template int32_t roll_mult<float>(float *odata, float *idata, int32_t N, int32_t M,
                              float alpha, CarmaDevice *device);

template int32_t roll_mult<double>(double *odata, double *idata, int32_t N, int32_t M,
                               double alpha, CarmaDevice *device);

template <class T>
__global__ void avg_krnl(T *data, T *p_sum, int32_t N) {
  T *sdata = SharedMemory<T>();
  // Load shared memory
  int32_t tid = threadIdx.x + blockDim.x * blockIdx.x;
  int32_t sid = threadIdx.x;

  if (tid < N)
    sdata[sid] = data[tid];
  else
    sdata[sid] = 0;

  __syncthreads();

  reduce_krnl(sdata, blockDim.x, sid);

  __syncthreads();

  if (threadIdx.x == 0) p_sum[blockIdx.x] = sdata[0];
}

template <class T>
__global__ void remove_avg_krnl(T *data, int32_t N, T avg) {
  int32_t tid = threadIdx.x + blockDim.x * blockIdx.x;
  while (tid < N) {
    data[tid] -= avg;
    tid += blockDim.x * gridDim.x;
  }
}

template <class T>
int32_t remove_avg(T *data, int32_t N, CarmaDevice *device) {
  int32_t nb_threads = 0, nb_blocks = 0;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);
  dim3 grid(nb_blocks), threads(nb_threads);
  int32_t smemSize = nb_threads * sizeof(T);

  T p_sum_c[nb_blocks];
  T *p_sum;
  carma_safe_call(hipMalloc((void **)&(p_sum), sizeof(T) * nb_blocks));

  avg_krnl<<<grid, threads, smemSize>>>(data, p_sum, N);
  carma_check_msg("avg_krnl<<<>>> execution failed\n");
  carma_safe_call(
      hipMemcpy(p_sum_c, p_sum, nb_blocks * sizeof(T), hipMemcpyDeviceToHost));
  carma_safe_call(hipFree(p_sum));

  T avg = 0;
  for (int32_t i = 0; i < nb_blocks; i++) {
    avg += p_sum_c[i];
  }
  avg /= N;
  remove_avg_krnl<<<grid, threads>>>(data, N, avg);
  carma_check_msg("remove_avg_krnl<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

template int32_t remove_avg<float>(float *data, int32_t N, CarmaDevice *device);
template int32_t remove_avg<double>(double *data, int32_t N, CarmaDevice *device);

__global__ void conv_krnl(hipFloatComplex *odata, hipFloatComplex *idata, int32_t N) {
  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  hipFloatComplex tmp;

  while (tid < N) {
    tmp.x = idata[tid].x * odata[tid].x - idata[tid].y * odata[tid].y;
    tmp.y = idata[tid].y * odata[tid].x + idata[tid].x * odata[tid].y;
    odata[tid] = tmp;
    tid += blockDim.x * gridDim.x;
  }
}

int32_t convolve(hipFloatComplex *d_odata, hipFloatComplex *d_idata, int32_t N,
             CarmaDevice *device) {
  int32_t nb_threads = 0, nb_blocks = 0;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);

  dim3 grid(nb_blocks), threads(nb_threads);

  conv_krnl<<<grid, threads>>>(d_odata, d_idata, N);

  carma_check_msg("conv_kernel<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}

__global__ void convmod_krnl(hipFloatComplex *odata, hipFloatComplex *idata,
                             int32_t mod, int32_t N) {
  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  hipFloatComplex tmp;

  while (tid < N) {
    tmp.x = (idata[tid].x * odata[tid].x - idata[tid].y * odata[tid].y) / mod;
    tmp.y = (idata[tid].y * odata[tid].x + idata[tid].x * odata[tid].y) / mod;
    odata[tid] = tmp;
    tid += blockDim.x * gridDim.x;
  }
}

int32_t convolve_modulate(hipFloatComplex *d_odata, hipFloatComplex *d_idata, int32_t mod,
                      int32_t N, CarmaDevice *device) {
  int32_t nb_threads = 0, nb_blocks = 0;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);

  dim3 grid(nb_blocks), threads(nb_threads);

  convmod_krnl<<<grid, threads>>>(d_odata, d_idata, mod, N);

  carma_check_msg("conv_kernel<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}

template <class T>
__global__ void mult_krnl(T *i_data, T *scale, int32_t N) {
  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    i_data[tid] = i_data[tid] * scale[tid];
    tid += blockDim.x * gridDim.x;
  }
}

template <class T>
__global__ void mult_krnl(T *i_data, T *scale, T gain, int32_t N) {
  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    i_data[tid] = i_data[tid] * scale[tid] * gain;
    tid += blockDim.x * gridDim.x;
  }
}

template <class T>
__global__ void mult_krnl(T *i_data, T gain, int32_t N) {
  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    i_data[tid] = i_data[tid] * gain;
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void mult_int_krnl(float *o_data, float *i_data, float gain, int32_t N) {
  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    o_data[tid] = gain * i_data[tid] + o_data[tid];
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void mult_int_krnl(float *o_data, float *i_data, float *scale,
                              float gain, int32_t N) {
  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    o_data[tid] = gain * (i_data[tid] * scale[tid]) + o_data[tid];
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void mult_int_krnl(float *o_data, float *i_data, float *scale,
                              float gain, int32_t N, int32_t istart) {
  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  tid += istart;

  while (tid < N) {
    o_data[tid] = gain * (i_data[tid] * scale[tid]) + o_data[tid];
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void add_md_krnl(float *o_matrix, float *i_matrix, float *i_vector,
                            int32_t N) {
  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    o_matrix[tid * (N + 1)] = i_matrix[tid * (N + 1)] + i_vector[tid];
    tid += blockDim.x * gridDim.x;
  }
}

template <class T>
int32_t mult_vect(T *d_data, T *scale, int32_t N, CarmaDevice *device) {
  int32_t nb_blocks, nb_threads;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);
  dim3 grid(nb_blocks), threads(nb_threads);

  mult_krnl<<<grid, threads>>>(d_data, scale, N);

  carma_check_msg("mult_kernel<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}
template int32_t mult_vect<float>(float *d_data, float *scale, int32_t N,
                              CarmaDevice *device);
template int32_t mult_vect<double>(double *d_data, double *scale, int32_t N,
                               CarmaDevice *device);

template <class T>
int32_t mult_vect(T *d_data, T *scale, T gain, int32_t N, CarmaDevice *device) {
  int32_t nb_blocks, nb_threads;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);
  dim3 grid(nb_blocks), threads(nb_threads);

  mult_krnl<<<grid, threads>>>(d_data, scale, gain, N);

  carma_check_msg("mult_kernel<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}

template int32_t mult_vect<float>(float *d_data, float *scale, float gain, int32_t N,
                              CarmaDevice *device);
template int32_t mult_vect<double>(double *d_data, double *scale, double gain,
                               int32_t N, CarmaDevice *device);

template <class T>
int32_t mult_vect(T *d_data, T gain, int32_t N, CarmaDevice *device) {
  int32_t nb_blocks, nb_threads;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);
  dim3 grid(nb_blocks), threads(nb_threads);

  mult_krnl<<<grid, threads>>>(d_data, gain, N);

  carma_check_msg("mult_kernel<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}

template int32_t mult_vect<float>(float *d_data, float gain, int32_t N,
                              CarmaDevice *device);
template int32_t mult_vect<double>(double *d_data, double gain, int32_t N,
                               CarmaDevice *device);

int32_t mult_int(float *o_data, float *i_data, float *scale, float gain, int32_t N,
             CarmaDevice *device, CarmaStreams *streams) {
  int32_t nb_threads = 0, nb_blocks = 0;

  int32_t nstreams = streams->get_nb_streams();
  get_num_blocks_and_threads(device, N / nstreams, nb_blocks, nb_threads);

  dim3 grid(nb_blocks), threads(nb_threads);

  for (int32_t i = 0; i < nstreams; i++) {
    mult_int_krnl<<<grid, threads, 0, streams->get_stream(i)>>>(
        o_data, i_data, scale, gain, N, i * nb_blocks * nb_threads);
    carma_check_msg("multint_kernel<<<>>> execution failed\n");
  }

  return EXIT_SUCCESS;
}

int32_t mult_int(float *o_data, float *i_data, float *scale, float gain, int32_t N,
             CarmaDevice *device) {
  int32_t nb_threads = 0, nb_blocks = 0;

  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);

  dim3 grid(nb_blocks), threads(nb_threads);

  mult_int_krnl<<<grid, threads>>>(o_data, i_data, scale, gain, N);
  carma_check_msg("multint_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

int32_t mult_int(float *o_data, float *i_data, float gain, int32_t N,
             CarmaDevice *device) {
  int32_t nb_threads = 0, nb_blocks = 0;

  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);

  dim3 grid(nb_blocks), threads(nb_threads);

  mult_int_krnl<<<grid, threads>>>(o_data, i_data, gain, N);
  carma_check_msg("multint_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

int32_t add_md(float *o_matrix, float *i_matrix, float *i_vector, int32_t N,
           CarmaDevice *device) {
  int32_t nb_threads = 0, nb_blocks = 0;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);
  dim3 grid(nb_blocks), threads(nb_threads);

  add_md_krnl<<<grid, threads>>>(o_matrix, i_matrix, i_vector, N);
  carma_check_msg("add_md_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}
