#include "hip/hip_runtime.h"
#include <sutra_turbu.h>
#include <hipblas.h>

extern __shared__ float cache_shm[];

__global__ void vonkarman_krnl(hipFloatComplex *odata, float *idata, float k0,
    int nalias, int nx, int ny, int blockSize) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  cache_shm[threadIdx.x + threadIdx.y * blockSize] = 0.0f;

  if ((x < nx) && (y < ny)) {
    // generate von karman spectrum
    for (int i = -nalias; i <= nalias; i++) {
      for (int j = -nalias; j <= nalias; j++) {
        if ((i == 0) && (j == 0)) {
          float xc = nx / 2;
          float yc = ny / 2;
          float tmp = sqrtf(
              (xc - x) * (xc - x) + (yc - y) * (yc - y) + k0 * k0);
          if (tmp > 1.)
            cache_shm[threadIdx.x + threadIdx.y * blockSize] =
                (6.88f * 0.00969f) * pow(tmp, -1.83333f);
          else
            cache_shm[threadIdx.x + threadIdx.y * blockSize] =
                (6.88f * 0.00969f);
        } else {
          float xc = x * nx + nx / 2;
          float yc = y * ny + ny / 2;
          cache_shm[threadIdx.x + threadIdx.y * blockSize] += (6.88f * 0.00969f)
              * pow(sqrtf((xc - x) * (xc - x) + (yc - y) * (yc - y) + k0 * k0),
                  -1.83333f);
        }
      }
    }

    odata[x + y * nx].x = cache_shm[threadIdx.x + threadIdx.y * blockSize]
        * cosf(2.0f * 3.14159f * idata[x + y * nx]);
    odata[x + y * nx].y = cache_shm[threadIdx.x + threadIdx.y * blockSize]
        * sinf(2.0f * 3.14159f * idata[x + y * nx]);
  }

  if ((x == 0) && (y == 0)) {
    odata[x + y * nx].x = 0.0f;
    odata[x + y * nx].y = 0.0f;
  }
}

int gene_vonkarman(hipFloatComplex *d_odata, float *d_idata, float k0,
    int nalias, int nx, int ny, int block_size) {
  int nnx = nx + block_size - nx % block_size; // find next multiple of BLOCK_SZ
  int nny = ny + block_size - ny % block_size;
  dim3 blocks(nnx / block_size, nny / block_size), threads(block_size,
      block_size);

  int smemSize = (block_size + 1) * (block_size + 1) * sizeof(float);

  vonkarman_krnl<<<blocks, threads, smemSize>>>(d_odata, d_idata, k0, nalias,
      nx, ny, block_size);

  cutilCheckMsg("raytrace_kernel<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}

__global__ void dphix_krnl(float *odata, float *idata, int N, int iter,
    int nx) {

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid + iter < N) {
    if (tid % nx < nx - iter)
      odata[tid] = (idata[tid] - idata[tid + iter])
          * (idata[tid] - idata[tid + iter]);
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void dphiy_krnl(float *odata, float *idata, int N, int iter,
    int nx) {

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid + iter * nx < N) {
    odata[tid] = (idata[tid] - idata[tid + iter * nx])
        * (idata[tid] - idata[tid + iter * nx]);
    tid += blockDim.x * gridDim.x;
  }
}

int norm_pscreen(float *d_odata, float *d_idata, int nx, int ny,
    float norm_fact, int device) {
  float sfx, sfy, norm = 0;
  int nthreads = 0, nblocks = 0;
  getNumBlocksAndThreads(device, nx * ny, nblocks, nthreads);

  dim3 grid(nblocks), threads(nthreads);

  hipblasHandle_t cublas_handle;
  hipblasCreate(&cublas_handle);

  int npts = 5;
  for (int i = 1; i < npts + 1; i++) {
    cutilSafeCall(hipMemset(d_odata, 0, sizeof(float) * nx * ny));
    dphix_krnl<<<grid, threads>>>(d_odata, d_idata, nx * ny, i, nx);
    cutilCheckMsg("dphix_kernel<<<>>> execution failed\n");
    //sfx  = hipblasSasum(nx*ny,d_odata,1)/((nx-i)*ny);
    // here we can use asum because the initial array is positive (result of a square)
    hipblasSasum(cublas_handle, nx * ny, d_odata, 1, &sfx);

    cutilSafeCall(hipMemset(d_odata, 0, sizeof(float) * nx * ny));
    dphiy_krnl<<<grid, threads>>>(d_odata, d_idata, nx * ny, i, nx);
    cutilCheckMsg("dphiy_kernel<<<>>> execution failed\n");
    //sfy  = hipblasSasum(nx*ny,d_odata,1)/((ny-i)*nx);
    hipblasSasum(cublas_handle, nx * ny, d_odata, 1, &sfy);

    //norm += sqrtf((sfx/((nx-i)*ny) + sfy/((ny-i)*nx))/2.0f)/sqrtf(6.88*pow(i,1.66));
    norm += sqrtf(sfx / ((nx - i) * ny)) / sqrtf(6.88 * pow(i, 1.66));
  }
  norm /= npts;

  cutilSafeCall(hipMemset(d_odata, 0, sizeof(float) * nx * ny));
  //hipblasSaxpy(nx*ny,1.0f/norm*norm_fact, d_idata, 1, d_odata, 1);
  norm = (1.0f / norm) * norm_fact;
  hipblasSaxpy(cublas_handle, nx * ny, &norm, d_idata, 1, d_odata, 1);

  hipblasDestroy(cublas_handle);
  return EXIT_SUCCESS;
}

/*
 __global__ void cgetscreen_krnl(float *odata, hipFloatComplex *idata, int N, int nx)
 {

 int tid = threadIdx.x + blockIdx.x * blockDim.x;

 while (tid < N) {
 odata[tid] = idata[tid].x;
 tid += blockDim.x * gridDim.x;
 }
 }

 int cgetscreen(float *d_odata,hipFloatComplex *d_idata,int N,int device)
 {

 int nthreads = 0,nblocks = 0;
 getNumBlocksAndThreads(device, N, nblocks, nthreads);

 dim3 grid(nblocks), threads(nthreads);

 cgetrealp_krnl<<<grid, threads>>>(d_odata, d_idata, N);

 return EXIT_SUCCESS;
 }


 */
