#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
//  This file is part of COMPASS <https://anr-compass.github.io/compass/>
//
//  Copyright (C) 2011-2019 COMPASS Team <https://github.com/ANR-COMPASS>
//  All rights reserved.
//  Distributed under GNU - LGPL
//
//  COMPASS is free software: you can redistribute it and/or modify it under the terms of the GNU Lesser 
//  General Public License as published by the Free Software Foundation, either version 3 of the License, 
//  or any later version.
//
//  COMPASS: End-to-end AO simulation tool using GPU acceleration 
//  The COMPASS platform was designed to meet the need of high-performance for the simulation of AO systems. 
//  
//  The final product includes a software package for simulating all the critical subcomponents of AO, 
//  particularly in the context of the ELT and a real-time core based on several control approaches, 
//  with performances consistent with its integration into an instrument. Taking advantage of the specific 
//  hardware architecture of the GPU, the COMPASS tool allows to achieve adequate execution speeds to
//  conduct large simulation campaigns called to the ELT. 
//  
//  The COMPASS platform can be used to carry a wide variety of simulations to both testspecific components 
//  of AO of the E-ELT (such as wavefront analysis device with a pyramid or elongated Laser star), and 
//  various systems configurations such as multi-conjugate AO.
//
//  COMPASS is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the 
//  implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  
//  See the GNU Lesser General Public License for more details.
//
//  You should have received a copy of the GNU Lesser General Public License along with COMPASS. 
//  If not, see <https://www.gnu.org/licenses/lgpl-3.0.txt>.
// -----------------------------------------------------------------------------

//! \file      sutra_centroider_tcog.cu
//! \ingroup   libsutra
//! \class     sutra_centroider_tcog
//! \brief     this class provides the centroider_tcog features to COMPASS
//! \author    COMPASS Team <https://github.com/ANR-COMPASS>
//! \version   4.3.2
//! \date      2011/01/28
//! \copyright GNU Lesser General Public License

#include <sutra_centroider_tcog.h>
#include <carma_utils.cuh>

template <class T, int Nthreads>
__global__ void centroids(float *d_img, T *d_centroids, T *ref, int *validx,
                          int *validy, float *d_intensities, float threshold,
                          unsigned int npix, unsigned int size, T scale,
                          T offset, unsigned int nelem_thread) {
  if (blockDim.x > Nthreads) {
    if (threadIdx.x == 0) printf("Wrong size argument\n");
    return;
  }
  // Specialize BlockReduce for a 1D block of 128 threads on type int
  typedef hipcub::BlockReduce<float, Nthreads> BlockReduce;
  // Allocate shared memory for BlockReduce
  __shared__ typename BlockReduce::TempStorage temp_storage;

  float idata = 0;
  float xdata = 0;
  float ydata = 0;
  // load shared mem
  unsigned int tid = threadIdx.x;
  unsigned int xvalid = validx[blockIdx.x];
  unsigned int yvalid = validy[blockIdx.x];
  unsigned int x, y;
  int idim;

  for (int cc = 0; cc < nelem_thread; cc++) {
    x = ((tid * nelem_thread + cc) % npix);
    y = ((tid * nelem_thread + cc) / npix);
    // idim = tid * nelem_thread + cc + (blockDim.x * nelem_thread) *
    // blockIdx.x;
    idim = (x + xvalid) + (y + yvalid) * size;
    if (idim < size * size) {
      float data_thresh =
          (d_img[idim] > threshold) ? d_img[idim] - threshold : 0.f;
      idata += data_thresh;
      xdata += data_thresh * x;
      ydata += data_thresh * y;
      d_img[idim] = data_thresh;
    }
  }

  // sdata[tid] = (i < N) ? g_idata[i] * x : 0;
  __syncthreads();

  float intensity = BlockReduce(temp_storage).Sum(idata, blockDim.x);
  __syncthreads();
  float slopex = BlockReduce(temp_storage).Sum(xdata, blockDim.x);
  __syncthreads();
  float slopey = BlockReduce(temp_storage).Sum(ydata, blockDim.x);

  // write result for this block to global mem
  if (tid == 0) {
    d_centroids[blockIdx.x] =
        (T(slopex / (intensity + 1.e-6)) - offset) * scale - ref[blockIdx.x];
    d_centroids[blockIdx.x + gridDim.x] =
        (T(slopey / (intensity + 1.e-6)) - offset) * scale -
        ref[blockIdx.x + gridDim.x];
    d_intensities[blockIdx.x] = intensity;
  }
}

template <class T>
void get_centroids(int size, int threads, int blocks, int npix, float *d_img,
                   T *d_centroids, T *ref, int *validx, int *validy,
                   float *intensities, float threshold, float scale,
                   float offset, carma_device *device) {
  int maxThreads = device->get_properties().maxThreadsPerBlock;
  unsigned int nelem_thread = 1;
  while ((threads / nelem_thread > maxThreads) ||
         (threads % nelem_thread != 0)) {
    nelem_thread++;
  }

  threads /= nelem_thread;
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  if (threads <= 16)
    centroids<T, 16><<<dimGrid, 16>>>(d_img, d_centroids, ref, validx, validy,
                                      intensities, threshold, npix, size,
                                      T(scale), T(offset), nelem_thread);
  else if (threads <= 36)
    centroids<T, 36><<<dimGrid, 36>>>(d_img, d_centroids, ref, validx, validy,
                                      intensities, threshold, npix, size,
                                      T(scale), T(offset), nelem_thread);

  else if (threads <= 64)
    centroids<T, 64><<<dimGrid, 64>>>(d_img, d_centroids, ref, validx, validy,
                                      intensities, threshold, npix, size,
                                      T(scale), T(offset), nelem_thread);
  else if (threads <= 100)
    centroids<T, 100><<<dimGrid, 100>>>(d_img, d_centroids, ref, validx, validy,
                                        intensities, threshold, npix, size,
                                        T(scale), T(offset), nelem_thread);
  else if (threads <= 144)
    centroids<T, 144><<<dimGrid, 144>>>(d_img, d_centroids, ref, validx, validy,
                                        intensities, threshold, npix, size,
                                        T(scale), T(offset), nelem_thread);
  else if (threads <= 256)
    centroids<T, 256><<<dimGrid, 256>>>(d_img, d_centroids, ref, validx, validy,
                                        intensities, threshold, npix, size,
                                        T(scale), T(offset), nelem_thread);
  else if (threads <= 512)
    centroids<T, 512><<<dimGrid, 512>>>(d_img, d_centroids, ref, validx, validy,
                                        intensities, threshold, npix, size,
                                        T(scale), T(offset), nelem_thread);
  else if (threads <= 1024)
    centroids<T, 1024><<<dimGrid, 1024>>>(
        d_img, d_centroids, ref, validx, validy, intensities, threshold, npix,
        size, T(scale), T(offset), nelem_thread);
  else
    printf("SH way too big !!!\n");

  carmaCheckMsg("centroids_kernel<<<>>> execution failed\n");
}

template void get_centroids<float>(int size, int threads, int blocks, int npix,
                                   float *d_img, float *d_centroids, float *ref,
                                   int *validx, int *validy, float *intensities,
                                   float threshold, float scale, float offset,
                                   carma_device *device);

template void get_centroids<double>(int size, int threads, int blocks, int npix,
                                    float *d_img, double *d_centroids,
                                    double *ref, int *validx, int *validy,
                                    float *intensities, float threshold,
                                    float scale, float offset,
                                    carma_device *device);
#ifdef CAN_DO_HALF
template void get_centroids<half>(int size, int threads, int blocks, int npix,
                                  float *d_img, half *d_centroids, half *ref,
                                  int *validx, int *validy, float *intensities,
                                  float threshold, float scale, float offset,
                                  carma_device *device);
#endif
// template <class T>
// void get_centroids(int size, int threads, int blocks, int n, T *d_idata,
//                    T *d_odata, T *alpha, T thresh, float scale, float offset,
//                    carma_device *device) {
//   int maxThreads = device->get_properties().maxThreadsPerBlock;
//   unsigned int nelem_thread = 1;
//   while ((threads / nelem_thread > maxThreads) ||
//          (threads % nelem_thread != 0)) {
//     nelem_thread++;
//   }
//   threads /= nelem_thread;
//   dim3 dimBlock(threads, 1, 1);
//   dim3 dimGrid(blocks, 1, 1);

//   // when there is only one warp per block, we need to allocate two warps
//   // worth of shared memory so that we don't index shared memory out of
//   bounds int smemSize =
//       (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);
//   centroidx<T><<<dimGrid, dimBlock, smemSize>>>(
//       d_idata, d_odata, alpha, thresh, n, size, scale, offset, nelem_thread);

//   carmaCheckMsg("centroidx_kernel<<<>>> execution failed\n");

//   centroidy<T><<<dimGrid, dimBlock, smemSize>>>(d_idata, &(d_odata[blocks]),
//                                                 alpha, thresh, n, size,
//                                                 scale, offset, nelem_thread);

//   carmaCheckMsg("centroidy_kernel<<<>>> execution failed\n");
// }

// template void get_centroids<float>(int size, int threads, int blocks, int n,
//                                    float *d_idata, float *d_odata, float
//                                    *alpha,
//                                    float thresh, float scale, float offset,
//                                    carma_device *device);

// template void get_centroids<double>(int size, int threads, int blocks, int n,
//                                     double *d_idata, double *d_odata,
//                                     double *alpha, double thresh, double
//                                     scale,
//                                     double offset, carma_device *device);
// template <class T>
// __global__ void centroidx(T *g_idata, T *g_odata, T *alpha, T thresh,
//                           unsigned int n, unsigned int N, float scale, float
//                           offset, unsigned int nelem_thread) {
//   T *sdata = SharedMemory<T>();

//   // load shared mem
//   unsigned int tid = threadIdx.x;
//   // unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
//   // unsigned int x = (tid % n) + 1;
//   unsigned int x;
//   int idim;
//   sdata[tid] = 0;
//   for (int cc = 0; cc < nelem_thread; cc++) {
//     x = ((tid * nelem_thread + cc) % n);
//     idim = tid * nelem_thread + cc + (blockDim.x * nelem_thread) *
//     blockIdx.x; if (idim < N)
//       sdata[tid] += (g_idata[idim] > thresh) ? g_idata[idim] * x : 0;
//     else
//       sdata[tid] += 0;
//   }

//   // if (i < N)
//   //   sdata[tid] = (g_idata[i] > thresh) ? g_idata[i] * x : 0;

//   __syncthreads();

//   reduce_krnl(sdata, blockDim.x, tid);

//   // write result for this block to global mem
//   if (tid == 0)
//     g_odata[blockIdx.x] =
//         ((sdata[0] * 1.0 / (alpha[blockIdx.x] + 1.e-6)) - offset) * scale;
// }

// template <class T>
// __global__ void centroidy(T *g_idata, T *g_odata, T *alpha, T thresh,
//                           unsigned int n, unsigned int N, float scale, float
//                           offset, unsigned int nelem_thread) {
//   T *sdata = SharedMemory<T>();

//   // load shared mem
//   unsigned int tid = threadIdx.x;
//   // unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
//   // unsigned int y = (tid / n) + 1;
//   unsigned int y;
//   int idim;
//   sdata[tid] = 0;
//   for (int cc = 0; cc < nelem_thread; cc++) {
//     y = ((tid * nelem_thread + cc) / n);
//     idim = tid * nelem_thread + cc + (blockDim.x * nelem_thread) *
//     blockIdx.x; if (idim < N)
//       sdata[tid] += (g_idata[idim] > thresh) ? g_idata[idim] * y : 0;
//     else
//       sdata[tid] += 0;
//   }

//   // if (i < N)
//   //   sdata[tid] = (g_idata[i] > thresh) ? g_idata[i] * y : 0;

//   __syncthreads();

//   reduce_krnl(sdata, blockDim.x, tid);

//   // write result for this block to global mem
//   if (tid == 0)
//     g_odata[blockIdx.x] =
//         ((sdata[0] * 1.0 / (alpha[blockIdx.x] + 1.e-6)) - offset) * scale;
// }
