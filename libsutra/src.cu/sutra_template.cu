#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
//  This file is part of COMPASS <https://anr-compass.github.io/compass/>
//
//  Copyright (C) 2011-2023 COMPASS Team <https://github.com/ANR-COMPASS>
//  All rights reserved.

// -----------------------------------------------------------------------------

//! \file      sutra_template.cu
//! \ingroup   libsutra
//! \class     SutraTemplate
//! \brief     this class provides a class template to COMPASS
//! \author    COMPASS Team <https://github.com/ANR-COMPASS>
//! \version   5.5.0
//! \date      2022/01/24

#include <sutra_template.hpp>
#include "carma_utils.cuh"

template <class T>
__global__ void comp_aotemplate_krnl(T *g_idata, T *g_odata, int32_t sh_size,
                                     int32_t N) {
  T *sdata = SharedMemory<T>();

  // load shared mem
  uint32_t tid = threadIdx.x;
  uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < N) {
    // fill shared mem with data
    sdata[tid] = g_idata[i];
  }

  __syncthreads();

  if (i < N) {
    // write result for this block to global mem
    g_odata[i] =
        sin((sdata[tid] - sdata[(tid + 1) % sh_size]) * 2.0f * CARMA_PI);
  }
}

template <class T>
void comp_aotemplate(int32_t threads, int32_t blocks, T *d_idata, T *d_odata, int32_t N) {
  dim3 dimBlock(threads, 1, 1);
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  int32_t smemSize =
      (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);
  comp_aotemplate_krnl<T>
      <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, smemSize, N);

  carma_check_msg("comp_aotemplate_kernel<<<>>> execution failed\n");
}

template void comp_aotemplate<float>(int32_t threads, int32_t blocks, float *d_idata,
                                     float *d_odata, int32_t N);

template void comp_aotemplate<double>(int32_t threads, int32_t blocks, double *d_idata,
                                      double *d_odata, int32_t N);
