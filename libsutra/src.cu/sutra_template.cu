#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
//  This file is part of COMPASS <https://anr-compass.github.io/compass/>
//
//  Copyright (C) 2011-2022 COMPASS Team <https://github.com/ANR-COMPASS>
//  All rights reserved.
//  Distributed under GNU - LGPL
//
//  COMPASS is free software: you can redistribute it and/or modify it under the terms of the GNU Lesser
//  General Public License as published by the Free Software Foundation, either version 3 of the License,
//  or any later version.
//
//  COMPASS: End-to-end AO simulation tool using GPU acceleration
//  The COMPASS platform was designed to meet the need of high-performance for the simulation of AO systems.
//
//  The final product includes a software package for simulating all the critical subcomponents of AO,
//  particularly in the context of the ELT and a real-time core based on several control approaches,
//  with performances consistent with its integration into an instrument. Taking advantage of the specific
//  hardware architecture of the GPU, the COMPASS tool allows to achieve adequate execution speeds to
//  conduct large simulation campaigns called to the ELT.
//
//  The COMPASS platform can be used to carry a wide variety of simulations to both testspecific components
//  of AO of the E-ELT (such as wavefront analysis device with a pyramid or elongated Laser star), and
//  various systems configurations such as multi-conjugate AO.
//
//  COMPASS is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the
//  implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.
//  See the GNU Lesser General Public License for more details.
//
//  You should have received a copy of the GNU Lesser General Public License along with COMPASS.
//  If not, see <https://www.gnu.org/licenses/lgpl-3.0.txt>.
// -----------------------------------------------------------------------------

//! \file      sutra_template.cu
//! \ingroup   libsutra
//! \class     SutraTemplate
//! \brief     this class provides a class template to COMPASS
//! \author    COMPASS Team <https://github.com/ANR-COMPASS>
//! \version   5.2.1
//! \date      2022/01/24
//! \copyright GNU Lesser General Public License

#include <sutra_template.h>
#include "carma_utils.cuh"

template <class T>
__global__ void comp_aotemplate_krnl(T *g_idata, T *g_odata, int sh_size,
                                     int N) {
  T *sdata = SharedMemory<T>();

  // load shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < N) {
    // fill shared mem with data
    sdata[tid] = g_idata[i];
  }

  __syncthreads();

  if (i < N) {
    // write result for this block to global mem
    g_odata[i] =
        sin((sdata[tid] - sdata[(tid + 1) % sh_size]) * 2.0f * CARMA_PI);
  }
}

template <class T>
void comp_aotemplate(int threads, int blocks, T *d_idata, T *d_odata, int N) {
  dim3 dimBlock(threads, 1, 1);
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  int smemSize =
      (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);
  comp_aotemplate_krnl<T>
      <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, smemSize, N);

  carma_check_msg("comp_aotemplate_kernel<<<>>> execution failed\n");
}

template void comp_aotemplate<float>(int threads, int blocks, float *d_idata,
                                     float *d_odata, int N);

template void comp_aotemplate<double>(int threads, int blocks, double *d_idata,
                                      double *d_odata, int N);
