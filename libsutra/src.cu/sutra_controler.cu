#include "hip/hip_runtime.h"
#include <sutra_controller.h>


/*
 _  __                    _     
| |/ /___ _ __ _ __   ___| |___ 
| ' // _ \ '__| '_ \ / _ \ / __|
| . \  __/ |  | | | |  __/ \__ \
|_|\_\___|_|  |_| |_|\___|_|___/
                                
 */


__global__ void shift_krnl(float *data, int offset, int N)
{

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    data[tid] = data[tid+offset*N]; 
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void mult_krnl(float *i_data, float *scale, int N)
{

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    i_data[tid] = i_data[tid]*scale[tid]; 
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void mult_int_krnl(float *o_data, float *i_data, float *scale, float gain, int N)
{

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    o_data[tid] = gain *( i_data[tid] * scale[tid]) + o_data[tid]; 
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void mult_int_krnl(float *o_data, float *i_data, float *scale, float gain, int N, int istart)
{

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  tid    += istart;

  if (tid < N) {
    o_data[tid] = gain *( i_data[tid] * scale[tid]) + o_data[tid]; 
  }
}

/*
 _                           _                   
| |    __ _ _   _ _ __   ___| |__   ___ _ __ ___ 
| |   / _` | | | | '_ \ / __| '_ \ / _ \ '__/ __|
| |__| (_| | |_| | | | | (__| | | |  __/ |  \__ \
|_____\__,_|\__,_|_| |_|\___|_| |_|\___|_|  |___/
                                                 
 */


int shift_buf(float *d_data,int offset, int N,int device)
{

  struct hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, device);
    
  int maxThreads = deviceProperties.maxThreadsPerBlock;
  int nBlocks = deviceProperties.multiProcessorCount*8;
  int nThreads = (N + nBlocks -1)/nBlocks;

  if (nThreads > maxThreads) {
    nThreads = maxThreads;
    nBlocks = (N + nThreads  -1)/nThreads;
  }

  dim3 grid(nBlocks), threads(nThreads);

  shift_krnl<<<grid, threads>>>(d_data, offset, N);

  cutilCheckMsg("shift_kernel<<<>>> execution failed\n");
   return EXIT_SUCCESS;
}

int mult_vect(float *d_data,float *scale, int N,int device)
{

  struct hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, device);
    
  int maxThreads = deviceProperties.maxThreadsPerBlock;
  int nBlocks = deviceProperties.multiProcessorCount*8;
  int nThreads = (N + nBlocks -1)/nBlocks;

  if (nThreads > maxThreads) {
    nThreads = maxThreads;
    nBlocks = (N + nThreads  -1)/nThreads;
  }

  dim3 grid(nBlocks), threads(nThreads);

  mult_krnl<<<grid, threads>>>(d_data, scale, N);

  cutilCheckMsg("mult_kernel<<<>>> execution failed\n");
   return EXIT_SUCCESS;
}

int mult_int(float *o_data,float *i_data,float *scale, float gain, int N,int device, carma_streams *streams)
{

  int nthreads = 0,nblocks = 0;

  int nstreams = streams->get_nbStreams();
  getNumBlocksAndThreads(device, N/nstreams, nblocks, nthreads);

  dim3 grid(nblocks), threads(nthreads);

  for(int i = 0; i < nstreams; i++) {
    mult_int_krnl<<<grid, threads, 0, streams->get_stream(i)>>>(o_data,i_data, scale, gain, N, i*nblocks*nthreads);
  cutilCheckMsg("multint_kernel<<<>>> execution failed\n");
  }

   return EXIT_SUCCESS;
}

int mult_int(float *o_data,float *i_data,float *scale, float gain, int N,int device)
{

  int nthreads = 0,nblocks = 0;

  getNumBlocksAndThreads(device, N, nblocks, nthreads);

  dim3 grid(nblocks), threads(nthreads);

  mult_int_krnl<<<grid, threads>>>(o_data,i_data, scale, gain, N);
  cutilCheckMsg("multint_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}
