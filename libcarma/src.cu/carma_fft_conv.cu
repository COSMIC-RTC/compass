#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
//  This file is part of COMPASS <https://anr-compass.github.io/compass/>
//
//  Copyright (C) 2011-2023 COMPASS Team <https://github.com/ANR-COMPASS>
//  All rights reserved.

// -----------------------------------------------------------------------------

//! \file      carma_fft_conv.cu
//! \ingroup   libcarma
//! \brief     this file provides fft convolution CUDA kernels
//! \author    COMPASS Team <https://github.com/ANR-COMPASS>
//! \version   5.5.0
//! \date      2022/01/24

#include <carma_obj.h>
#include <convolutionFFT2D_common.h>

__global__ void fftconv_upadkrnl(float *odata, float *idata, int fftW,
                                 int dataW, int N, int n) {
  __shared__ float cache[BLOCK_SZ][BLOCK_SZ];

  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  // int tid = x + y *blockDim.x * gridDim.x;

  if (y * fftW + x < N)
    cache[BLOCK_SZ - 1 - threadIdx.x][BLOCK_SZ - 1 - threadIdx.y] =
        idata[y * fftW + x];

  __syncthreads();

  if (y * dataW + x < n)
    odata[y * dataW + x] =
        cache[BLOCK_SZ - 1 - threadIdx.x][BLOCK_SZ - 1 - threadIdx.y];
}

int fftconv_unpad_old(float *d_odata, float *d_idata, int fftW, int dataH,
                      int dataW, int N, int n) {
  dim3 blocks(dataH / BLOCK_SZ, dataW / BLOCK_SZ), threads(BLOCK_SZ, BLOCK_SZ);

  fftconv_upadkrnl<<<blocks, threads>>>(d_odata, d_idata, fftW, dataW, N, n);

  return EXIT_SUCCESS;
}

__global__ void unpad_krnl(float *odata, float *idata, int fftW, int dataW,
                           int N, int n, int nim) {
  const int y = blockDim.y * blockIdx.y + threadIdx.y;
  const int x = blockDim.x * blockIdx.x + threadIdx.x;
  const int z = blockDim.z * blockIdx.z + threadIdx.z;

  int kz_src = z * N;
  int kz_dst = z * n;

  if ((y * fftW + x < N) && (z < nim)) {
    odata[y * dataW + x + kz_dst] = idata[y * fftW + x + kz_src];
  }
}

int fftconv_unpad(float *d_odata, float *d_idata, int fftW, int dataH,
                  int dataW, int N, int n, int nim) {
  dim3 threads(16, 8, 8);
  dim3 grid(i_div_up(dataW, threads.x), i_div_up(dataH, threads.y),
            i_div_up(nim, threads.z));

  unpad_krnl<<<grid, threads>>>(d_odata, d_idata, fftW, dataW, N, n, nim);
  carma_check_msg("unpad_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}
