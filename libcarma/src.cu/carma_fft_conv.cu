#include "hip/hip_runtime.h"
// This file is part of COMPASS <https://github.com/COSMIC-RTC/compass>
//
// COMPASS is free software: you can redistribute it and/or modify it under the terms of the GNU
// Lesser General Public License as published by the Free Software Foundation, either version 3 of
// the License, or any later version.
//
// COMPASS is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without
// even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
// Lesser General Public License for more details.
//
// You should have received a copy of the GNU Lesser General Public License along with COMPASS. If
// not, see <https://www.gnu.org/licenses/>
//
//  Copyright (C) 2011-2024 COSMIC Team <https://github.com/COSMIC-RTC/compass>

//! \file      carma_fft_conv.cu
//! \ingroup   libcarma
//! \brief     this file provides fft convolution CUDA kernels
//! \author    COSMIC Team <https://github.com/COSMIC-RTC/compass>
//! \date      2022/01/24

#include <carma_obj.hpp>
#include <convolutionFFT2D_common.h>

__global__ void fftconv_upadkrnl(float *odata, float *idata, int32_t fftW,
                                 int32_t dataW, int32_t N, int32_t n) {
  __shared__ float cache[BLOCK_SZ][BLOCK_SZ];

  int32_t x = threadIdx.x + blockIdx.x * blockDim.x;
  int32_t y = threadIdx.y + blockIdx.y * blockDim.y;
  // int32_t tid = x + y *blockDim.x * gridDim.x;

  if (y * fftW + x < N)
    cache[BLOCK_SZ - 1 - threadIdx.x][BLOCK_SZ - 1 - threadIdx.y] =
        idata[y * fftW + x];

  __syncthreads();

  if (y * dataW + x < n)
    odata[y * dataW + x] =
        cache[BLOCK_SZ - 1 - threadIdx.x][BLOCK_SZ - 1 - threadIdx.y];
}

int32_t fftconv_unpad_old(float *d_odata, float *d_idata, int32_t fftW, int32_t dataH,
                      int32_t dataW, int32_t N, int32_t n) {
  dim3 blocks(dataH / BLOCK_SZ, dataW / BLOCK_SZ), threads(BLOCK_SZ, BLOCK_SZ);

  fftconv_upadkrnl<<<blocks, threads>>>(d_odata, d_idata, fftW, dataW, N, n);

  return EXIT_SUCCESS;
}

__global__ void unpad_krnl(float *odata, float *idata, int32_t fftW, int32_t dataW,
                           int32_t N, int32_t n, int32_t nim) {
  const int32_t y = blockDim.y * blockIdx.y + threadIdx.y;
  const int32_t x = blockDim.x * blockIdx.x + threadIdx.x;
  const int32_t z = blockDim.z * blockIdx.z + threadIdx.z;

  int32_t kz_src = z * N;
  int32_t kz_dst = z * n;

  if ((y * fftW + x < N) && (z < nim)) {
    odata[y * dataW + x + kz_dst] = idata[y * fftW + x + kz_src];
  }
}

int32_t fftconv_unpad(float *d_odata, float *d_idata, int32_t fftW, int32_t dataH,
                  int32_t dataW, int32_t N, int32_t n, int32_t nim) {
  dim3 threads(16, 8, 8);
  dim3 grid(i_div_up(dataW, threads.x), i_div_up(dataH, threads.y),
            i_div_up(nim, threads.z));

  unpad_krnl<<<grid, threads>>>(d_odata, d_idata, fftW, dataW, N, n, nim);
  carma_check_msg("unpad_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}
