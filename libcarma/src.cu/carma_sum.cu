#include "hip/hip_runtime.h"
#include <carma_obj.h>

bool
isPow2(unsigned int x) {
  return ((x & (x - 1)) == 0);
}

unsigned int
nextPow2(unsigned int x) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}

void
sumGetNumBlocksAndThreads(int n, int device, int &blocks, int &threads) {

  struct hipDeviceProp_t deviceProperties;

  hipGetDeviceProperties(&deviceProperties, device);

  int maxThreads = deviceProperties.maxThreadsPerBlock;
  int maxBlocks = deviceProperties.multiProcessorCount;

  threads = (n < maxThreads * 2) ? nextPow2((n + 1) / 2) : maxThreads;
  blocks = (n + (threads * 2 - 1)) / (threads * 2);
  blocks = MIN(maxBlocks, blocks);

}

template<class T>
  struct SharedMemory {
    __device__
    inline
    operator T*() {
      extern __shared__ int __smem[];
      return (T*) __smem;
    }

    __device__
    inline
    operator const T*() const {
      extern __shared__ int __smem[];
      return (T*) __smem;
    }
  };

// specialize for double to avoid unaligned memory 
// access compile errors
template<>
  struct SharedMemory<double> {
    __device__
    inline
    operator double*() {
      extern __shared__ double __smem_d[];
      return (double*) __smem_d;
    }

    __device__
    inline
    operator const double*() const {
      extern __shared__ double __smem_d[];
      return (double*) __smem_d;
    }
  };
/*
 Parallel sum reduction using shared memory
 - takes log(n) steps for n input elements
 - uses n threads
 - only works for power-of-2 arrays
 */

/*
 This version adds multiple elements per thread sequentially.  This reduces the overall
 cost of the algorithm while keeping the work complexity O(n) and the step complexity 
 O(log n).
 (Brent's Theorem optimization)

 Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory. 
 In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.  
 If blockSize > 32, allocate blockSize*sizeof(T) bytes.
 */
// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template<class T, unsigned int blockSize, bool nIsPow2>
  __global__ void
  reduce6(T *g_idata, T *g_odata, unsigned int n) {
    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockSize * 2 + threadIdx.x;
    unsigned int gridSize = blockSize * 2 * gridDim.x;

    T mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n) {
      mySum += g_idata[i];
      // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
      if (nIsPow2 || i + blockSize < n)
        mySum += g_idata[i + blockSize];
      i += gridSize;
    }

    // each thread puts its local sum into shared memory 
    sdata[tid] = mySum;
    __syncthreads();

    // do reduction in shared mem
    if (blockSize >= 1024) {
      if (tid < 512) {
        sdata[tid] = mySum = mySum + sdata[tid + 512];
      }
      __syncthreads();
    }
    if (blockSize >= 512) {
      if (tid < 256) {
        sdata[tid] = mySum = mySum + sdata[tid + 256];
      }
      __syncthreads();
    }
    if (blockSize >= 256) {
      if (tid < 128) {
        sdata[tid] = mySum = mySum + sdata[tid + 128];
      }
      __syncthreads();
    }
    if (blockSize >= 128) {
      if (tid < 64) {
        sdata[tid] = mySum = mySum + sdata[tid + 64];
      }
      __syncthreads();
    }

#ifndef __DEVICE_EMULATION__
    if (tid < 32)
#endif
        {
      // now that we are using warp-synchronous programming (below)
      // we need to declare our shared memory volatile so that the compiler
      // doesn't reorder stores to it and induce incorrect behavior.
      volatile T* smem = sdata;
      if (blockSize >= 64) {
        smem[tid] = mySum = mySum + smem[tid + 32];
        __syncthreads();
      }
      if (blockSize >= 32) {
        smem[tid] = mySum = mySum + smem[tid + 16];
        __syncthreads();
      }
      if (blockSize >= 16) {
        smem[tid] = mySum = mySum + smem[tid + 8];
        __syncthreads();
      }
      if (blockSize >= 8) {
        smem[tid] = mySum = mySum + smem[tid + 4];
        __syncthreads();
      }
      if (blockSize >= 4) {
        smem[tid] = mySum = mySum + smem[tid + 2];
        __syncthreads();
      }
      if (blockSize >= 2) {
        smem[tid] = mySum = mySum + smem[tid + 1];
        __syncthreads();
      }
    }

    // write result for this block to global mem 
    if (tid == 0)
      g_odata[blockIdx.x] = sdata[0];
  }

template<class T>
  void
  reduce(int size, int threads, int blocks, T *d_idata, T *d_odata) {
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps 
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize =
        (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);

    if (isPow2(size)) {
      switch (threads) {
      case 1024:
        reduce6<T, 1024, true> <<<dimGrid, dimBlock, smemSize>>>(d_idata,
            d_odata, size);
        break;
      case 512:
        reduce6<T, 512, true> <<<dimGrid, dimBlock, smemSize>>>(d_idata,
            d_odata, size);
        break;
      case 256:
        reduce6<T, 256, true> <<<dimGrid, dimBlock, smemSize>>>(d_idata,
            d_odata, size);
        break;
      case 128:
        reduce6<T, 128, true> <<<dimGrid, dimBlock, smemSize>>>(d_idata,
            d_odata, size);
        break;
      case 64:
        reduce6<T, 64, true> <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata,
            size);
        break;
      case 32:
        reduce6<T, 32, true> <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata,
            size);
        break;
      case 16:
        reduce6<T, 16, true> <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata,
            size);
        break;
      case 8:
        reduce6<T, 8, true> <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata,
            size);
        break;
      case 4:
        reduce6<T, 4, true> <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata,
            size);
        break;
      case 2:
        reduce6<T, 2, true> <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata,
            size);
        break;
      case 1:
        reduce6<T, 1, true> <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata,
            size);
        break;
      }
    } else {
      switch (threads) {
      case 1024:
        reduce6<T, 1024, false> <<<dimGrid, dimBlock, smemSize>>>(d_idata,
            d_odata, size);
        break;
      case 512:
        reduce6<T, 512, false> <<<dimGrid, dimBlock, smemSize>>>(d_idata,
            d_odata, size);
        break;
      case 256:
        reduce6<T, 256, false> <<<dimGrid, dimBlock, smemSize>>>(d_idata,
            d_odata, size);
        break;
      case 128:
        reduce6<T, 128, false> <<<dimGrid, dimBlock, smemSize>>>(d_idata,
            d_odata, size);
        break;
      case 64:
        reduce6<T, 64, false> <<<dimGrid, dimBlock, smemSize>>>(d_idata,
            d_odata, size);
        break;
      case 32:
        reduce6<T, 32, false> <<<dimGrid, dimBlock, smemSize>>>(d_idata,
            d_odata, size);
        break;
      case 16:
        reduce6<T, 16, false> <<<dimGrid, dimBlock, smemSize>>>(d_idata,
            d_odata, size);
        break;
      case 8:
        reduce6<T, 8, false> <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata,
            size);
        break;
      case 4:
        reduce6<T, 4, false> <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata,
            size);
        break;
      case 2:
        reduce6<T, 2, false> <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata,
            size);
        break;
      case 1:
        reduce6<T, 1, false> <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata,
            size);
        break;
      }
    }

  }

template void
reduce<int>(int size, int threads, int blocks, int *d_idata, int *d_odata);

template void
reduce<float>(int size, int threads, int blocks, float *d_idata,
    float *d_odata);

template void
reduce<double>(int size, int threads, int blocks, double *d_idata,
    double *d_odata);
