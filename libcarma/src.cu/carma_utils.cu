#include "hip/hip_runtime.h"
// This file is part of COMPASS <https://github.com/COSMIC-RTC/compass>
//
// COMPASS is free software: you can redistribute it and/or modify it under the terms of the GNU Lesser
// General Public License as published by the Free Software Foundation, either version 3 of the 
// License, or any later version.
//
// COMPASS is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; 
// without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. 
// See the GNU Lesser General Public License for more details.
//
// You should have received a copy of the GNU Lesser General Public License along with COMPASS. 
// If not, see <https://www.gnu.org/licenses/>
//
//  Copyright (C) 2011-2024 COSMIC Team <https://github.com/COSMIC-RTC/compass>

//! \file      carma_utils.cu
//! \ingroup   libcarma
//! \brief     this file provides utilities CUDA kernels
//! \author    COSMIC Team <https://github.com/COSMIC-RTC/compass>
//! \version   5.5.0
//! \date      2022/01/24

#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include "carma_utils.cuh"
#include "carma_utils.hpp"

template <class T_data>
__global__ void find_nnz_krnl(T_data *d_data, int32_t *colind, int32_t *d_nnz, int32_t N) {
  int32_t *sdata = SharedMemory<int32_t>();
  int32_t tid = threadIdx.x + blockDim.x * blockIdx.x;
  int32_t sid = threadIdx.x;
  if (tid == 0) d_nnz[0] = 0;

  // Load shared memory with 1 if d_data[tid]!= 0, with 0 else
  if (tid < N) {
    sdata[sid] = (d_data[tid] != 0);
    colind[tid] = (sdata[sid]) ? tid : N + tid;  // Init colind for further sort
  } else {
    sdata[sid] = 0;
  }
  __syncthreads();
  reduce_krnl(sdata, blockDim.x, sid);
  __syncthreads();

  if (threadIdx.x == 0)
    //		intensities[blockIdx.x] = sdata[0];
    atomicAdd(d_nnz, sdata[0]);
}
template <class T_data>
int32_t find_nnz(T_data *d_data, int32_t *colind, int32_t N, int32_t *d_nnz, int32_t &h_nnz,
             CarmaDevice *device) {
  int32_t nb_threads = 0, nb_blocks = 0;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);
  dim3 grid(nb_blocks), threads(nb_threads);
  int32_t smemSize = nb_threads * sizeof(int32_t);

  find_nnz_krnl<<<grid, threads, smemSize>>>(d_data, colind, d_nnz, N);
  carma_check_msg("find_nnz_krnl<<<>>> execution failed\n");

  // wrap raw pointer with a device_ptr
  thrust::device_ptr<int32_t> dev_ptr(colind);

  thrust::sort(dev_ptr, dev_ptr + N);
  carma_safe_call(hipMemcpy(&h_nnz, d_nnz, sizeof(int32_t), hipMemcpyDeviceToHost));

  return EXIT_SUCCESS;
}

template int32_t find_nnz<float>(float *d_data, int32_t *colind, int32_t N, int32_t *d_nnz,
                             int32_t &h_nnz, CarmaDevice *device);
template int32_t find_nnz<double>(double *d_data, int32_t *colind, int32_t N, int32_t *d_nnz,
                              int32_t &h_nnz, CarmaDevice *device);

template <class T_data>
__global__ void fill_sparse_vect_krnl(T_data *dense_data, int32_t *colind_sorted,
                                      T_data *values, int32_t *colind, int32_t *rowind,
                                      int32_t nnz) {
  int32_t tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid == 0) rowind[0] = 0;
  if (tid == 1) rowind[1] = nnz;

  // Load shared memory with 1 if d_data[tid]!= 0, with 0 else
  if (tid < nnz) {
    values[tid] = dense_data[colind_sorted[tid]];
    colind[tid] = colind_sorted[tid];
  }
  __syncthreads();
}
template <class T_data>
int32_t fill_sparse_vect(T_data *dense_data, int32_t *colind_sorted, T_data *values,
                     int32_t *colind, int32_t *rowind, int32_t nnz, CarmaDevice *device) {
  int32_t nb_threads = 0, nb_blocks = 0;
  get_num_blocks_and_threads(device, nnz, nb_blocks, nb_threads);
  dim3 grid(nb_blocks), threads(nb_threads);

  fill_sparse_vect_krnl<<<grid, threads>>>(dense_data, colind_sorted, values,
                                           colind, rowind, nnz);
  carma_check_msg("fill_sparse_vect_krnl<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}
template int32_t fill_sparse_vect<float>(float *dense_data, int32_t *colind_sorted,
                                     float *values, int32_t *colind, int32_t *rowind,
                                     int32_t nnz, CarmaDevice *device);
template int32_t fill_sparse_vect<double>(double *dense_data, int32_t *colind_sorted,
                                      double *values, int32_t *colind, int32_t *rowind,
                                      int32_t nnz, CarmaDevice *device);

__global__ void float_to_double_krnl(float *i_data, double *o_data, int32_t N) {
  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid < N) {
    o_data[tid] = (double)i_data[tid];
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void double_to_float_krnl(double *i_data, float *o_data, int32_t N) {
  int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid < N) {
    o_data[tid] = (float)i_data[tid];
    tid += blockDim.x * gridDim.x;
  }
}

int32_t float_to_double(float *i_data, double *o_data, int32_t N, CarmaDevice *device) {
  int32_t nb_threads = 0, nb_blocks = 0;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);
  dim3 grid(nb_blocks), threads(nb_threads);
  float_to_double_krnl<<<grid, threads>>>(i_data, o_data, N);
  carma_check_msg("float_to_double_krnl<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

int32_t double_to_float(double *i_data, float *o_data, int32_t N, CarmaDevice *device) {
  int32_t nb_threads = 0, nb_blocks = 0;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);
  dim3 grid(nb_blocks), threads(nb_threads);
  double_to_float_krnl<<<grid, threads>>>(i_data, o_data, N);
  carma_check_msg("float_to_double_krnl<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

template <typename T>
__global__ void fill_array_krnl(T *d_data, T value, int32_t N) {
  int32_t tid = threadIdx.x + blockDim.x * blockIdx.x;
  while (tid < N) {
    d_data[tid] = value;
    tid += blockDim.x * gridDim.x;
  }
}

template <typename T_data>
int32_t fill_array_with_value(T_data *d_data, T_data value, int32_t N,
                          CarmaDevice *device) {
  int32_t nb_threads = 0, nb_blocks = 0;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);
  dim3 grid(nb_blocks), threads(nb_threads);
  fill_array_krnl<<<grid, threads>>>(d_data, value, N);
  carma_check_msg("fill_array_with_value\n");

  return EXIT_SUCCESS;
}

template int32_t fill_array_with_value<float>(float *d_data, float value, int32_t N,
                                          CarmaDevice *device);
template int32_t fill_array_with_value<double>(double *d_data, double value, int32_t N,
                                           CarmaDevice *device);
template int32_t fill_array_with_value<int32_t>(int32_t *d_data, int32_t value, int32_t N,
                                        CarmaDevice *device);
template int32_t fill_array_with_value<uint32_t>(uint32_t *d_data,
                                                 uint32_t value, int32_t N,
                                                 CarmaDevice *device);
template int32_t fill_array_with_value<uint16_t>(uint16_t *d_data, uint16_t value,
                                             int32_t N, CarmaDevice *device);
template int32_t fill_array_with_value<hipFloatComplex>(hipFloatComplex *d_data,
                                                   hipFloatComplex value, int32_t N,
                                                   CarmaDevice *device);
template int32_t fill_array_with_value<hipDoubleComplex>(hipDoubleComplex *d_data,
                                                    hipDoubleComplex value,
                                                    int32_t N,
                                                    CarmaDevice *device);