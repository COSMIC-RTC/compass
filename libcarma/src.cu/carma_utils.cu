#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
//  This file is part of COMPASS <https://anr-compass.github.io/compass/>
//
//  Copyright (C) 2011-2023 COMPASS Team <https://github.com/ANR-COMPASS>
//  All rights reserved.

// -----------------------------------------------------------------------------

//! \file      carma_utils.cu
//! \ingroup   libcarma
//! \brief     this file provides utilities CUDA kernels
//! \author    COMPASS Team <https://github.com/ANR-COMPASS>
//! \version   5.4.2
//! \date      2022/01/24

#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include "carma_utils.cuh"
#include "carma_utils.h"

template <class T_data>
__global__ void find_nnz_krnl(T_data *d_data, int *colind, int *d_nnz, int N) {
  int *sdata = SharedMemory<int>();
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int sid = threadIdx.x;
  if (tid == 0) d_nnz[0] = 0;

  // Load shared memory with 1 if d_data[tid]!= 0, with 0 else
  if (tid < N) {
    sdata[sid] = (d_data[tid] != 0);
    colind[tid] = (sdata[sid]) ? tid : N + tid;  // Init colind for further sort
  } else {
    sdata[sid] = 0;
  }
  __syncthreads();
  reduce_krnl(sdata, blockDim.x, sid);
  __syncthreads();

  if (threadIdx.x == 0)
    //		intensities[blockIdx.x] = sdata[0];
    atomicAdd(d_nnz, sdata[0]);
}
template <class T_data>
int find_nnz(T_data *d_data, int *colind, int N, int *d_nnz, int &h_nnz,
             CarmaDevice *device) {
  int nb_threads = 0, nb_blocks = 0;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);
  dim3 grid(nb_blocks), threads(nb_threads);
  int smemSize = nb_threads * sizeof(int);

  find_nnz_krnl<<<grid, threads, smemSize>>>(d_data, colind, d_nnz, N);
  carma_check_msg("find_nnz_krnl<<<>>> execution failed\n");

  // wrap raw pointer with a device_ptr
  thrust::device_ptr<int> dev_ptr(colind);

  thrust::sort(dev_ptr, dev_ptr + N);
  carma_safe_call(hipMemcpy(&h_nnz, d_nnz, sizeof(int), hipMemcpyDeviceToHost));

  return EXIT_SUCCESS;
}

template int find_nnz<float>(float *d_data, int *colind, int N, int *d_nnz,
                             int &h_nnz, CarmaDevice *device);
template int find_nnz<double>(double *d_data, int *colind, int N, int *d_nnz,
                              int &h_nnz, CarmaDevice *device);

template <class T_data>
__global__ void fill_sparse_vect_krnl(T_data *dense_data, int *colind_sorted,
                                      T_data *values, int *colind, int *rowind,
                                      int nnz) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid == 0) rowind[0] = 0;
  if (tid == 1) rowind[1] = nnz;

  // Load shared memory with 1 if d_data[tid]!= 0, with 0 else
  if (tid < nnz) {
    values[tid] = dense_data[colind_sorted[tid]];
    colind[tid] = colind_sorted[tid];
  }
  __syncthreads();
}
template <class T_data>
int fill_sparse_vect(T_data *dense_data, int *colind_sorted, T_data *values,
                     int *colind, int *rowind, int nnz, CarmaDevice *device) {
  int nb_threads = 0, nb_blocks = 0;
  get_num_blocks_and_threads(device, nnz, nb_blocks, nb_threads);
  dim3 grid(nb_blocks), threads(nb_threads);

  fill_sparse_vect_krnl<<<grid, threads>>>(dense_data, colind_sorted, values,
                                           colind, rowind, nnz);
  carma_check_msg("fill_sparse_vect_krnl<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}
template int fill_sparse_vect<float>(float *dense_data, int *colind_sorted,
                                     float *values, int *colind, int *rowind,
                                     int nnz, CarmaDevice *device);
template int fill_sparse_vect<double>(double *dense_data, int *colind_sorted,
                                      double *values, int *colind, int *rowind,
                                      int nnz, CarmaDevice *device);

__global__ void float_to_double_krnl(float *i_data, double *o_data, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid < N) {
    o_data[tid] = (double)i_data[tid];
    tid += blockDim.x * gridDim.x;
  }
}

__global__ void double_to_float_krnl(double *i_data, float *o_data, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid < N) {
    o_data[tid] = (float)i_data[tid];
    tid += blockDim.x * gridDim.x;
  }
}

int float_to_double(float *i_data, double *o_data, int N, CarmaDevice *device) {
  int nb_threads = 0, nb_blocks = 0;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);
  dim3 grid(nb_blocks), threads(nb_threads);
  float_to_double_krnl<<<grid, threads>>>(i_data, o_data, N);
  carma_check_msg("float_to_double_krnl<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

int double_to_float(double *i_data, float *o_data, int N, CarmaDevice *device) {
  int nb_threads = 0, nb_blocks = 0;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);
  dim3 grid(nb_blocks), threads(nb_threads);
  double_to_float_krnl<<<grid, threads>>>(i_data, o_data, N);
  carma_check_msg("float_to_double_krnl<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

#ifdef CAN_DO_HALF
__global__ void float_to_half_array_krnl(float *source, half *dest, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid < N) {
    dest[tid] = __float2half(source[tid]);
    tid += blockDim.x * gridDim.x;
  }
}

half *float_to_half_array(float *source, int N, CarmaDevice *device) {
  int nb_threads = 0, nb_blocks = 0;
  half *dest;
  carma_safe_call(hipMalloc((void **)&(dest), sizeof(half) * N));
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);
  dim3 grid(nb_blocks), threads(nb_threads);
  float_to_half_array_krnl<<<grid, threads>>>(source, dest, N);
  carma_check_msg("float_to_half_array_krnl\n");

  return dest;
}

__global__ void half_to_float_array_krnl(half *source, float *dest, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid < N) {
    dest[tid] = __half2float(source[tid]);
    tid += blockDim.x * gridDim.x;
  }
}

float *half_to_float_array(half *source, int N, CarmaDevice *device) {
  int nb_threads = 0, nb_blocks = 0;
  float *dest;
  carma_safe_call(hipMalloc((void **)&(dest), sizeof(float) * N));
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);
  dim3 grid(nb_blocks), threads(nb_threads);
  half_to_float_array_krnl<<<grid, threads>>>(source, dest, N);
  carma_check_msg("half_to_float_array_krnl\n");

  return dest;
}

__global__ void copy_from_float_to_half_krnl(const float *data, half *dest, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid < N) {
    dest[tid] = __float2half(data[tid]);
    tid += blockDim.x * gridDim.x;
  }
}

int copy_from_float_to_half(const float *h_data, half *d_dest, int N,
                        CarmaDevice *device) {
  int nb_threads = 0, nb_blocks = 0;
  float *d_data;
  carma_safe_call(hipMalloc((void **)&d_data, sizeof(float) * N));
  carma_safe_call(
      hipMemcpy(d_data, h_data, sizeof(float) * N, hipMemcpyHostToDevice));

  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);
  dim3 grid(nb_blocks), threads(nb_threads);
  copy_from_float_to_half_krnl<<<grid, threads>>>(d_data, d_dest, N);
  carma_check_msg("copy_from_float_to_half_krnl\n");
  hipFree(d_data);

  return EXIT_SUCCESS;
}

__global__ void copy_from_half_to_float_krnl(const half *data, float *dest, int N) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid < N) {
    dest[tid] = __half2float(data[tid]);
    tid += blockDim.x * gridDim.x;
  }
}

int copy_from_half_to_float(const half *d_data, float *h_dest, int N,
                        CarmaDevice *device) {
  int nb_threads = 0, nb_blocks = 0;
  float *d_dest;
  carma_safe_call(hipMalloc((void **)&d_dest, sizeof(float) * N));
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);
  dim3 grid(nb_blocks), threads(nb_threads);
  copy_from_half_to_float_krnl<<<grid, threads>>>(d_data, d_dest, N);
  carma_check_msg("copy_from_half_to_float_krnl\n");
  carma_safe_call(
      hipMemcpy(h_dest, d_dest, sizeof(float) * N, hipMemcpyDeviceToHost));

  hipFree(d_dest);

  return EXIT_SUCCESS;
}
#endif

template <typename T>
__global__ void fill_array_krnl(T *d_data, T value, int N) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  while (tid < N) {
    d_data[tid] = value;
    tid += blockDim.x * gridDim.x;
  }
}

template <typename T_data>
int fill_array_with_value(T_data *d_data, T_data value, int N,
                          CarmaDevice *device) {
  int nb_threads = 0, nb_blocks = 0;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);
  dim3 grid(nb_blocks), threads(nb_threads);
  fill_array_krnl<<<grid, threads>>>(d_data, value, N);
  carma_check_msg("fill_array_with_value\n");

  return EXIT_SUCCESS;
}

template int fill_array_with_value<float>(float *d_data, float value, int N,
                                          CarmaDevice *device);
template int fill_array_with_value<double>(double *d_data, double value, int N,
                                           CarmaDevice *device);
template int fill_array_with_value<int>(int *d_data, int value, int N,
                                        CarmaDevice *device);
template int fill_array_with_value<unsigned int>(unsigned int *d_data,
                                                 unsigned int value, int N,
                                                 CarmaDevice *device);
template int fill_array_with_value<uint16_t>(uint16_t *d_data, uint16_t value,
                                             int N, CarmaDevice *device);
template int fill_array_with_value<hipFloatComplex>(hipFloatComplex *d_data,
                                                   hipFloatComplex value, int N,
                                                   CarmaDevice *device);
template int fill_array_with_value<hipDoubleComplex>(hipDoubleComplex *d_data,
                                                    hipDoubleComplex value,
                                                    int N,
                                                    CarmaDevice *device);

#ifdef CAN_DO_HALF
template int fill_array_with_value<half>(half *d_data, half value, int N,
                                         CarmaDevice *device);
#endif
