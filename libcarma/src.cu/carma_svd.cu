
#include <hip/hip_runtime.h>
// -----------------------------------------------------------------------------
//  This file is part of COMPASS <https://anr-compass.github.io/compass/>
//
//  Copyright (C) 2011-2023 COMPASS Team <https://github.com/ANR-COMPASS>
//  All rights reserved.

// -----------------------------------------------------------------------------

//! \file      carma_svd.cu
//! \ingroup   libcarma
//! \brief     this file provides SVD CUDA kernels
//! \author    COMPASS Team <https://github.com/ANR-COMPASS>
//! \version   5.4.0
//! \date      2022/01/24

#if 0
#include <carma_svd.h>

__global__ void kernel_setidd(double *d,int N) {
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx<N) {
    d[idx+N*idx] = 1.0;
  }
}

int carma_setidd(double *d,int n) {
  int blockSize = 8;
  int nb_blocks = n / blockSize + (n % blockSize == 0?0:1);

  kernel_setidd <<< nb_blocks, blockSize >>> ((double *)d,n);

  return EXIT_SUCCESS;
}

__global__ void kernel_setidf(float *d,int N) {
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx<N) {
    d[idx+N*idx] = 1.0;
  }
}

int carma_setidf(float *d,int n) {
  int blockSize = 8;
  int nb_blocks = n / blockSize + (n % blockSize == 0?0:1);

  kernel_setidf <<< nb_blocks, blockSize >>> ((float *)d,n);

  return EXIT_SUCCESS;
}
#endif
