#include "hip/hip_runtime.h"
#include <carma_obj.h>

// two variables : blocks and threads (per block)
// can define a grid of blocks (x,y) cf ex 2d
// thread cooperation : inside a block threads can cooperate
// shared memory works with blocks
// allocate shared for a block : __shared__ float cache[threadsPerBlock]
// then it is asigned to a block : cacheIndex = threadIdx.x
// tid moves by the same amount : blockDim.x * gridDim.x
// use __syncthreads() to synchronize
// never put syncthreads in a conditional of thread id
// in case you use shared mem, your limited by threadsPerBlock hence 
// the size of the shared mem => nblocks = (N+threadsPerBlock-1)/threadsPerBlock
// to get size of shmem per block : deviceProperties.sharedMemPerBlock
// for very simple kernels = not limited by shm size : 
// a good optimization could be threadsPerBlock = deviceProperties.maxThreadsPerBlock
// keeping in mind that the minimum number of blocks is given by the 
// number of cuda cores for instance on my gt330M i have 6 MP x 8 cores = 48 cores
// then comes threads per block so if we are not limited by shared mem :
/*
minBlocks = deviceProperties.multiProcessorCount
tmpThreads = N / minBlocks;
if (tmpThreads > deviceProperties.maxThreadsPerBlock) {
nThreads = maxThreadsPerBlock
nBlocks  = (N+threadsPerBlock-1)/threadsPerBlock
} else {
nBlocks = minBlocks;
nThreads = (N + minBlocks - 1)/minBlocks;
}
*/

/*
short	2 bytes	
int	4 bytes	
long	4 bytes	
float	4 bytes	
double	8 bytes	
 */
// for shmem : 1 float = 4 bytes => shmem can contain 
// nb_elem = deviceProperties.sharedMemPerBlock/4 floats
// seems not to work on my laptop ... maybe my x server is already asking a lot ...
// worth a try on blast asap
// on my laptop, have to divide by 4 ...
// 
//

template <class T> __device__ T carma_sin(T data);
template <> __device__ float carma_sin(float data )
{
  return  sinf(data);
}

template <> __device__ double carma_sin(double data )
{
return sin(data);
}


template<class T> __global__ void generic1d(T *odata, T *idata, int N)
{

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    odata[tid] = carma_sin(2.0f * idata[tid]);
    tid += blockDim.x * gridDim.x;
  }
}

template<class T> int launch_generic1d(T *d_odata,T *d_idata,int N)
{

  struct hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, 0);
    
  int maxThreads = deviceProperties.maxThreadsPerBlock;
  int nBlocks = deviceProperties.multiProcessorCount*8;
  int nThreads = (N + nBlocks -1)/nBlocks;

  if (nThreads > maxThreads) {
    nThreads = maxThreads;
    nBlocks = (N + nThreads  -1)/nThreads;
  }

  dim3 grid(nBlocks), threads(nThreads);
  //  dim3 grid(128), threads(128);

  generic1d<<<grid, threads>>>(d_odata, d_idata, N);

   return EXIT_SUCCESS;
}


template int launch_generic1d<float>(float *d_odata,float *d_idata,int N);

template int launch_generic1d<double>(double *d_odata,double *d_idata,int N);


template<class T> __global__ void generic2d(T *odata, T *idata, int N)
{
  __shared__ T cache[BLOCK_SZ][BLOCK_SZ];

  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int tid = x + y *blockDim.x * gridDim.x;

  cache[BLOCK_SZ-1-threadIdx.x][BLOCK_SZ-1-threadIdx.y] =  carma_sin(2.0f * idata[tid]);

  __syncthreads();

  odata[tid] = cache[BLOCK_SZ-1-threadIdx.x][BLOCK_SZ-1-threadIdx.y];
}


template<class T> int launch_generic2d(T *d_odata,T *d_idata,int N1, int N2)
{

  dim3 blocks(N1/BLOCK_SZ,N2/BLOCK_SZ), threads(BLOCK_SZ,BLOCK_SZ);
  int N = N1 * N2;

  generic2d<<<blocks, threads>>>(d_odata, d_idata, N);

   return EXIT_SUCCESS;
}

template int launch_generic2d<float>(float *d_odata,float *d_idata,int N1, int N2);
template int launch_generic2d<double>(double *d_odata,double *d_idata,int N1, int N2);

template<class T> __global__ void krnl_fillindex(T *odata, T *idata, int *indx, int N)
{

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    odata[tid] = idata[indx[tid]];
    tid += blockDim.x * gridDim.x;
  }
}

template<class T> int fillindex(T *d_odata,T *d_idata,int *indx,int N)
{

  struct hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, 0);
    
  int maxThreads = deviceProperties.maxThreadsPerBlock;
  int nBlocks = deviceProperties.multiProcessorCount*8;
  int nThreads = (N + nBlocks -1)/nBlocks;

  if (nThreads > maxThreads) {
    nThreads = maxThreads;
    nBlocks = (N + nThreads  -1)/nThreads;
  }

  dim3 grid(nBlocks), threads(nThreads);
  //  dim3 grid(128), threads(128);

  krnl_fillindex<<<grid, threads>>>(d_odata, d_idata,indx, N);

  return EXIT_SUCCESS;
}


template int fillindex<float>(float *d_odata,float *d_idata,int *indx,int N);

template int fillindex<double>(double *d_odata,double *d_idata,int *indx,int N);

template<class T> __global__ void krnl_fillvalues(T *odata, unsigned int *indx, int N)
{

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    odata[indx[tid]] = 1;
    tid += blockDim.x * gridDim.x;
  }
}

template<class T> int fillvalues(T *d_odata,unsigned int *indx,int N)
{

  struct hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, 0);
    
  int maxThreads = deviceProperties.maxThreadsPerBlock;
  int nBlocks = deviceProperties.multiProcessorCount*8;
  int nThreads = (N + nBlocks -1)/nBlocks;

  if (nThreads > maxThreads) {
    nThreads = maxThreads;
    nBlocks = (N + nThreads  -1)/nThreads;
  }

  dim3 grid(nBlocks), threads(nThreads);
  //  dim3 grid(128), threads(128);

  krnl_fillvalues<<<grid, threads>>>(d_odata, indx, N);

  return EXIT_SUCCESS;
}


template int fillvalues<float>(float *d_odata,unsigned int *indx,int N);

template int fillvalues<double>(double *d_odata,unsigned int *indx,int N);

template int fillvalues<unsigned int>(unsigned int *d_odata,unsigned int *indx,int N);


template<class T> __global__ void getarray2d_krnl(T *odata, T *idata, int tidx0, int Ncol,int NC, int N)
{

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int tidB;

  while (tid < N) {
    if (Ncol > 1) tidB = tidx0 + (tid/Ncol)*NC + (tid%Ncol);
    else tidB = tidx0 + tid*NC;
    odata[tid] = idata[tidB];
    tid += blockDim.x * gridDim.x;
  }
}

template<class T> int getarray2d(T *d_odata,T *d_idata,int x0, int Ncol,int NC, int N)
{

  struct hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, 0);
    
  int maxThreads = deviceProperties.maxThreadsPerBlock;
  int nBlocks = deviceProperties.multiProcessorCount*8;
  int nThreads = (N + nBlocks -1)/nBlocks;

  if (nThreads > maxThreads) {
    nThreads = maxThreads;
    nBlocks = (N + nThreads  -1)/nThreads;
  }

  dim3 grid(nBlocks), threads(nThreads);
  //  dim3 grid(128), threads(128);

  getarray2d_krnl<<<grid, threads>>>(d_odata, d_idata, x0, Ncol,NC,N);

  cutilCheckMsg("getarray2d_kernel<<<>>> execution failed\n");

   return EXIT_SUCCESS;
}


template int getarray2d<float>(float *d_odata,float *d_idata,int x0, int Ncol,int NC, int N);

template int getarray2d<double>(double *d_odata,double *d_idata,int x0, int Ncol,int NC, int N);


template<class T> __global__ void fillarray2d_krnl(T *odata, T *idata, int tidx0, int Ncol,int NC, int N)
{

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int tidB;

  while (tid < N) {
    if (Ncol > 1) tidB = tidx0 + (tid/Ncol)*NC + (tid%Ncol);
    else tidB = tidx0 + tid*NC;
    odata[tidB] = idata[tid];
    tid += blockDim.x * gridDim.x;
  }
}

template<class T> int fillarray2d(T *d_odata,T *d_idata,int x0, int Ncol,int NC, int N)
{

  struct hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, 0);
    
  int maxThreads = deviceProperties.maxThreadsPerBlock;
  int nBlocks = deviceProperties.multiProcessorCount*8;
  int nThreads = (N + nBlocks -1)/nBlocks;

  if (nThreads > maxThreads) {
    nThreads = maxThreads;
    nBlocks = (N + nThreads  -1)/nThreads;
  }

  dim3 grid(nBlocks), threads(nThreads);
  //  dim3 grid(128), threads(128);

  fillarray2d_krnl<<<grid, threads>>>(d_odata, d_idata, x0, Ncol,NC,N);

  cutilCheckMsg("fillarray2d_kernel<<<>>> execution failed\n");

   return EXIT_SUCCESS;
}


template int fillarray2d<float>(float *d_odata,float *d_idata,int x0, int Ncol,int NC, int N);

template int fillarray2d<double>(double *d_odata,double *d_idata,int x0, int Ncol,int NC, int N);

template<class T> __global__ void plus_krnl(T *idata, T alpha, int N)
{

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    idata[tid] += alpha;
    tid += blockDim.x * gridDim.x;
  }
}

template<class T> int carma_plus(T *d_odata,T alpha,int N)
{

  struct hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, 0);
    
  int maxThreads = deviceProperties.maxThreadsPerBlock;
  int nBlocks = deviceProperties.multiProcessorCount*8;
  int nThreads = (N + nBlocks -1)/nBlocks;

  if (nThreads > maxThreads) {
    nThreads = maxThreads;
    nBlocks = (N + nThreads  -1)/nThreads;
  }

  dim3 grid(nBlocks), threads(nThreads);
  //  dim3 grid(128), threads(128);

  plus_krnl<<<grid, threads>>>(d_odata, alpha, N);

  cutilCheckMsg("plus_kernel<<<>>> execution failed\n");

   return EXIT_SUCCESS;
}

template int carma_plus<float>(float *d_odata,float alpha,int N);

template int carma_plus<double>(double *d_odata,double alpha,int N);


template<class T> __global__ void plusai_krnl(T *odata, T* idata, int i, int sgn, int N)
{

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    if (sgn == 1) odata[tid] += idata[i];
    else odata[tid] -= idata[i];
    tid += blockDim.x * gridDim.x;
  }
}

template<class T> int carma_plusai(T *d_odata,T *i_data, int i,int sgn, int N)
{

  struct hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, 0);
    
  int maxThreads = deviceProperties.maxThreadsPerBlock;
  int nBlocks = deviceProperties.multiProcessorCount*8;
  int nThreads = (N + nBlocks -1)/nBlocks;

  if (nThreads > maxThreads) {
    nThreads = maxThreads;
    nBlocks = (N + nThreads  -1)/nThreads;
  }

  dim3 grid(nBlocks), threads(nThreads);
  //  dim3 grid(128), threads(128);

  plusai_krnl<<<grid, threads>>>(d_odata, i_data, i, sgn, N);

  cutilCheckMsg("plusai_kernel<<<>>> execution failed\n");

   return EXIT_SUCCESS;
}

template int carma_plusai<float>(float *d_odata,float *d_idata, int i,int sgn, int N);

template int carma_plusai<double>(double *d_odata,double *d_idata, int i,int sgn, int N);

template<class T> __global__ void fillarray2d2_krnl(T *odata, T *idata, int tidx0, int Ncol,int NC, int N)
{

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int tidB;

  while (tid < N) {
    if (Ncol > 1) tidB = tidx0 + (tid/Ncol)*NC + (tid%Ncol);
    else tidB = tidx0 + tid*NC;
    odata[tidB] = idata[N-tid-1];
    tid += blockDim.x * gridDim.x;
  }
}

template<class T> int fillarray2d2(T *d_odata,T *d_idata,int x0, int Ncol,int NC, int N)
{

  struct hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, 0);
    
  int maxThreads = deviceProperties.maxThreadsPerBlock;
  int nBlocks = deviceProperties.multiProcessorCount*8;
  int nThreads = (N + nBlocks -1)/nBlocks;

  if (nThreads > maxThreads) {
    nThreads = maxThreads;
    nBlocks = (N + nThreads  -1)/nThreads;
  }

  dim3 grid(nBlocks), threads(nThreads);
  //  dim3 grid(128), threads(128);

  fillarray2d2_krnl<<<grid, threads>>>(d_odata, d_idata, x0, Ncol,NC,N);

  cutilCheckMsg("fillarray2d_kernel<<<>>> execution failed\n");

   return EXIT_SUCCESS;
}


template int fillarray2d2<float>(float *d_odata,float *d_idata,int x0, int Ncol,int NC, int N);

template int fillarray2d2<double>(double *d_odata,double *d_idata,int x0, int Ncol,int NC, int N);

