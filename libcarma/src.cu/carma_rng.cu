#include "hip/hip_runtime.h"
// -----------------------------------------------------------------------------
//  This file is part of COMPASS <https://anr-compass.github.io/compass/>
//
//  Copyright (C) 2011-2023 COMPASS Team <https://github.com/ANR-COMPASS>
//  All rights reserved.

// -----------------------------------------------------------------------------

//! \file      carma_rng.cu
//! \ingroup   libcarma
//! \brief     this file provides RNG CUDA kernels
//! \author    COMPASS Team <https://github.com/ANR-COMPASS>
//! \version   5.4.4
//! \date      2022/01/24


#include <assert.h>
#include <carma_obj.h>
#include <hiprand_normal.h>
#include <hiprand_uniform.h>

#define CARMA_NYI_DEV                 \
  {                                   \
    printf("Method not implemented"); \
    assert(0);                        \
  }

// PRNG init kernel
__global__ void initPRNG(hiprandState *s, int n, int *seed, int offset) {
  const int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id < n) hiprand_init(seed[id], threadIdx.x, offset, &s[id]);
}

int carma_prng_init(int *seed, const int nb_threads, const int nb_blocks,
                    hiprandState *state) {
  dim3 grid(nb_blocks);
  dim3 threads(nb_threads);

  // Initialise RNG
  initPRNG<<<grid, threads>>>(state, nb_threads * nb_blocks, seed, nb_threads);
  carma_check_msg("initRNG<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

template <class T>
__forceinline__ __device__ void carma_curand_uniform_gen(
    hiprandState *state, T (*fct)(hiprandState *), T *res, int n, float beta) {
  const int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  const int delta = blockDim.x * gridDim.x;
  for (int idx = tidx; idx < n; idx += delta)
    res[idx] = beta * res[idx] + fct(&state[tidx]);
}

__forceinline__ __device__ void carma_curand_uniform_dev(hiprandState *s,
                                                         float *d, int n,
                                                         float beta) {
  carma_curand_uniform_gen(s, hiprand_uniform, d, n, beta);
}

__forceinline__ __device__ void carma_curand_uniform_dev(hiprandState *s,
                                                         double *d, int n,
                                                         float beta) {
  carma_curand_uniform_gen(s, hiprand_uniform_double, d, n, beta);
}

template <class T>
__global__ void carma_curand_uniform(hiprandState *s, T *d, int n, float beta) {
  carma_curand_uniform_dev(s, d, n, beta);
}

template <>
__global__ void carma_curand_uniform(hiprandState *s, int *d, int n,
                                     float beta) CARMA_NYI_DEV;

template <>
__global__ void carma_curand_uniform(hiprandState *s, unsigned *d, int n,
                                     float beta) CARMA_NYI_DEV;

template <>
__global__ void carma_curand_uniform(hiprandState *s, hipFloatComplex *d, int n,
                                     float beta) {
  carma_curand_uniform_gen(s, hiprand_uniform, (float *)d, n * 2, beta);
}

template <>
__global__ void carma_curand_uniform(hiprandState *s, hipDoubleComplex *d, int n,
                                     float beta) {
  carma_curand_uniform_gen(s, hiprand_uniform_double, (double *)d, n * 2, beta);
}

template <class T>
__forceinline__ __device__ void carma_curand_normal_gen(hiprandState *state,
                                                        T (*fct)(hiprandState *),
                                                        T *res, int n,
                                                        float alpha,
                                                        float beta) {
  const int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  const int delta = blockDim.x * gridDim.x;
  for (int idx = tidx; idx < n; idx += delta)
    res[idx] = beta * res[idx] + alpha * fct(&state[tidx]);
}

template <class T>
__forceinline__ __device__ void carma_curand_normal_dev(
    hiprandState *s, T *d, int n, float alpha, float beta) CARMA_NYI_DEV

    template <>
    __forceinline__ __device__
    void carma_curand_normal_dev(hiprandState *s, float *d, int n, float alpha,
                                 float beta) {
  carma_curand_normal_gen(s, hiprand_normal, d, n, alpha, beta);
}

template <>
__forceinline__ __device__ void carma_curand_normal_dev(hiprandState *s,
                                                        double *d, int n,
                                                        float alpha,
                                                        float beta) {
  carma_curand_normal_gen(s, hiprand_normal_double, d, n, alpha, beta);
}

template <class T>
__global__ void carma_curand_normal(hiprandState *s, T *d, int n, float alpha,
                                    float beta) {
  carma_curand_normal_dev(s, d, n, alpha, beta);
}

template <class T>
__global__ void carma_curand_poisson(hiprandState *state, T *res, int n) {
  const int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  const int delta = blockDim.x * gridDim.x;
  for (int idx = tidx; idx < n; idx += delta)
    res[idx] = (T)hiprand_poisson(&state[tidx], (double)res[idx]);
}

template __global__ void carma_curand_poisson(hiprandState *s, int *d, int n);
template __global__ void carma_curand_poisson(hiprandState *s, unsigned int *d,
                                              int n);
template __global__ void carma_curand_poisson(hiprandState *s, float *d, int n);
template __global__ void carma_curand_poisson(hiprandState *s, double *d, int n);
template <>
__global__ void carma_curand_poisson(hiprandState *s, hipFloatComplex *d,
                                     int n) CARMA_NYI_DEV template <>
__global__ void carma_curand_poisson(hiprandState *s, hipDoubleComplex *d,
                                     int n) CARMA_NYI_DEV

    /*
    template< class T_data, T_data (*ptr_sqrt)(T_data val),
        T_data (*ptr_log)(T_data val), T_data (*ptr_lgamma)(T_data val),
        T_data (*ptr_tan)(T_data val), T_data (*ptr_floor)(T_data val),
        T_data (*ptr_exp)(T_data val)>
    __global__ void carma_curand_montagn(hiprandState *state, T_data *res, int n)
    { T_data xm; T_data tmp, sq, alxm, g, oldm = (-1.0); T_data em, t, y;

      const int tidx = blockIdx.x * blockDim.x + threadIdx.x;
      const int delta = blockDim.x * gridDim.x;
      for (int idx = tidx; idx < n; idx += delta) {
        xm = res[idx];
        //xm = (T_data)results[idx];
        if (xm > 0.0f) {
          if (xm != oldm) {
            oldm = xm;
            sq = ptr_sqrt(2.0f * xm);
            alxm = ptr_log(xm);
            g = xm * alxm - ptr_lgamma(xm + 1.0f);
          }
          do {
            do {
              tmp = hiprand_uniform(&state[tidx]);
              y = ptr_tan(CARMA_PI * tmp);
              em = sq * y + xm;
            } while (em < 0.0f);
            em = ptr_floor(em);
            t = 0.9f * (1.0 + y * y) * ptr_exp(em * alxm - ptr_lgamma(em + 1.0f)
    - g); tmp = hiprand_uniform(&state[tidx]); } while (tmp > t); } else em =
    0.0f; res[idx] = xm;
      }
    }

    template<float, sqrtf, logf, lgammaf, tanf, floorf, expf>
    __global__ void carma_curand_montagn(hiprandState *state, float *res, int n);
    template<double, sqrt, log, lgamma, tan, floor, exp>
    __global__ void carma_curand_montagn(hiprandState *state, double *res, int
    n);
    */

    template <class T>
    __global__ void carma_curand_montagn_krn(hiprandState *state, T *res,
                                             int n) CARMA_NYI_DEV

    template <>
    __global__
    void carma_curand_montagn_krn(hiprandState *state, float *res, int n) {
  float xm;
  float tmp, sq, alxm, g, oldm = (-1.0);
  float em, t, y;

  const int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  const int delta = blockDim.x * gridDim.x;
  for (int idx = tidx; idx < n; idx += delta) {
    xm = res[idx];
    // xm = (float)results[idx];
    if (xm > 0.0f) {
      if (xm != oldm) {
        oldm = xm;
        sq = sqrtf(2.0f * xm);
        alxm = logf(xm);
        g = xm * alxm - lgammaf(xm + 1.0f);
      }
      do {
        do {
          tmp = hiprand_uniform(&state[tidx]);
          y = tanf(CARMA_PI * tmp);
          em = sq * y + xm;
        } while (em < 0.0f);
        em = floorf(em);
        t = 0.9f * (1.0 + y * y) * expf(em * alxm - lgammaf(em + 1.0f) - g);
        tmp = hiprand_uniform(&state[tidx]);
      } while (tmp > t);
    } else
      em = 0.0f;
    res[idx] = xm;
  }
}

template <>
__global__ void carma_curand_montagn_krn(hiprandState *state, double *res,
                                         int n) {
  double tmp;
  const int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  const int delta = blockDim.x * gridDim.x;
  for (int idx = tidx; idx < n; idx += delta) {
    tmp = hiprand_uniform(&state[tidx]);
    res[idx] = tmp;
  }
}

template <class T>
int carma_curand_montagn(hiprandState *state, T *d_odata, int N,
                         CarmaDevice *device) {
  int nb_blocks, nb_threads;
  get_num_blocks_and_threads(device, N, nb_blocks, nb_threads);

  dim3 grid(nb_blocks), threads(nb_threads);
  //  dim3 grid(128), threads(128);

  carma_curand_montagn_krn<<<grid, threads>>>(state, d_odata, N);

  return EXIT_SUCCESS;
}

template int carma_curand_montagn<float>(hiprandState *state, float *d_odata,
                                         int N, CarmaDevice *device);

template int carma_curand_montagn<double>(hiprandState *state, double *d_odata,
                                          int N, CarmaDevice *device);

template int carma_curand_montagn<hipFloatComplex>(hiprandState *state,
                                                  hipFloatComplex *d_odata,
                                                  int N, CarmaDevice *device);

template int carma_curand_montagn<hipDoubleComplex>(hiprandState *state,
                                                   hipDoubleComplex *d_odata,
                                                   int N, CarmaDevice *device);

template int carma_curand_montagn<int>(hiprandState *state, int *d_odata, int N,
                                       CarmaDevice *device);

template int carma_curand_montagn<unsigned int>(hiprandState *state,
                                                unsigned int *d_odata, int N,
                                                CarmaDevice *device);

template <>
int carma_curand_montagn<uint16_t>(hiprandState *state, uint16_t *d_odata, int N,
                                   CarmaDevice *device) {
  CARMA_NYI_DEV;
  return EXIT_FAILURE;
}

template <class T>
int carma_prng_cu(T *results, const int nb_threads, const int nb_blocks,
                  hiprandState *state, char gtype, int n, float alpha,
                  float beta) {
  // dim3 grid(1);
  dim3 threads(2 * nb_threads);

  if (gtype == 'U')
    carma_curand_uniform<<<nb_blocks, nb_threads>>>(state, results, n, beta);
  if (gtype == 'N')
    carma_curand_normal<<<nb_blocks, nb_threads>>>(state, results, n, alpha, beta);
  if (gtype == 'P') {
    carma_curand_poisson<<<nb_blocks, nb_threads>>>(state, results, n);
  }
  carma_check_msg("PRNG<<<>>> execution failed\n");
  return EXIT_SUCCESS;
}

template int carma_prng_cu(int *results, const int nb_threads, const int nb_blocks,
                           hiprandState *state, char gtype, int n, float alpha,
                           float beta);
template int carma_prng_cu(unsigned int *results, const int nb_threads,
                           const int nb_blocks, hiprandState *state, char gtype,
                           int n, float alpha, float beta);

template int carma_prng_cu(float *results, const int nb_threads,
                           const int nb_blocks, hiprandState *state, char gtype,
                           int n, float alpha, float beta);
template int carma_prng_cu(double *results, const int nb_threads,
                           const int nb_blocks, hiprandState *state, char gtype,
                           int n, float alpha, float beta);
template int carma_prng_cu(hipFloatComplex *results, const int nb_threads,
                           const int nb_blocks, hiprandState *state, char gtype,
                           int n, float alpha, float beta);
template int carma_prng_cu(hipDoubleComplex *results, const int nb_threads,
                           const int nb_blocks, hiprandState *state, char gtype,
                           int n, float alpha, float beta);

template <>
int carma_prng_cu(uint16_t *results, const int nb_threads, const int nb_blocks,
                  hiprandState *state, char gtype, int n, float alpha,
                  float beta) {
  CARMA_NYI_DEV;
  return EXIT_FAILURE;
}
